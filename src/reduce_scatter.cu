#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <cassert>

#include "core.h"
#include "common_kernel.h"
#include "copy_kernel.h"
#include "enqueue.h"
#include "reduce_kernel.h"

/* HIERARCHY
 *
 * The data is split into CHUNKS, and each CHUNK is split into NUM_SUBCHUNKS
 * SUBCHUNKS, where each SUBCHUNK is an independent, complete reduction. Each
 * GPU has a buffer that can fit an entire CHUNK, so that all SUBCHUNKS can be
 * processed without checking that the buffer on the receiving GPU is empty. A
 * SUBCHUNK is split into NUM_GPUS SLICES and each GPU works on a different
 * SLICE at the same time. Before moving on the the next SLICE in the reduction
 * algorithm, the GPU has to check whether it has received the data from the
 * previous GPU it needs for this SLICE. To hide the latency of this
 * communication, each GPU processes all the SLICES of all the SUBCHUNKS in
 * sequence before moving on to the next SLICE. Each SLICE is split into a
 * certain number of UNROLLS (determined by the buffer size) and each thread
 * performs UNROLL_COUNT single-data-element operations inside an UNROLL. As the
 * name suggests, the UNROLL_COUNT operations within an UNROLL are unrolled.
*/

// Number of threads used to perform copies, etc. Must be multiple of 32.
// An additional thread is used to handle threadfences, so the CUDA blocks
// have dimension NUM_THREADS+1.
#define NUM_THREADS     256

// Each thread unrolls the innermost loop of the copy or reduction operations
// to this many single-data-element instructions
#define UNROLL_COUNT    8

#define UNROLL_SIZE     (UNROLL_COUNT * NUM_THREADS)

// To hide the latency associated with the synchronization between different
// subchunks, we interleave the independent subchunks so that more data can be
// transferred while the sync is in progress. This is the number of subchunks
// that are active at the same time
#define NUM_SUBCHUNKS   2

/*
 * numGPUs BLOCKs consisting of recvcount words each
 * BLOCK is split up into NumChunks CHUNKs
 * CHUNK is split up into NUM_SUBCHUNKS SUBCHUNKs
 * SUBCHUNK consists of exactly one SLICE
 * SLICE is most efficiently processed in multiples of UNROLL_SIZE
 *
 * The algorithm has numGPUs steps and each step processes a SLICE (i.e.
 * SUBCHUNK) of a different BLOCK. Only data of the BLOCKs not resident on the
 * GPU need to be communicated, hence (numGPUs - 1) BLOCKs. So the buffer needs
 * to have room for (numGPUs - 1) SLICEs.
 */


// do not encode the subchunk number into the flag, because there is a separate
// flag for each subchunk

// If this is called with STEP, it means that we just finished processing the
// data for step STEP on this GPU, which is the data required on the next GPU
// for step STEP + 1, so we signal the next GPU that its data for step STEP + 1
// is available. This is called by one particular consumer warp and so we select
// the first thread in the warp to set the flag.
#define SIGNAL_NEW_DATA_AVAILABLE(chunk, subchunk, step)                        \
    do {                                                                        \
      args.NextNewDataAvailableFlag[0] =                                        \
          2*((chunk) * args.NumGPUs + (step)) + subchunk + 1;                   \
    } while (0)

// This is called by all producer threads, but only thread 0 spins on the flag,
// all threads synchronize after thread 0 is done spinning.
#define WAIT_FOR_NEW_DATA(chunk, subchunk, step)                                \
    do {                                                                        \
      if (tid == 0) {                                                           \
        Wait([=] {                                                              \
          return ((volatile int *)args.ThisNewDataAvailableFlag)[0] >=          \
              2*((chunk) * args.NumGPUs + (step)) + subchunk - 1;               \
        });                                                                     \
      }                                                                         \
      BAR(sync, 1, NUM_THREADS);                                                \
    } while (0)

// If this is called with CHUNK, it means that this GPU has just finished
// processing the chunk CHUNK and so the previous GPU can start with CHUNK + 1
#define SIGNAL_CHUNK_DONE(chunk, subchunk)                                      \
    do {                                                                        \
      args.PrevChunkDoneFlag[0] = 2*(chunk) + subchunk + 1;                     \
    } while (0)

// This is called by all producer threads, but only thread 0 spins on the flag,
// all threads synchronize after thread 0 is done spinning.
#define WAIT_FOR_CHUNK(chunk, subchunk)                                       \
    do {                                                                      \
      if (tid == 0) {                                                         \
        Wait([=] {                                                            \
          return ((volatile int *)args.ThisChunkDoneFlag)[0] >=               \
              2*(chunk) + subchunk - 1;                                       \
        });                                                                   \
      }                                                                       \
      BAR(sync, 1, NUM_THREADS);                                              \
    } while (0)


__device__ inline void getSliceSizeAndChunkSize(int *sliceSize, int slice,
    int numSlices, int numBigSlices, int numSmallSlices, int bigSliceN,
    int smallSliceN, int lastSliceN) {
  if (slice < numBigSlices) {
    *sliceSize = bigSliceN;
  } else {
    *sliceSize = (slice < numBigSlices + numSmallSlices) ? smallSliceN
        : ((slice == numSlices - 1) ? lastSliceN : 0);
  }

/*  if (threadIdx.x == 0)
    printf("[sliceSize=%d] slice=%d numSlices=%d "
        "numBigSlices=%d numSmallSlices=%d bigSliceN=%d smallSliceN=%d "
        "lastSliceN=%d\n", *sliceSize, slice, numSlices, numBigSlices,
        numSmallSlices, bigSliceN, smallSliceN, lastSliceN);
*/
}

template<typename T>
struct ReduceScatterKernelArgs {
  // general parameters
  int ThisId;
  int NumGPUs;
  int N;
  int * UserFromRing;

  // some pre-computed sizes
  int SliceSize;
  int ChunkSize;
  int NumChunks;

  int BufferSliceStride;
  int BufferMisalignedN;

  // local and remote input, output, and buffer
  const T * __restrict__ ThisInput;
  volatile T * __restrict__ ThisOutput;
  volatile T * __restrict__ ThisBuffer;
  volatile T * __restrict__ NextBuffer;

  // local and remote flags
  volatile int * __restrict__ ThisNewDataAvailableFlag;
  volatile int * __restrict__ NextNewDataAvailableFlag;
  volatile int * __restrict__ ThisChunkDoneFlag;
  volatile int * __restrict__ PrevChunkDoneFlag;
};

__device__ inline int GetBlock(const int index, const int step,
    const int * const userFromRing, const int numGPUs) {
  return userFromRing[(numGPUs + index - 1 - step) % numGPUs];
}

template<int THREADS, int UNROLL, class FUNC, typename T>
__global__ void ReduceScatterKernel(const ReduceScatterKernelArgs<T> args) {
  if (args.N == 0) return;
  int tid = threadIdx.x;

  for (int chunk = 0; chunk < args.NumChunks; ++chunk) {
    // calculate slice size.  for all chunks except (possibly) the last one,
    // this will just be args.SliceSize. For the last one, it may be smaller
    int bigSliceN   = args.SliceSize;
    int smallSliceN = 0;
    int lastSliceN  = 0;
    int numSlices   = NUM_SUBCHUNKS;
    int numBigSlices   = numSlices;
    int numSmallSlices = 0;

    // last chunk
    if ((chunk + 1 == args.NumChunks) && (args.N % args.ChunkSize > 0))
      CalcLastChunk<THREADS, UNROLL, T>(&bigSliceN, &smallSliceN, &lastSliceN,
          &numSlices, &numBigSlices, &numSmallSlices, args.N, args.NumChunks,
          args.ChunkSize);


    // this offset is only applied to Data pointers, not to Buffer pointers,
    // since we only have one buffer per chunk
    int chunkOffset = chunk * args.ChunkSize;

    // step 0: push data to next GPU
    int step = 0;
    int block = GetBlock(args.ThisId, step, args.UserFromRing, args.NumGPUs);
    int blockOffset = chunkOffset + block * args.N;
    int bufferOffset = block * NUM_SUBCHUNKS * args.BufferSliceStride +
        ((block * args.BufferMisalignedN) % alignof(PackType));
    int sliceSize;

    if (tid < NUM_THREADS) {
      for(int s=0; s<NUM_SUBCHUNKS; ++s) {
        getSliceSizeAndChunkSize(&sliceSize, s, numSlices, numBigSlices,
            numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

        WAIT_FOR_CHUNK(chunk, s);
        Copy<UNROLL, THREADS>(
            args.NextBuffer + bufferOffset,
            args.ThisInput + blockOffset,
            sliceSize);
        __syncthreads();
        bufferOffset += sliceSize;
        blockOffset += sliceSize;
      }
    } else { // Is consumer
      for(int s=0; s<NUM_SUBCHUNKS; ++s) {
        __syncthreads();
        SIGNAL_NEW_DATA_AVAILABLE(chunk, s, step);
      }
    }

    // steps j with 0 < j < k - 1, where k = number of GPUs: reduce and copy to
    // next GPU
    for (step = 1; step < args.NumGPUs - 1; ++step) {
      int block = GetBlock(args.ThisId, step, args.UserFromRing, args.NumGPUs);
      int blockOffset = chunkOffset + block * args.N;
      int bufferOffset = block * NUM_SUBCHUNKS * args.BufferSliceStride +
          ((block * args.BufferMisalignedN) % alignof(PackType));

      if (tid < NUM_THREADS) {
        for(int s=0; s<NUM_SUBCHUNKS; ++s) {
            getSliceSizeAndChunkSize(&sliceSize, s, numSlices, numBigSlices,
                numSmallSlices, bigSliceN, smallSliceN, lastSliceN);
          WAIT_FOR_NEW_DATA(chunk, s, step);
          Reduce<UNROLL, THREADS, FUNC>(
              args.NextBuffer + bufferOffset,
              args.ThisBuffer + bufferOffset,
              args.ThisInput + blockOffset,
              sliceSize);
          __syncthreads();
          bufferOffset += sliceSize;
          blockOffset += sliceSize;
        }
      } else {
        for(int s=0; s<NUM_SUBCHUNKS; ++s) {
          __syncthreads();
          SIGNAL_NEW_DATA_AVAILABLE(chunk, s, step);
        }
      }
    }

    // step k - 1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    step = args.NumGPUs - 1;
    block = GetBlock(args.ThisId, step, args.UserFromRing, args.NumGPUs);
    blockOffset = chunkOffset + block * args.N;
    bufferOffset = block * NUM_SUBCHUNKS * args.BufferSliceStride +
        ((block * args.BufferMisalignedN) % alignof(PackType));

    if (tid < NUM_THREADS) {
      int outputOffset = 0;
      for (int s=0; s<NUM_SUBCHUNKS; ++s) {
        getSliceSizeAndChunkSize(&sliceSize, s, numSlices, numBigSlices,
            numSmallSlices, bigSliceN, smallSliceN, lastSliceN);
        WAIT_FOR_NEW_DATA(chunk, s, step);
        Reduce<UNROLL, THREADS, FUNC>(
            args.ThisOutput + (chunkOffset + outputOffset),
            args.ThisBuffer + bufferOffset,
            args.ThisInput + blockOffset,
            sliceSize);
        __syncthreads();
        outputOffset += sliceSize;
        bufferOffset += sliceSize;
        blockOffset += sliceSize;
      }
    } else {
      for (int s=0; s<NUM_SUBCHUNKS; ++s) {
        __syncthreads();
        SIGNAL_NEW_DATA_AVAILABLE(chunk, s, step);

        // signal that chunk is done if this is not the last chunk
        if (chunk + 1 < args.NumChunks) {
          SIGNAL_CHUNK_DONE(chunk, s);
        }
      }
    }
  }

  // wait for the last data to be pushed to us
  if (tid < NUM_THREADS) {
    WAIT_FOR_NEW_DATA(args.NumChunks, NUM_SUBCHUNKS-1, 0);

    if (tid == 0) {
      args.ThisNewDataAvailableFlag[tid] = 0;
      args.ThisChunkDoneFlag[tid] = 0;
    }
  }
}

template<class FUNC, typename T>
ncclResult_t ncclReduceScatterWithTypeAndFunc(const void* sendbuff,
    void* recvbuff, const int recvcount, ncclComm* comm, hipStream_t stream) {
  if (recvcount == 0) {
    return ncclSuccess;
  }
  int index = comm->ncclId;

  int blockSizeInBytes = recvcount * sizeof(T);
  int misalignedBytes = blockSizeInBytes % alignof(uint64_t);

  assert((int)((misalignedBytes / sizeof(T)) * sizeof(T)) == misalignedBytes);

  int misalignedN = misalignedBytes / sizeof(T);
  assert(misalignedN < (int)(sizeof(uint64_t) / sizeof(T)));

  int paddingN = (misalignedN > 0) ? sizeof(uint64_t) / sizeof(T) : 0;

  // There is one slice per GPU, so a slice can be at most bufferN / numGPUs,
  // where bufferN is the number of elements of type T that fit into the buffer.
  // For efficiency, we want the slice size to be a multiple of UNROLL_SIZE
  int bufferN = comm->buffSize / sizeof(T);
  // we only need buffer for k slices and k*k paddings (we need k paddings per
  // block and we have k blocks)
  int bufferNPerSlice = (bufferN - NUM_SUBCHUNKS * comm->nDev * paddingN) /
      (NUM_SUBCHUNKS * comm->nDev);
  int sliceSize = (bufferNPerSlice / UNROLL_SIZE) * UNROLL_SIZE;

  int nextId = (index + 1) % comm->nDev;
  int prevId = (index + comm->nDev - 1) % comm->nDev;

  ReduceScatterKernelArgs<T> args;

  args.ThisId = index;
  args.NumGPUs = comm->nDev;
  args.N = recvcount;

  /* Block j must end up in recvbuff[j], which lives on device with logical
   * index comm->ringFromUser[j]. But the block ordering does not necessarily
   * follow the ring ordering. Hence the order in which a particular GPU
   * processes the different blocks (the correspondence between the step in
   * the reduction algorithm and the block on which a GPU operates in that
   * particular step) is not the same as the ring order.
   *
   * Say we have 4 GPUs and comm->userFromRing = { 1, 2, 0, 3 }. Then there are 4
   * step in the reduction algorithm and block 0 needs to end up device 2,
   * block 1 on device 0, block 2 on device 1, and block 3 needs to end up on
   * device 3. In the last step of the algorithm, each GPU must be processing
   * the block that will end up on that GPU. The blocks that a GPU has to
   * process in the previous steps is determined by the next step because each
   * GPU only hands off data to the next GPU in the ring.
   *
   * In the above example, we get the following table of which block is
   * processed by each GPU in a given step. The columns correspond to the
   * different GPUs while the rows are the steps in the algorithm.
   *
   *      GPU 0   1   2   3
   * step
   *    0     3   1   2   0
   *    1     0   3   1   2
   *    2     2   0   3   1
   *    3     1   2   0   3
   *
   * We note the the rows in the above table are just comm->userFromRing in the last
   * step and the list is cyclicly permuted to the left for each previous
   * step. The columns, which are what the individual GPUs need to know, are
   * comm->userFromRing traversed backwards and starting at index k-1 for GPU k.
   * These columns are what we put into args.BlockVsStep to tell the GPU which
   * block it needs to be processing at a particular step. */
  args.UserFromRing = comm->devUserFromRing;

  args.SliceSize = sliceSize;
  args.ChunkSize = NUM_SUBCHUNKS * args.SliceSize;

  // don't reduce this if we cut the slice size in half below, because if that
  // happens, the last chunk will be larger than the other chunks, and we will
  // need the extra buffer space
  args.BufferSliceStride = args.SliceSize + paddingN;

  args.BufferMisalignedN = misalignedN;

  // avoid a case where we have one or more big chunks and one tiny one
  int remainder = args.N % args.ChunkSize;
  if ((args.N > args.ChunkSize) && (remainder > 0) &&
      (args.N < 5 * args.ChunkSize) && (2 * remainder < args.ChunkSize)) {
    args.SliceSize /= 2;
    args.ChunkSize = NUM_SUBCHUNKS * args.SliceSize;

    // round down so we end up with a big last chunk
    args.NumChunks = args.N / args.ChunkSize;
  } else {
    // round up
    args.NumChunks = (args.N + args.ChunkSize - 1) / args.ChunkSize;
  }

//  printf("sliceSize = %i, chunkSize = %i, numChunks = %i, sliceStride = %i, misalignedN = %i\n", args.SliceSize, args.ChunkSize, args.NumChunks, args.BufferSliceStride, args.BufferMisalignedN);

  args.ThisInput = (const T*)sendbuff;
  args.ThisOutput = (volatile T*)recvbuff;
  args.ThisBuffer = (volatile T*)comm->local[prevId]->buff;
  args.NextBuffer = (volatile T*)comm->remote[nextId]->buff;

  // we need 2 * NUM_SUBCHUNKS flags, so use the first NUM_SUBCHUNKS flags
  // to signal the next GPU that new data is available and the following
  // NUM_SUBCHUNKS to signal the previous GPU that a chunk is finished
  args.ThisNewDataAvailableFlag = comm->local[prevId]->flags;
  args.NextNewDataAvailableFlag = comm->remote[nextId]->flags;
  args.ThisChunkDoneFlag = comm->local[nextId]->flags + 1;
  args.PrevChunkDoneFlag = comm->remote[prevId]->flags + 1;

  ReduceScatterKernel<NUM_THREADS, UNROLL_COUNT, FUNC, T>
      <<<1, NUM_THREADS + NUM_SUBCHUNKS * WARP_SIZE, 0, stream>>>(args);
  return ncclSuccess;
}

template<typename T>
ncclResult_t ncclReduceScatterWithType(const void* sendbuff, void* recvbuff,
    int recvcount, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  switch (op) {
  case ncclSum:
    return ncclReduceScatterWithTypeAndFunc<FuncSum<T>, T>(
        sendbuff, recvbuff, recvcount, comm, stream);
  case ncclProd:
    return ncclReduceScatterWithTypeAndFunc<FuncProd<T>, T>(
        sendbuff, recvbuff, recvcount, comm, stream);
  case ncclMax:
    return ncclReduceScatterWithTypeAndFunc<FuncMax<T>, T>(
        sendbuff, recvbuff, recvcount, comm, stream);
  case ncclMin:
    return ncclReduceScatterWithTypeAndFunc<FuncMin<T>, T>(
        sendbuff, recvbuff, recvcount, comm, stream);
  }
  return ncclInvalidOperation;
}

class ReduceScatterFunctor {
public:
  ncclResult_t operator()(const void* sendbuff, void* recvbuff,
      int recvcount, ncclDataType_t datatype, ncclRedOp_t op, int /*root*/,
      ncclComm* comm, hipStream_t stream) {

    switch (datatype) {
    case ncclChar:
      return ncclReduceScatterWithType<char>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
    case ncclInt:
      return ncclReduceScatterWithType<int>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
#ifdef CUDA_HAS_HALF
    case ncclHalf:
      return ncclReduceScatterWithType<half>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
#endif
    case ncclFloat:
      return ncclReduceScatterWithType<float>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
    case ncclDouble:
      return ncclReduceScatterWithType<double>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
    case ncclInt64:
      return ncclReduceScatterWithType<long long>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
    case ncclUint64:
      return ncclReduceScatterWithType<unsigned long long>(sendbuff, recvbuff, recvcount,
          op, comm, stream);
    }
    return ncclInvalidType;
  }
};

extern "C" DSOGLOBAL
ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff,
    int recvcount, ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm,
    hipStream_t stream) {
  return enqueue(ReduceScatterFunctor(), sendbuff, recvbuff, recvcount,
      datatype, op, 0, comm, stream);
}
