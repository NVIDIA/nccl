#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and boffset for buffer sync
#define NEXT_STEP \
  step++; \
  boffset += sliceSize; \
  if (boffset == buffSize) boffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void ReduceKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  __shared__ DevRing<T> ring;

  LoadRing<THREADS>(args.ring, &ring);
  __syncthreads();

  if (tid == 0) {
    WaitFlag prevCommOp(ring.prevOpCounter, 0);
    WaitFlag nextCommOp(ring.nextOpCounter, 0);
    prevCommOp.wait(args.opIndex);
    nextCommOp.wait(args.opIndex);
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring.recvFlagFromNext, (1-NUM_BUFCHUNKS)*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring.recvFlagFromPrev, 0);
  PostFlag postDoneToPrev(ring.sendFlagToPrev, 0);
  PostFlag postReadyToNext(ring.sendFlagToNext, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = args.nRanks;
  const int rank = ring.userRank[0];
  const int prevRank = ring.userRank[nranks-1];
  const int root = args.root;
  const int buffSize = args.buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int boffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput =  args.ThisOutput;
  T * __restrict__ prevInput = ring.recvBuffer;
  T * __restrict__ nextOutput =  ring.sendBuffer;

  for (int offset = 0; offset < size; offset += sliceSize) {
    int maxOffset = size-offset;
    if (prevRank == root) {
      Prims::Copy(
          thisInput + offset,
          nextOutput + boffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext,
          postReadyToNext);
    } else if (rank == root) {
      Prims::Reduce(
          prevInput  + boffset,
          thisInput + offset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitReadyFromPrev,
          postDoneToPrev);
    } else {
      Prims::ReduceCopy(
          thisInput + offset,
          prevInput + boffset,
          thisOutput + offset,
          nextOutput + boffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);
    }
    NEXT_STEP; // Increases step, boffset
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    if (rank != root) {
      // Wait for last update from next then reset the flag
      waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
      *ring.recvFlagFromNext = 0;
    }

    if (prevRank != root) {
      // reset the flag
      *ring.recvFlagFromPrev = 0;
    }

    incrementOpCounter(&args);
  }
}

#define THREADS 512
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingReduce(const void* sendbuff, void* recvbuff, const int count, const int root,
    ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, root, count, comm);
    LAUNCH_KERNEL(ReduceKernel, THREADS, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template<typename> class RedOp>
class ReduceFunctor {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int root, ncclComm* comm, hipStream_t stream) {
    return RingReduce<RedOp<T>, T>(sendbuff, recvbuff, count, root, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  return enqueue<ReduceFunctor>(sendbuff, recvbuff, count, datatype, op, root, comm, stream);
}

