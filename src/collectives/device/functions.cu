#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "common.h"

__device__ struct ncclShmemData* ncclShmem;

#define NCCL_FUNC5(func, algo, redop, type) \
  NCCL_FUNC_NAME(func, algo, LL,     redop, type), \
  NCCL_FUNC_NAME(func, algo, LL128,  redop, type), \
  NCCL_FUNC_NAME(func, algo, SIMPLE, redop, type)

#define NCCL_FUNC4(func, redop, type) \
  NCCL_FUNC5(func, TREE,    redop, type), \
  NCCL_FUNC5(func, RING,    redop, type), \
  NCCL_FUNC5(func, COLLNET, redop, type)

// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, uint8_t), \
  NCCL_FUNC4(func, redop, int32_t), \
  NCCL_FUNC4(func, redop, uint32_t), \
  NCCL_FUNC4(func, redop, int64_t), \
  NCCL_FUNC4(func, redop, uint64_t), \
  NCCL_FUNC4(func, redop, half), \
  NCCL_FUNC4(func, redop, float), \
  NCCL_FUNC4(func, redop, double), \
  NCCL_FUNC4(func, redop, hip_bfloat16)
#define NCCL_FUNCS3B(func, redop) \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t), \
  NCCL_FUNC4(func, redop, int8_t)

// Must be consistent with ncclRedOp_t
#define NCCL_FUNCS2A(func) \
  NCCL_FUNCS3A(func, Sum ), \
  NCCL_FUNCS3A(func, Prod), \
  NCCL_FUNCS3A(func, Max ), \
  NCCL_FUNCS3A(func, Min )
#define NCCL_FUNCS2B(func) \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum)

// Must be consistent with ncclFunc_t
#define NCCL_FUNCS() { \
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),\
  NCCL_FUNCS2B(Broadcast), \
  NCCL_FUNCS2A(Reduce), \
  NCCL_FUNCS2B(AllGather), \
  NCCL_FUNCS2A(ReduceScatter), \
  NCCL_FUNCS2A(AllReduce) }

// Must be consistent with the ncclFuncSet enum
__device__ ncclKern_t ncclFuncs[1+NCCL_NUM_FUNCTIONS*ncclNumOps*ncclNumTypes*NCCL_NUM_ALGORITHMS*NCCL_NUM_PROTOCOLS] = {
// Don't try to initialize the host shadow copy of this device-side global
// variable. There is no host pointer to a device-side function, which
// confuses clang. This will be fixed in the next clang release.
#if __CUDA_ARCH__
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),
  NCCL_FUNCS2B(Broadcast),
  NCCL_FUNCS2A(Reduce),
  NCCL_FUNCS2B(AllGather),
  NCCL_FUNCS2A(ReduceScatter),
  NCCL_FUNCS2A(AllReduce)
#endif
};

// Workaround for https://reviews.llvm.org/D55580
__device__ void ncclWorkaroundClangD55580() {}
