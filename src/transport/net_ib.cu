#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "nccl.h"
#include "core.h"
#include "socket.h"
#include "net.h"
#include "topo.h"
#include "utils.h"
#include "param.h"

#include <assert.h>
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <poll.h>
#include <sys/types.h>
#include <unistd.h>

#include "ibvwrap.h"

#define USE_RDMA_WRITE 1
#define USE_RDMA_SEND_INLINE 0
#define MAXNAMESIZE 64
static char ncclIbIfName[MAX_IF_NAME_SIZE];
static union socketAddress ncclIbIfAddr;
static int ncclNIbDevs = -1;
struct ncclIbDev {
  int device;
  uint8_t port;
  ibv_context* context;
  char devName[MAXNAMESIZE];
};

#define MAX_IB_PORT 15
struct userIbDev {
  char devName[MAXNAMESIZE];
  uint16_t port_en;
};

#define MAX_IB_DEVS 16
struct ncclIbDev ncclIbDevs[MAX_IB_DEVS];
struct userIbDev userIbDevs[MAX_IB_DEVS];
pthread_mutex_t ncclIbLock = PTHREAD_MUTEX_INITIALIZER;

NCCL_PARAM(IbGidIndex, "IB_GID_INDEX", 0);
NCCL_PARAM(IbTimeout, "IB_TIMEOUT", 14);
NCCL_PARAM(IbRetryCnt, "IB_RETRY_CNT", 7);
NCCL_PARAM(IbSl, "IB_SL", 0);
NCCL_PARAM(IbTc, "IB_TC", 0);

// Allocate memory to be potentially ibv_reg_mr'd. This needs to be
// allocated on separate pages as those pages will be marked DONTFORK
// and if they are shared, that could cause a crash in a child process
static ncclResult_t ncclIbMalloc(void** ptr, size_t size) {
  size_t page_size = sysconf(_SC_PAGESIZE);
  void* p;
  int size_aligned = ROUNDUP(size, page_size);
  int ret = posix_memalign(&p, page_size, size_aligned);
  if (ret != 0) return ncclSystemError;
  memset(p, 0, size);
  *ptr = p;
  return ncclSuccess;
}

pthread_t ncclIbAsyncThread;
static void* ncclIbAsyncThreadMain(void* args) {
  struct ibv_context* context = (struct ibv_context*)args;
  while (1) {
    struct ibv_async_event event;
    if (ncclSuccess != wrap_ibv_get_async_event(context, &event)) { break; }
    char *str;
    if (ncclSuccess != wrap_ibv_event_type_str(&str, event.event_type)) { break; }
    if (event.event_type != IBV_EVENT_COMM_EST)
      WARN("NET/IB : Got async event : %s", str);
    if (ncclSuccess != wrap_ibv_ack_async_event(&event)) { break; }
  }
  return NULL;
}

NCCL_PARAM(IbDisable, "IB_DISABLE", 0);

ncclResult_t ncclIbInit(ncclDebugLogger_t logFunction) {
  if(wrap_ibv_symbols() != ncclSuccess) { return ncclInternalError; }
  if (ncclParamIbDisable()) return ncclInternalError;

  if (ncclNIbDevs == -1) {
    pthread_mutex_lock(&ncclIbLock);
    wrap_ibv_fork_init();
    if (ncclNIbDevs == -1) {
      ncclNIbDevs = 0;
      if (findInterfaces(ncclIbIfName, &ncclIbIfAddr, MAX_IF_NAME_SIZE, 1) != 1) {
        WARN("NET/IB : No IP interface found.");
        return ncclInternalError;
      }
      INFO(NCCL_INIT|NCCL_NET,"NET/IB : Using interface %s for sideband communication", ncclIbIfName);

      // Detect IB cards
      int nIbDevs;
      struct ibv_device** devices;

      // Check if user defined which IB device:port to use
      char* userIbEnv = getenv("NCCL_IB_HCA");
      struct netIf userIfs[MAX_IB_DEVS];
      bool searchNot = userIbEnv && userIbEnv[0] == '^';
      int nUserIfs = parseStringList(userIbEnv, userIfs, MAX_IB_DEVS);

      if (ncclSuccess != wrap_ibv_get_device_list(&devices, &nIbDevs)) return ncclInternalError;

      for (int d=0; d<nIbDevs; d++) {
        struct ibv_context * context;
        if (ncclSuccess != wrap_ibv_open_device(&context, devices[d])) {
          WARN("NET/IB : Unable to open device %s", devices[d]->name);
          continue;
        }
        int found = 0;
        if (context) {
          struct ibv_device_attr devAttr;
          if (ncclSuccess != wrap_ibv_query_device(context, &devAttr)) {
            WARN("NET/IB : Unable to query device %s", devices[d]->name);
            continue;
          }
          for (int port = 1; port <= devAttr.phys_port_cnt; port++) {
            struct ibv_port_attr portAttr;
            if (ncclSuccess != wrap_ibv_query_port(context, port, &portAttr)) {
              WARN("NET/IB : Unable to query port %d", port);
              continue;
            }
            if (portAttr.state != IBV_PORT_ACTIVE) continue;
            if (portAttr.link_layer != IBV_LINK_LAYER_INFINIBAND
                && portAttr.link_layer != IBV_LINK_LAYER_ETHERNET) continue;

            // check against user specified HCAs/ports
            if (! (matchIfList(devices[d]->name, port, userIfs, nUserIfs) ^ searchNot)) {
              continue;
            }
            INFO(NCCL_INIT|NCCL_NET,"NET/IB: [%d] %s:%d/%s ", d, devices[d]->name, port,
                portAttr.link_layer == IBV_LINK_LAYER_INFINIBAND ? "IB" : "RoCE");
            ncclIbDevs[ncclNIbDevs].device = d;
            ncclIbDevs[ncclNIbDevs].port = port;
            ncclIbDevs[ncclNIbDevs].context = context;
            strncpy(ncclIbDevs[ncclNIbDevs].devName, devices[d]->name, MAXNAMESIZE);
            ncclNIbDevs++;
            found++;
            pthread_create(&ncclIbAsyncThread, NULL, ncclIbAsyncThreadMain, context);
          }

          if (found == 0) { if (ncclSuccess != wrap_ibv_close_device(context)) { return ncclInternalError; } }
        }
      }
      if (nIbDevs && (ncclSuccess != wrap_ibv_free_device_list(devices))) { return ncclInternalError; };
    }
    pthread_mutex_unlock(&ncclIbLock);
  }
  return ncclSuccess;
}

ncclResult_t ncclIbDevices(int* ndev) {
  *ndev = ncclNIbDevs;
  return ncclSuccess;
}

ncclResult_t ncclIbPciPath(int dev, char** path) {
  char devicepath[PATH_MAX];
  snprintf(devicepath, PATH_MAX, "/sys/class/infiniband/%s/device", ncclIbDevs[dev].devName);
  *path = realpath(devicepath, NULL);
  if (*path == NULL) {
    WARN("Could not find real path of %s", devicepath);
    return ncclSystemError;
  }
  return ncclSuccess;
}

// Detect whether GDR can work on a given NIC with the current CUDA device
// Returns :
// ncclSuccess : GDR works
// ncclSystemError : no module or module loaded but not supported by GPU
ncclResult_t ncclIbGdrSupport(int ibDev) {
  static int moduleLoaded = -1;
  if (moduleLoaded == -1) {
    moduleLoaded = (access("/sys/kernel/mm/memory_peers/nv_mem/version", F_OK) == -1) ? 0 : 1;
  }
  if (moduleLoaded == 0) return ncclSystemError;
  ncclResult_t ret = ncclSystemError;
  void* ptr;
  if (hipMalloc(&ptr, sizeof(int)) == hipSuccess) {
    struct ibv_mr* mr;
    struct ibv_pd* pd;
    if (wrap_ibv_alloc_pd(&pd, ncclIbDevs[ibDev].context) == ncclSuccess) {
      if ((mr = wrap_direct_ibv_reg_mr(pd, ptr, sizeof(int), IBV_ACCESS_LOCAL_WRITE|IBV_ACCESS_REMOTE_WRITE|IBV_ACCESS_REMOTE_READ)) != NULL) {
        ret = ncclSuccess;
        wrap_ibv_dereg_mr(mr);
      }
      wrap_ibv_dealloc_pd(pd);
    }
    hipFree(ptr);
  }
  return ret;
}

ncclResult_t ncclIbPtrSupport(int dev, int* supportedTypes) {
  *supportedTypes = NCCL_PTR_HOST;

  int cudaDev;
  CUDACHECK(hipGetDevice(&cudaDev));

  if (ncclIbGdrSupport(dev) != ncclSuccess) {
    INFO(NCCL_INIT|NCCL_NET,"NET/IB : GPU Direct RDMA Disabled for GPU %d / HCA %s (no module or not supported by GPU)", cudaDev, ncclIbDevs[dev].devName);
    return ncclSuccess;
  }
  *supportedTypes |= NCCL_PTR_CUDA;
  return ncclSuccess;
}

static ncclResult_t GetSocketAddr(union socketAddress* addr) {
  memcpy(addr, &ncclIbIfAddr, sizeof(*addr));
  return ncclSuccess;
}

#define MAX_REQUESTS 128

struct ncclIbQpInfo {
  uint32_t lid;
  uint8_t ib_port;
  uint32_t qpn;

  // For RoCE
  uint64_t spn;
  uint64_t iid;
  enum ibv_mtu mtu;

  // FIFO RDMA info
  uint32_t fifoRkey;
  uint64_t fifoAddr;
};

struct ncclIbHandle {
  union socketAddress connectAddr;
};

struct ncclIbMr {
  struct ibv_mr* mr;
  int refcnt;
};

struct ncclIbVerbs {
  struct ibv_pd* pd;
  struct ibv_cq* cq;
  struct ncclIbMr mrPool[MAX_REQUESTS];
  int mrRotation;
};

struct ncclIbRequest {
  int used;
  int type;
  struct ncclIbVerbs* verbs;
  struct ncclIbMr * ibMr;
  int done;
  int size;
  int free;
};

struct ncclIbListenComm {
  int dev;
  int fd;
};

struct ncclIbSendFifo {
  uint64_t addr;
  int      size;
  uint32_t seq;
  uint32_t rkey;
  uint32_t ready;
};

struct ncclIbSendComm {
  struct ncclIbSendFifo fifo[MAX_REQUESTS];
  struct ncclIbRequest reqs[MAX_REQUESTS];
  uint32_t fifoHead;
  int fd;
  int ready;
  struct ncclIbVerbs verbs;
  struct ibv_qp* qp;
  struct ibv_mr* fifoMr;
};

struct ncclIbGpuFlush {
  int enabled;
  int hostMem;
  struct ibv_mr* hostMr;
  struct ibv_sge sge;
  struct ibv_qp* qp;
};

struct ncclIbRemFifo {
  struct ncclIbSendFifo elems[MAX_REQUESTS];
  uint64_t addr;
  uint32_t rkey;
  uint32_t tail;
  uint32_t flags;
  struct ibv_mr* mr;
  struct ibv_sge sge;
};

struct ncclIbRecvComm {
  struct ncclIbRemFifo remFifo;
  struct ncclIbRequest reqs[MAX_REQUESTS];
  int fd;
  int ready;
  struct ncclIbVerbs verbs;
  struct ibv_qp* qp;
  struct ncclIbGpuFlush gpuFlush;
};

ncclResult_t ncclIbInitVerbs(ibv_context* ctx, struct ncclIbVerbs* verbs) {
  NCCLCHECK(wrap_ibv_alloc_pd(&verbs->pd, ctx));
  NCCLCHECK(wrap_ibv_create_cq(&verbs->cq, ctx, MAX_REQUESTS, NULL, NULL, 0));
  return ncclSuccess;
}

ncclResult_t ncclIbDestroyVerbs(struct ncclIbVerbs* verbs) {
  NCCLCHECK(wrap_ibv_destroy_cq(verbs->cq));
  NCCLCHECK(wrap_ibv_dealloc_pd(verbs->pd));
  return ncclSuccess;
}

ncclResult_t ncclIbCreateQp(uint8_t ib_port, struct ncclIbVerbs* verbs, int access_flags, struct ibv_qp** qp) {
  struct ibv_qp_init_attr qpInitAttr;
  memset(&qpInitAttr, 0, sizeof(struct ibv_qp_init_attr));
  qpInitAttr.send_cq = verbs->cq;
  qpInitAttr.recv_cq = verbs->cq;
  qpInitAttr.qp_type = IBV_QPT_RC;
  qpInitAttr.cap.max_send_wr = MAX_REQUESTS;
  qpInitAttr.cap.max_recv_wr = MAX_REQUESTS;
  qpInitAttr.cap.max_send_sge = 1;
  qpInitAttr.cap.max_recv_sge = 1;
  qpInitAttr.cap.max_inline_data = 0;
  NCCLCHECK(wrap_ibv_create_qp(qp, verbs->pd, &qpInitAttr));
  struct ibv_qp_attr qpAttr;
  memset(&qpAttr, 0, sizeof(struct ibv_qp_attr));
  qpAttr.qp_state = IBV_QPS_INIT;
  qpAttr.pkey_index = 0;
  qpAttr.port_num = ib_port;
  qpAttr.qp_access_flags = access_flags;
  NCCLCHECK(wrap_ibv_modify_qp(*qp, &qpAttr, IBV_QP_STATE | IBV_QP_PKEY_INDEX | IBV_QP_PORT | IBV_QP_ACCESS_FLAGS));
  return ncclSuccess;
}

ncclResult_t ncclIbRtrQp(ibv_qp* qp, struct ncclIbQpInfo* info) {
  struct ibv_qp_attr qpAttr;
  memset(&qpAttr, 0, sizeof(struct ibv_qp_attr));
  qpAttr.qp_state = IBV_QPS_RTR;
  qpAttr.path_mtu = info->mtu;
  qpAttr.dest_qp_num = info->qpn;
  qpAttr.rq_psn = 0;
  qpAttr.max_dest_rd_atomic = 1;
  qpAttr.min_rnr_timer = 12;
  if (info->lid == 0) {
    qpAttr.ah_attr.is_global = 1;
    qpAttr.ah_attr.grh.dgid.global.subnet_prefix = info->spn;
    qpAttr.ah_attr.grh.dgid.global.interface_id = info->iid;
    qpAttr.ah_attr.grh.flow_label = 0;
    qpAttr.ah_attr.grh.sgid_index = ncclParamIbGidIndex();
    qpAttr.ah_attr.grh.hop_limit = 255;
    qpAttr.ah_attr.grh.traffic_class = ncclParamIbTc();
  } else {
    qpAttr.ah_attr.is_global = 0;
    qpAttr.ah_attr.dlid = info->lid;
  }
  qpAttr.ah_attr.sl = ncclParamIbSl();
  qpAttr.ah_attr.src_path_bits = 0;
  qpAttr.ah_attr.port_num = info->ib_port;
  NCCLCHECK(wrap_ibv_modify_qp(qp, &qpAttr, IBV_QP_STATE | IBV_QP_AV | IBV_QP_PATH_MTU | IBV_QP_DEST_QPN | IBV_QP_RQ_PSN | IBV_QP_MAX_DEST_RD_ATOMIC | IBV_QP_MIN_RNR_TIMER));
  return ncclSuccess;
}

ncclResult_t ncclIbRtsQp(ibv_qp* qp) {
  struct ibv_qp_attr qpAttr;
  memset(&qpAttr, 0, sizeof(struct ibv_qp_attr));
  qpAttr.qp_state = IBV_QPS_RTS;
  qpAttr.timeout = ncclParamIbTimeout();
  qpAttr.retry_cnt = ncclParamIbRetryCnt();
  qpAttr.rnr_retry = 7;
  qpAttr.sq_psn = 0;
  qpAttr.max_rd_atomic = 1;
  NCCLCHECK(wrap_ibv_modify_qp(qp, &qpAttr, IBV_QP_STATE | IBV_QP_TIMEOUT | IBV_QP_RETRY_CNT | IBV_QP_RNR_RETRY | IBV_QP_SQ_PSN | IBV_QP_MAX_QP_RD_ATOMIC));
  return ncclSuccess;
}


ncclResult_t ncclIbListen(int dev, void* opaqueHandle, void** listenComm) {
  struct ncclIbListenComm* comm;
  NCCLCHECK(ncclCalloc(&comm, 1));
  struct ncclIbHandle* handle = (struct ncclIbHandle*) opaqueHandle;
  static_assert(sizeof(struct ncclIbHandle) < NCCL_NET_HANDLE_MAXSIZE, "ncclIbHandle size too large");
  comm->dev = dev;
  NCCLCHECK(GetSocketAddr(&(handle->connectAddr)));
  NCCLCHECK(createListenSocket(&comm->fd, &handle->connectAddr));
  *listenComm = comm;
  return ncclSuccess;
}

ncclResult_t ncclIbConnect(int dev, void* opaqueHandle, void** sendComm) {
  struct ncclIbSendComm* comm;
  NCCLCHECK(ncclIbMalloc((void**)&comm, sizeof(struct ncclIbSendComm)));

  struct ncclIbHandle* handle = (struct ncclIbHandle*) opaqueHandle;
  NCCLCHECK(connectAddress(&comm->fd, &handle->connectAddr));
  *sendComm = comm;

  // IB Setup
  ibv_context* ctx = ncclIbDevs[dev].context;
  NCCLCHECK(ncclIbInitVerbs(ctx, &comm->verbs));
  uint8_t ib_port = ncclIbDevs[dev].port;
  NCCLCHECK(ncclIbCreateQp(ib_port, &comm->verbs, IBV_ACCESS_REMOTE_WRITE, &comm->qp));

  // Send my QP Info to receiver through the socket. Hope this won't block.
  struct ibv_port_attr portAttr;
  NCCLCHECK(wrap_ibv_query_port(ctx, ib_port, &portAttr));
  struct ncclIbQpInfo qpInfo;
  qpInfo.ib_port = ib_port;
  qpInfo.qpn = comm->qp->qp_num;
  qpInfo.mtu = portAttr.active_mtu;

  // Prepare my fifo
  NCCLCHECK(wrap_ibv_reg_mr(&comm->fifoMr, comm->verbs.pd, comm->fifo, sizeof(struct ncclIbSendFifo)*MAX_REQUESTS, IBV_ACCESS_LOCAL_WRITE|IBV_ACCESS_REMOTE_WRITE|IBV_ACCESS_REMOTE_READ));
  qpInfo.fifoRkey = comm->fifoMr->rkey;
  qpInfo.fifoAddr = (uint64_t)comm->fifo;

  // RoCE support
  qpInfo.lid = portAttr.lid;
  if (qpInfo.lid) { // IB
    INFO(NCCL_INIT|NCCL_NET,"NET/IB: Dev %d Port %d qpn %d mtu %d LID %d", dev, ib_port, qpInfo.qpn, qpInfo.mtu, qpInfo.lid);
  } else { // RoCE
    union ibv_gid gid;
    NCCLCHECK(wrap_ibv_query_gid(ctx, ib_port, ncclParamIbGidIndex(), &gid));
    qpInfo.spn = gid.global.subnet_prefix;
    qpInfo.iid = gid.global.interface_id;
    INFO(NCCL_INIT|NCCL_NET,"NET/IB: Dev %d Port %d qpn %d mtu %d GID %ld (%lX/%lX)", dev, ib_port, qpInfo.qpn, qpInfo.mtu, ncclParamIbGidIndex(), qpInfo.spn, qpInfo.iid);
  }

  NCCLCHECK(socketSend(comm->fd, &qpInfo, sizeof(qpInfo)));
  return ncclSuccess;
}

NCCL_PARAM(IbGdrFlushDisable, "GDR_FLUSH_DISABLE", 0);

ncclResult_t ncclIbAccept(void* listenComm, void** recvComm) {
  struct ncclIbListenComm* lComm = (struct ncclIbListenComm*)listenComm;
  struct ncclIbRecvComm* rComm;
  NCCLCHECK(ncclIbMalloc((void**)&rComm, sizeof(struct ncclIbRecvComm)));

  struct sockaddr_in sockaddr;
  socklen_t socklen = sizeof(struct sockaddr_in);
  SYSCHECKVAL(accept(lComm->fd, (struct sockaddr*)&sockaddr, &socklen), "accept", rComm->fd);
  struct ncclIbQpInfo remQpInfo;
  NCCLCHECK(socketReceive(rComm->fd, &remQpInfo, sizeof(remQpInfo)));

  // IB setup
  ibv_context* ctx = ncclIbDevs[lComm->dev].context;
  uint8_t ib_port = ncclIbDevs[lComm->dev].port;
  struct ibv_port_attr portAttr;
  NCCLCHECK(wrap_ibv_query_port(ctx, ib_port, &portAttr));
  union ibv_gid gid;
  NCCLCHECK(wrap_ibv_query_gid(ctx, ib_port, ncclParamIbGidIndex(), &gid));

  // QP Creation
  NCCLCHECK(ncclIbInitVerbs(ctx, &rComm->verbs));
  NCCLCHECK(ncclIbCreateQp(ib_port, &rComm->verbs, IBV_ACCESS_REMOTE_WRITE, &rComm->qp));

  // Adjust the MTU
  remQpInfo.mtu = (enum ibv_mtu)std::min(remQpInfo.mtu, portAttr.active_mtu);

  // Setup QP
  struct ibv_qp* qp = rComm->qp;
  NCCLCHECK(ncclIbRtrQp(qp, &remQpInfo));
  NCCLCHECK(ncclIbRtsQp(qp));

  // Retain remote fifo info and prepare my RDMA ops
  rComm->remFifo.rkey = remQpInfo.fifoRkey;
  rComm->remFifo.addr = remQpInfo.fifoAddr;
  NCCLCHECK(wrap_ibv_reg_mr(&rComm->remFifo.mr, rComm->verbs.pd, &rComm->remFifo.elems, sizeof(struct ncclIbSendFifo)*MAX_REQUESTS, IBV_ACCESS_REMOTE_WRITE|IBV_ACCESS_LOCAL_WRITE|IBV_ACCESS_REMOTE_READ));
  rComm->remFifo.sge.length = sizeof(struct ncclIbSendFifo);
  rComm->remFifo.sge.lkey = rComm->remFifo.mr->lkey;

#if USE_RDMA_SEND_INLINE
  // Determine whether the remFifo element data can be sent INLINE
  struct ibv_qp_attr attr;
  struct ibv_qp_init_attr init_attr;
  NCCLCHECK(wrap_ibv_query_qp(qp, &attr, IBV_QP_CAP, &init_attr));
  if (init_attr.cap.max_inline_data >= rComm->remFifo.sge.length) rComm->remFifo.flags = IBV_SEND_INLINE;
#endif

  // Allocate Flush dummy buffer for GPU Direct RDMA
  rComm->gpuFlush.enabled = (ncclIbGdrSupport(lComm->dev) == 0) && (ncclParamIbGdrFlushDisable() == 0) ? 1 : 0;
  if (rComm->gpuFlush.enabled) {
    NCCLCHECK(wrap_ibv_reg_mr(&rComm->gpuFlush.hostMr, rComm->verbs.pd, &rComm->gpuFlush.hostMem, sizeof(int), IBV_ACCESS_LOCAL_WRITE));
    rComm->gpuFlush.sge.addr = (uint64_t)&rComm->gpuFlush.hostMem;
    rComm->gpuFlush.sge.length = 1;
    rComm->gpuFlush.sge.lkey = rComm->gpuFlush.hostMr->lkey;
    NCCLCHECK(ncclIbCreateQp(ib_port, &rComm->verbs, IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_READ, &rComm->gpuFlush.qp));
    struct ncclIbQpInfo localQpInfo = {
      .lid=portAttr.lid,
      .ib_port=ib_port,
      .qpn=rComm->gpuFlush.qp->qp_num,
      .spn=gid.global.subnet_prefix,
      .iid=gid.global.interface_id,
      .mtu=portAttr.active_mtu
    };
    NCCLCHECK(ncclIbRtrQp(rComm->gpuFlush.qp, &localQpInfo));
    NCCLCHECK(ncclIbRtsQp(rComm->gpuFlush.qp));
  }

  // Fill Handle
  struct ncclIbQpInfo qpInfo = {
    .lid=portAttr.lid,
    .ib_port=ib_port,
    .qpn=qp->qp_num,
    .spn=gid.global.subnet_prefix,
    .iid=gid.global.interface_id,
    .mtu=remQpInfo.mtu
  };

  NCCLCHECK(socketSend(rComm->fd, &qpInfo, sizeof(qpInfo)));
  *recvComm = rComm;
  return ncclSuccess;
}

ncclResult_t ncclIbGetRequest(struct ncclIbRequest* reqs, struct ncclIbRequest** req) {
  for (int i=0; i<MAX_REQUESTS; i++) {
    struct ncclIbRequest* r = reqs+i;
    if (r->used == 0) {
      r->used = 1;
      r->type = 0;
      r->verbs = NULL;
      r->ibMr = NULL;
      r->done = 0;
      r->size = -1;
      r->free = 0;
      *req = r;
      return ncclSuccess;
    }
  }
  WARN("NET/IB : unable to allocate requests");
  *req = NULL;
  return ncclInternalError;
}

ncclResult_t ncclSendCheck(struct ncclIbSendComm* comm) {
  struct ncclIbQpInfo remQpInfo;
  struct ibv_qp* qp = comm->qp;

  // Do not block on this receive, return if not ready.
  int bytes = 0;
  NCCLCHECK(socketProgress(NCCL_SOCKET_RECV, comm->fd, &remQpInfo, sizeof(remQpInfo), &bytes));
  if (bytes == 0) return ncclSuccess; // Try again later
  NCCLCHECK(socketWait(NCCL_SOCKET_RECV, comm->fd, &remQpInfo, sizeof(remQpInfo), &bytes));

  NCCLCHECK(ncclIbRtrQp(qp, &remQpInfo));
  NCCLCHECK(ncclIbRtsQp(qp));
  comm->ready = 1;

  // Block until this is done. It *should* not block indefinitely.
  NCCLCHECK(socketSend(comm->fd, &comm->ready, sizeof(int)));

  return ncclSuccess;
}

ncclResult_t ncclRecvCheck(struct ncclIbRecvComm* comm) {
  // Do not block on this receive, return if not ready.
  int bytes = 0;
  NCCLCHECK(socketProgress(NCCL_SOCKET_RECV, comm->fd, &comm->ready, sizeof(int), &bytes));
  if (bytes == 0) return ncclSuccess; // Try again later
  NCCLCHECK(socketWait(NCCL_SOCKET_RECV, comm->fd, &comm->ready, sizeof(int), &bytes));
  return ncclSuccess;
}

ncclResult_t ncclIbTest(void* request, int* done, int* size);

#define REG_ALIGN (4096)

// Cache previous MRs to avoid registering/unregistering for each Isend/Irecv
ncclResult_t ncclIbGetMr(struct ncclIbVerbs* verbs, void* data, int size, struct ncclIbMr** mrRet) {
  uint64_t addr = (uint64_t)data;
  int elem = -1;
  assert(size > 0);

  // Look for an already existing MR
  for (int i=0; i<MAX_REQUESTS; i++) {
    if (verbs->mrPool[i].mr == NULL) continue;
    uint64_t regAddr = (uint64_t)verbs->mrPool[i].mr->addr;
    uint64_t regSize = (uint64_t)verbs->mrPool[i].mr->length;
    if (regAddr <= addr && addr+size <= regAddr+regSize) {
      *mrRet = verbs->mrPool+i;
      verbs->mrPool[i].refcnt++;
      return ncclSuccess;
    }
  }

  // Find an unused element
  if (elem == -1) {
    elem = (verbs->mrRotation++);
    for (int i=0; i<MAX_REQUESTS; i++) {
      elem %= MAX_REQUESTS;
      if (verbs->mrPool[elem].refcnt > 0) elem++; else break;
    }
    if (verbs->mrPool[elem].refcnt > 0) {
      WARN("NET/IB : memory register : no MR available");
      return ncclInternalError;
    }
  }

  assert(elem < MAX_REQUESTS);
  assert(verbs->mrPool[elem].refcnt == 0);

  // Deregister / register
  uint64_t regAddr = addr & (~(REG_ALIGN-1));
  uint64_t regSize = addr+size - regAddr;
  regSize = ((regSize + REG_ALIGN-1) / REG_ALIGN ) * REG_ALIGN;
  if (verbs->mrPool[elem].mr) NCCLCHECK(wrap_ibv_dereg_mr(verbs->mrPool[elem].mr));
  NCCLCHECK(wrap_ibv_reg_mr(&verbs->mrPool[elem].mr, verbs->pd, (void*)regAddr, regSize, IBV_ACCESS_LOCAL_WRITE|IBV_ACCESS_REMOTE_WRITE|IBV_ACCESS_REMOTE_READ));
  *mrRet = verbs->mrPool+elem;
  verbs->mrPool[elem].refcnt++;
  TRACE(NCCL_INIT,"elem %d regAddr %lx size %ld rkey %x", elem, regAddr, regSize, (verbs->mrPool+elem)->mr->rkey);
  return ncclSuccess;
}

ncclResult_t ncclIbIsend(void* sendComm, void* data, int size, int type, void** request) {
  struct ncclIbSendComm* comm = (struct ncclIbSendComm*)sendComm;
  if (comm->ready == 0) NCCLCHECK(ncclSendCheck(comm));
  if (comm->ready == 0) { *request = NULL; return ncclSuccess; }

  // Wait for the receiver to have posted the corresponding receive
  volatile struct ncclIbSendFifo* slot = comm->fifo + (comm->fifoHead%MAX_REQUESTS);
  volatile uint32_t * readyPtr = &slot->ready;
  if (*readyPtr == 0) { *request = NULL; return ncclSuccess; }

  struct ncclIbRequest* req;
  NCCLCHECK(ncclIbGetRequest(comm->reqs, &req));
  req->type = type;
  req->verbs = &comm->verbs;
  req->size = size;

  struct ibv_send_wr wr;
  memset(&wr, 0, sizeof(wr));
  wr.wr_id = (uint64_t)req;

  struct ibv_sge sge;
  if (size == 0) {
    wr.sg_list = NULL;
    wr.num_sge = 0;
  } else {
    NCCLCHECK(ncclIbGetMr(&comm->verbs, data, size, &req->ibMr));
    sge.addr=(uintptr_t)data; sge.length=(unsigned int)size; sge.lkey=req->ibMr->mr->lkey;
    wr.sg_list = &sge;
    wr.num_sge = 1;
  }
  wr.opcode = IBV_WR_SEND;
  wr.send_flags = IBV_SEND_SIGNALED;

#if USE_RDMA_WRITE
  __sync_synchronize(); // order the readyPtr load against rkey load below
  // Sanity checks to catch user collective call count/size mismatches
  // plus any potential programming errors
  if (size > slot->size || slot->size <= 0 || slot->addr == 0 || slot->rkey == 0 || slot->seq != comm->fifoHead) {
    WARN("NET/IB : collective mismatch error local size %d remote %d addr %lx rkey %x seq %x/%x",
        size, slot->size, slot->addr, slot->rkey, slot->seq, comm->fifoHead);
    return ncclInternalError;
  }
  wr.opcode = IBV_WR_RDMA_WRITE_WITH_IMM;
  wr.wr.rdma.remote_addr = slot->addr;
  wr.wr.rdma.rkey = slot->rkey;
  wr.imm_data = size; // Send the message size via imm_data
  __sync_synchronize();
#endif
  // We must clear slot->ready, but reset other fields to aid
  // debugging and sanity checks
  slot->ready = 0;
  slot->addr = 0ULL;
  slot->rkey = slot->size = slot->seq = 0;
  comm->fifoHead++;

  struct ibv_send_wr* bad_wr;
  NCCLCHECK(wrap_ibv_post_send(comm->qp, &wr, &bad_wr));
  *request = req;
  return ncclSuccess;
}

ncclResult_t ncclIbPostFifo(struct ncclIbRecvComm* comm, uint32_t rkey, uint64_t addr, int size) {
  struct ibv_send_wr wr;
  memset(&wr, 0, sizeof(wr));
  struct ncclIbRequest* req;
  NCCLCHECK(ncclIbGetRequest(comm->reqs, &req));
  req->verbs = &comm->verbs;
  req->free = 1; // Not a user req ; free as soon as it is complete.
  wr.wr_id = (uint64_t)req;

  struct ncclIbSendFifo* localElem = comm->remFifo.elems + (comm->remFifo.tail % MAX_REQUESTS);
  localElem->addr = addr;
  localElem->rkey = rkey;
  localElem->ready = 1;
  localElem->size = size; // Sanity/Debugging
  localElem->seq = comm->remFifo.tail; // Sanity/Debugging
  wr.wr.rdma.remote_addr = comm->remFifo.addr + (comm->remFifo.tail % MAX_REQUESTS) * sizeof(struct ncclIbSendFifo);
  wr.wr.rdma.rkey = comm->remFifo.rkey;
  comm->remFifo.sge.addr = (uint64_t)localElem;
  wr.sg_list = &comm->remFifo.sge;
  wr.num_sge = 1;
  wr.opcode = IBV_WR_RDMA_WRITE;
  wr.send_flags = IBV_SEND_SIGNALED | comm->remFifo.flags; // IBV_SEND_INLINE

  struct ibv_send_wr* bad_wr;
  NCCLCHECK(wrap_ibv_post_send(comm->qp, &wr, &bad_wr));
  comm->remFifo.tail++;

  return ncclSuccess;
}

ncclResult_t ncclIbIrecv(void* recvComm, void* data, int size, int type, void** request) {
  struct ncclIbRecvComm* comm = (struct ncclIbRecvComm*)recvComm;
  if (comm->ready == 0) NCCLCHECK(ncclRecvCheck(comm));
  if (comm->ready == 0) { *request = NULL; return ncclSuccess; }

  struct ncclIbRequest* req;
  NCCLCHECK(ncclIbGetRequest(comm->reqs, &req));
  req->type = type;
  req->verbs = &comm->verbs;
  req->size = size;

  struct ibv_recv_wr wr;
  memset(&wr, 0, sizeof(wr));
  wr.wr_id = (uint64_t)req;

  struct ibv_sge sge;
  if (size == 0) {
    wr.sg_list = NULL;
    wr.num_sge = 0;
    req->ibMr = NULL;
  } else {
    NCCLCHECK(ncclIbGetMr(&comm->verbs, data, size, &req->ibMr));
    sge.addr=(uintptr_t)data; sge.length=(unsigned int)size; sge.lkey=req->ibMr->mr->lkey;
    wr.sg_list = &sge;
    wr.num_sge = 1;
  }

  struct ibv_recv_wr* bad_wr;
  NCCLCHECK(wrap_ibv_post_recv(comm->qp, &wr, &bad_wr));
  *request = req;

  // Post to FIFO to notify sender
  NCCLCHECK(ncclIbPostFifo(comm, req->ibMr->mr->rkey, (uint64_t)data, size));
  return ncclSuccess;
}

ncclResult_t ncclIbFlush(void* recvComm, void* data, int size) {
  struct ncclIbRecvComm* comm = (struct ncclIbRecvComm*)recvComm;
  if (comm->gpuFlush.enabled == 0 || size == 0) return ncclSuccess;

  struct ncclIbRequest* req;
  NCCLCHECK(ncclIbGetRequest(comm->reqs, &req));
  req->verbs = &comm->verbs;
  NCCLCHECK(ncclIbGetMr(&comm->verbs, data, 1, &req->ibMr));

  struct ibv_send_wr wr;
  memset(&wr, 0, sizeof(wr));
  wr.wr_id = (uint64_t)req;

  wr.wr.rdma.remote_addr = (uint64_t)data;
  wr.wr.rdma.rkey = req->ibMr->mr->rkey;
  wr.sg_list = &comm->gpuFlush.sge;
  wr.num_sge = 1;
  wr.opcode = IBV_WR_RDMA_READ;
  wr.send_flags = IBV_SEND_SIGNALED;

  struct ibv_send_wr* bad_wr;
  NCCLCHECK(wrap_ibv_post_send(comm->gpuFlush.qp, &wr, &bad_wr));

  int done = 0;
  while (done == 0) {
    NCCLCHECK((ncclResult_t)ncclIbTest(req, &done, NULL));
  }

  return ncclSuccess;
}

ncclResult_t ncclIbTest(void* request, int* done, int* size) {
  struct ncclIbRequest *r = (struct ncclIbRequest*)request;
  *done = 0;

  while (1) {
    if (r->done == 1) {
      *done = 1;
      if (size) *size = r->size;
      r->used = 0;
      return ncclSuccess;
    }

    int wrDone = 0;
    struct ibv_wc wc;
    NCCLCHECK(wrap_ibv_poll_cq(r->verbs->cq, 1, &wc, &wrDone));
    if (wrDone == 0) return ncclSuccess;

    if (wc.status != IBV_WC_SUCCESS) {
      WARN("NET/IB : Got completion with error %d, opcode %d, len %d, vendor err %d", wc.status, wc.opcode, wc.byte_len, wc.vendor_err);
      return ncclSystemError;
    }

    struct ncclIbRequest* doneReq = (struct ncclIbRequest*)wc.wr_id;
    if (doneReq) {
      if (wc.opcode == IBV_WC_RECV) {
        doneReq->size = wc.byte_len;
#if USE_RDMA_WRITE
      } else if (wc.opcode == IBV_WC_RECV_RDMA_WITH_IMM) {
        doneReq->size = wc.imm_data;
#endif
      }
      if (doneReq->ibMr != NULL) {
        doneReq->ibMr->refcnt--;
        if (doneReq->ibMr->refcnt < 0) WARN("NET/IB : doneReq %p MR %p refcount now %d", doneReq, doneReq->ibMr, doneReq->ibMr->refcnt);
      }
      doneReq->done = 1;
      if (doneReq->free == 1) {
        // This is an internal (FIFO post) req. Free it immediately.
        doneReq->used = 0;
      }
    }
  }
}

ncclResult_t ncclIbCloseSend(void* sendComm) {
  struct ncclIbSendComm* comm = (struct ncclIbSendComm*)sendComm;
  if (comm) {
    close(comm->fd);
    if (comm->qp != NULL) NCCLCHECK(wrap_ibv_destroy_qp(comm->qp));
    if (comm->fifoMr != NULL) NCCLCHECK(wrap_ibv_dereg_mr(comm->fifoMr));
    for (int i=0; i<MAX_REQUESTS; i++) {
      if (comm->verbs.mrPool[i].mr != NULL) {
        if (comm->verbs.mrPool[i].refcnt != 0) WARN("NET/IB : TX MR #%d has non-zero (%d) refcnt", i, comm->verbs.mrPool[i].refcnt);
        NCCLCHECK(wrap_ibv_dereg_mr(comm->verbs.mrPool[i].mr));
      }
    }
    NCCLCHECK(ncclIbDestroyVerbs(&comm->verbs));
    free(comm);
  }
  return ncclSuccess;
}

ncclResult_t ncclIbCloseRecv(void* recvComm) {
  struct ncclIbRecvComm* comm = (struct ncclIbRecvComm*)recvComm;
  if (comm) {
    close(comm->fd);
    if (comm->qp != NULL) NCCLCHECK(wrap_ibv_destroy_qp(comm->qp));
    if (comm->gpuFlush.enabled) {
      if (comm->gpuFlush.qp != NULL) NCCLCHECK(wrap_ibv_destroy_qp(comm->gpuFlush.qp));
      if (comm->gpuFlush.hostMr != NULL) NCCLCHECK(wrap_ibv_dereg_mr(comm->gpuFlush.hostMr));
    }
    if (comm->remFifo.mr != NULL) NCCLCHECK(wrap_ibv_dereg_mr(comm->remFifo.mr));
    for (int i=0; i<MAX_REQUESTS; i++) {
      if (comm->verbs.mrPool[i].mr != NULL) {
        if (comm->verbs.mrPool[i].refcnt != 0) WARN("NET/IB : RX MR #%d has non-zero (%d) refcnt", i, comm->verbs.mrPool[i].refcnt);
        NCCLCHECK(wrap_ibv_dereg_mr(comm->verbs.mrPool[i].mr));
      }
    }
    NCCLCHECK(ncclIbDestroyVerbs(&comm->verbs));
    free(comm);
  }
  return ncclSuccess;
}

ncclResult_t ncclIbCloseListen(void* listenComm) {
  struct ncclIbListenComm* comm = (struct ncclIbListenComm*)listenComm;
  if (comm) {
    close(comm->fd);
    free(comm);
  }
  return ncclSuccess;
}

ncclNet_t ncclNetIb = {
  "IB",
  ncclIbInit,
  ncclIbDevices,
  ncclIbPciPath,
  ncclIbPtrSupport,
  ncclIbListen,
  ncclIbConnect,
  ncclIbAccept,
  ncclIbIsend,
  ncclIbIrecv,
  ncclIbFlush,
  ncclIbTest,
  ncclIbCloseSend,
  ncclIbCloseRecv,
  ncclIbCloseListen
};

