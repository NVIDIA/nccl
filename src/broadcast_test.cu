#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "nccl.h"
#include "test_utilities.h"


template<typename T>
void RunTest(T** buff, const int N, const ncclDataType_t type, const int root,
    ncclComm_t* const comms, const std::vector<int>& dList) {
  // initialize data
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  T* buffer = (T*)malloc(N * sizeof(T));
  T* result = (T*)malloc(N * sizeof(T));
  memset(result, 0, N * sizeof(T));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamCreate(s+i));

    if (i == root) {
      Randomize(buff[root], N, root);
      CUDACHECK(hipMemcpy(result, buff[root], N * sizeof(T),
          hipMemcpyDeviceToHost));
    } else {
      CUDACHECK(hipMemset(buff[i], 0, N * sizeof(T)));
    }

    CUDACHECK(hipDeviceSynchronize());
  }

  // warm up GPU
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    ncclBcast((void*)buff[i], std::min(32 * 1024, N), type, root, comms[i], s[i]);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

//  for (int n = 1; n <= N; n = n << 1)
  {
    int n = N;
    printf("%12i  %12i  %6s  %4i", (int)(n * sizeof(T)), n,
        TypeName(type).c_str(), root);

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      ncclBcast((void*)buff[i], n, type, root, comms[i], s[i]);
    }

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }

    auto stop = std::chrono::high_resolution_clock::now();

    double elapsedSec =
        std::chrono::duration_cast<std::chrono::duration<double>>(
            stop - start).count();
    double algbw = (double)(n * sizeof(T)) / 1.0E9  / elapsedSec;
    double busbw = algbw;

    double maxDelta = 0.0;
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      double tmpDelta = CheckDelta<T>(buff[i], result, n);
      maxDelta = std::max(tmpDelta, maxDelta);
    }

    printf("  %7.3f  %5.2f  %5.2f  %7.0le\n", elapsedSec * 1.0E3, algbw, busbw,
            maxDelta);
  }

  for(int i=0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamDestroy(s[i]));
  }
  free(s);
  free(buffer);
  free(result);
}

template<typename T>
void RunTests(const int N, const ncclDataType_t type, ncclComm_t* const comms,
    const std::vector<int>& dList) {
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  T** buff = (T**)malloc(nDev * sizeof(T*));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipMalloc(buff + i, N * sizeof(T)));
  }

  //for (int root = 1; root < 2; ++root) {
  for (int root = 0; root < nDev; ++root) {
    RunTest<T>(buff, N, type, root, comms, dList);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipFree(buff[i]));
  }

  free(buff);
}

void usage() {
  printf("Tests nccl Broadcast with user supplied arguments.\n"
      "    Usage: broadcast_test <data size in bytes> [number of GPUs] "
      "[GPU 0] [GPU 1] ...\n\n");
}

int main(int argc, char* argv[]) {
  int nVis = 0;
  CUDACHECK(hipGetDeviceCount(&nVis));

  unsigned long long N = 0;
  if (argc > 1) {
    int t = sscanf(argv[1], "%llu", &N);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  } else {
    printf("Error: must specify at least data size in bytes!\n\n");
    usage();
    exit(EXIT_FAILURE);
  }

  int nDev = nVis;
  if (argc > 2) {
    int t = sscanf(argv[2], "%d", &nDev);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  }
  std::vector<int> dList(nDev);
  for (int i = 0; i < nDev; ++i)
    dList[i] = i % nVis;

  if (argc > 3) {
    if (argc - 3 != nDev) {
      printf("Error: insufficient number of GPUs in list\n\n");
      usage();
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < nDev; ++i) {
      int t = sscanf(argv[3 + i], "%d", dList.data() + i);
      if (t == 0) {
        printf("Error: %s is not an integer!\n\n", argv[2 + i]);
        usage();
        exit(EXIT_FAILURE);
      }
    }
  }

  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);;
  ncclCommInitAll(comms, nDev, dList.data());

  printf("# Using devices\n");
  for (int g = 0; g < nDev; ++g) {
    int cudaDev;
    int rank;
    hipDeviceProp_t prop;
    ncclCommCuDevice(comms[g], &cudaDev);
    ncclCommUserRank(comms[g], &rank);
    CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
    printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev,
        prop.pciBusID, prop.name);
  }
  printf("\n");

  printf("# %10s  %12s  %6s  %4s  %7s  %5s  %5s  %7s\n",
      "bytes", "N", "type", "root", "time", "algbw", "busbw", "delta");

  RunTests<char>(N / sizeof(char), ncclChar, comms, dList);
  RunTests<int>(N / sizeof(int), ncclInt, comms, dList);
#ifdef CUDA_HAS_HALF
  RunTests<half>(N / sizeof(half), ncclHalf, comms, dList);
#endif
  RunTests<float>(N / sizeof(float), ncclFloat, comms, dList);
  RunTests<double>(N / sizeof(double), ncclDouble, comms, dList);
  RunTests<long long>(N / sizeof(long long), ncclInt64, comms, dList);
  RunTests<unsigned long long>(N / sizeof(unsigned long long), ncclUint64, comms, dList);

  printf("\n");

  for(int i = 0; i < nDev; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  exit(EXIT_SUCCESS);
}

