/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "nccl.h"
#include "core.h"
#include "socket.h"
#include "net.h"

#include <assert.h>
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <poll.h>
#include <limits.h>

/* Init functions */
static char ncclNetIfNames[MAX_IF_NAME_SIZE*MAX_IFS];
static union socketAddress ncclNetIfAddrs[MAX_IFS];
static int ncclNetIfs = -1;
pthread_mutex_t ncclSocketLock = PTHREAD_MUTEX_INITIALIZER;

ncclResult_t ncclSocketInit(ncclDebugLogger_t logFunction) {
  if (ncclNetIfs == -1) {
    pthread_mutex_lock(&ncclSocketLock);
    if (ncclNetIfs == -1) {
      ncclNetIfs = findInterfaces(ncclNetIfNames, ncclNetIfAddrs, MAX_IF_NAME_SIZE, MAX_IFS);
      if (ncclNetIfs <= 0) {
        WARN("NET/Socket : no interface found");
        return ncclInternalError;
      } else {
        char line[1024];
        char addrline[1024];
        line[0] = '\0';
        for (int i=0; i<ncclNetIfs; i++) {
          snprintf(line+strlen(line), 1023-strlen(line), " [%d]%s:%s", i, ncclNetIfNames+i*MAX_IF_NAME_SIZE,
              socketToString(&ncclNetIfAddrs[i].sa, addrline));
        }
        line[1023] = '\0';
        INFO(NCCL_INIT|NCCL_NET,"NET/Socket : Using%s", line);
      }
    }
    pthread_mutex_unlock(&ncclSocketLock);
  }
  return ncclSuccess;
}

ncclResult_t ncclSocketPtrSupport(int dev, int* supportedTypes) {
  *supportedTypes = NCCL_PTR_HOST;
  return ncclSuccess;
}

ncclResult_t ncclSocketDevices(int* ndev) {
  *ndev = ncclNetIfs;
  return ncclSuccess;
}

ncclResult_t ncclSocketPciPath(int dev, char** path) {
  char devicepath[PATH_MAX];
  snprintf(devicepath, PATH_MAX, "/sys/class/net/%s/device", ncclNetIfNames+dev*MAX_IF_NAME_SIZE);
  *path = realpath(devicepath, NULL);
  if (*path == NULL) {
    INFO(NCCL_NET|NCCL_INIT, "Could not find real path of %s", devicepath);
    return ncclSystemError;
  }
  return ncclSuccess;
}

static ncclResult_t GetSocketAddr(int dev, union socketAddress* addr) {
  if (dev >= ncclNetIfs) return ncclInternalError;
  memcpy(addr, ncclNetIfAddrs+dev, sizeof(*addr));
  return ncclSuccess;
}

/* Communication functions */

struct ncclSocketHandle {
  union socketAddress connectAddr;
};

struct ncclSocketRequest {
  int op;
  void* data;
  int size;
  int fd;
  int offset;
  int used;
};

struct ncclSocketReqs {
  struct ncclSocketRequest* requests;
};

struct ncclSocketComm {
  int fd;
  struct ncclSocketReqs reqs;
};

ncclResult_t ncclSocketNewComm(struct ncclSocketComm** comm) {
  NCCLCHECK(ncclCalloc(comm, 1));
  (*comm)->fd = -1;
  return ncclSuccess;
}

ncclResult_t ncclSocketCreateHandle(void* opaqueHandle, const char* str) {
  struct ncclSocketHandle* handle = (struct ncclSocketHandle*) opaqueHandle;
  NCCLCHECK(GetSocketAddrFromString(&(handle->connectAddr), str));
  return ncclSuccess;
}

ncclResult_t ncclSocketListen(int dev, void* opaqueHandle, void** listenComm) {
  struct ncclSocketHandle* handle = (struct ncclSocketHandle*) opaqueHandle;
  static_assert(sizeof(struct ncclSocketHandle) < NCCL_NET_HANDLE_MAXSIZE, "ncclSocketHandle size too large");
  // if dev >= 0, listen based on dev
  if (dev >= 0) {
    NCCLCHECK(GetSocketAddr(dev, &(handle->connectAddr)));
  } else if (dev == findSubnetIf) {
    // handle stores a remote address
    // need to find a local addr that is in the same network as the remote addr
    union socketAddress localAddr;
    char ifName[MAX_IF_NAME_SIZE];
    if (findInterfaceMatchSubnet(ifName, &localAddr, handle->connectAddr, MAX_IF_NAME_SIZE, 1) <= 0) {
      WARN("NET/Socket : No usable listening interface found");
      return ncclSystemError;
    }
    // pass the local address back
    memcpy(&handle->connectAddr, &localAddr, sizeof(handle->connectAddr));
  } // Otherwise, handle stores a local address
  struct ncclSocketComm* comm;
  NCCLCHECK(ncclSocketNewComm(&comm));
  NCCLCHECK(createListenSocket(&comm->fd, &handle->connectAddr));
  *listenComm = comm;
  return ncclSuccess;
}

ncclResult_t ncclSocketConnect(int dev, void* opaqueHandle, void** sendComm) {
  struct ncclSocketComm* comm;
  NCCLCHECK(ncclSocketNewComm(&comm));
  struct ncclSocketHandle* handle = (struct ncclSocketHandle*) opaqueHandle;
  NCCLCHECK(connectAddress(&comm->fd, &handle->connectAddr));
  *sendComm = comm;
  return ncclSuccess;
}

ncclResult_t ncclSocketAccept(void* listenComm, void** recvComm) {
  struct ncclSocketComm* lComm = (struct ncclSocketComm*)listenComm;
  struct ncclSocketComm* rComm;
  NCCLCHECK(ncclSocketNewComm(&rComm));
  struct sockaddr_in sockaddr;
  socklen_t socklen = sizeof(struct sockaddr_in);
  SYSCHECKVAL(accept(lComm->fd, (struct sockaddr*)&sockaddr, &socklen), "accept", rComm->fd);
  *recvComm = rComm;
  return ncclSuccess;
}

#define MAX_REQUESTS 128

ncclResult_t ncclSocketGetRequest(struct ncclSocketReqs* reqs, int op, void* data, int size, int fd, struct ncclSocketRequest** req) {
  if (reqs->requests == NULL) {
    NCCLCHECK(ncclCalloc(&reqs->requests, MAX_REQUESTS));
  }
  for (int i=0; i<MAX_REQUESTS; i++) {
    struct ncclSocketRequest* r = reqs->requests+i;
    if (r->used == 0) {
      r->op = op;
      r->data = data;
      r->size = size;
      r->fd = fd;
      r->offset = -1;
      r->used = 1;
      *req = r;
      return ncclSuccess;
    }
  }
  WARN("Socket : unable to allocate requests");
  return ncclInternalError;
}

ncclResult_t ncclSocketTest(void* request, int* done, int* size) {
  *done = 0;
  struct ncclSocketRequest *r = (struct ncclSocketRequest*)request;
  if (r == NULL) {
    WARN("NET/Socket : test called with NULL request");
    return ncclInternalError;
  }
  if (r->offset == -1) { /* try to send/recv size */
    int data = r->size;
    int offset = 0;
    NCCLCHECK(socketProgress(r->op, r->fd, &data, sizeof(int), &offset));

    if (offset == 0) return ncclSuccess; /* Not ready -- retry later */

    // Not sure we could ever receive less than 4 bytes, but just in case ...
    if (offset < sizeof(int)) NCCLCHECK(socketWait(r->op, r->fd, &data, sizeof(int), &offset));

    // Check size is less or equal to the size provided by the user
    if (r->op == NCCL_SOCKET_RECV && data > r->size) {
      WARN("NET/Socket : message truncated : receiving %d bytes instead of %d", data, r->size);
      return ncclInternalError;
    }
    r->size = data;
    r->offset = 0;
  }
  if (r->offset < r->size) {
    NCCLCHECK(socketProgress(r->op, r->fd, r->data, r->size, &r->offset));
  }
  if (r->offset == r->size) {
    if (size) *size = r->size;
    *done = 1;
    r->used = 0;
  }
  return ncclSuccess;
}

ncclResult_t ncclSocketRegMr(void* comm, void* data, int size, int type, void** mhandle) {
  return (type != NCCL_PTR_HOST) ? ncclInternalError : ncclSuccess;
}
ncclResult_t ncclSocketDeregMr(void* comm, void* mhandle) { return ncclSuccess; }

ncclResult_t ncclSocketIsend(void* sendComm, void* data, int size, void* mhandle, void** request) {
  struct ncclSocketComm* comm = (struct ncclSocketComm*)sendComm;
  NCCLCHECK(ncclSocketGetRequest(&comm->reqs, NCCL_SOCKET_SEND, data, size, comm->fd, (struct ncclSocketRequest**)request));
  return ncclSuccess;
}

ncclResult_t ncclSocketIrecv(void* recvComm, void* data, int size, void* mhandle, void** request) {
  struct ncclSocketComm* comm = (struct ncclSocketComm*)recvComm;
  NCCLCHECK(ncclSocketGetRequest(&comm->reqs, NCCL_SOCKET_RECV, data, size, comm->fd, (struct ncclSocketRequest**)request));
  return ncclSuccess;
}

ncclResult_t ncclSocketFlush(void* recvComm, void* data, int size, void* mhandle) {
  // We don't support CUDA pointers, so we don't need a flush operation
  return ncclInternalError;
}

ncclResult_t ncclSocketClose(void* opaqueComm) {
  struct ncclSocketComm* comm = (struct ncclSocketComm*)opaqueComm;
  if (comm) {
    free(comm->reqs.requests);
    close(comm->fd);
    free(comm);
  }
  return ncclSuccess;
}

ncclNet_t ncclNetSocket = {
  "Socket",
  ncclSocketInit,
  ncclSocketDevices,
  ncclSocketPciPath,
  ncclSocketPtrSupport,
  ncclSocketListen,
  ncclSocketConnect,
  ncclSocketAccept,
  ncclSocketRegMr,
  ncclSocketDeregMr,
  ncclSocketIsend,
  ncclSocketIrecv,
  ncclSocketFlush,
  ncclSocketTest,
  ncclSocketClose,
  ncclSocketClose,
  ncclSocketClose
};
