#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENCE.txt for license information
 ************************************************************************/

#include <algorithm>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "nccl.h"
#include "test_utilities.h"

int errors = 0;
double avg_bw = 0.0;
int avg_count = 0;
bool is_reduction = false;

template<typename T>
void RunTest(T** sendbuff, T** recvbuff, const int N, const ncclDataType_t type,
    ncclComm_t* const comms, const std::vector<int>& dList) {
  // initialize data
  int nDev = 0;
  NCCLCHECK(ncclCommCount(comms[0], &nDev));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  T* buffer = (T*)malloc(nDev * N * sizeof(T));
  T* result = (T*)malloc(nDev * N * sizeof(T));
  memset(buffer, 0, nDev * N * sizeof(T));
  memset(result, 0, nDev * N * sizeof(T));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamCreate(s+i));
    CUDACHECK(hipMemset(recvbuff[i], 0, nDev * N * sizeof(T)));
    Randomize(sendbuff[i], N, i);

    CUDACHECK(hipMemcpy(result + i * N, sendbuff[i], N * sizeof(T),
        hipMemcpyDeviceToHost));
  }

  // warm up GPU
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    NCCLCHECK(ncclAllGather((const void*)sendbuff[i], std::min(32 * 1024, N), type,
        (void*)recvbuff[i], comms[i], s[i]));
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //for (int n = 1; n <= N; n = n << 1)
  {
    int n = N;
    printf("%12i  %12i  %6s", (int)(n * sizeof(T)), n, TypeName(type).c_str());

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      NCCLCHECK(ncclAllGather((const void*)sendbuff[i], n, type, (void*)recvbuff[i], comms[i],
          s[i]));
    }

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }

    auto stop = std::chrono::high_resolution_clock::now();

    double elapsedSec =
        std::chrono::duration_cast<std::chrono::duration<double>>(
            stop - start).count();
    double algbw = (double)(n * sizeof(T)) / 1.0E9 * (double)(nDev - 1)
        / elapsedSec;
    double busbw = algbw;

    double maxDelta = 0.0;
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      double tmpDelta = CheckDelta<T>(recvbuff[i], result, nDev*N);
      maxDelta = std::max(tmpDelta, maxDelta);
    }

    printf("  %7.3f  %5.2f  %5.2f  %7.0le\n", elapsedSec * 1.0E3, algbw, busbw,
        maxDelta);

    if (maxDelta > deltaMaxValue(type, is_reduction)) errors++;
    avg_bw += busbw;
    avg_count++;

  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamDestroy(s[i]));
  }
  free(s);
  free(buffer);
  free(result);
}

template<typename T>
void RunTests(const int N, const ncclDataType_t type, ncclComm_t* const comms,
    const std::vector<int>& dList) {
  int nDev = 0;
  NCCLCHECK(ncclCommCount(comms[0], &nDev));
  T** sendbuff = (T**)malloc(nDev * sizeof(T*));
  T** recvbuff = (T**)malloc(nDev * sizeof(T*));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipMalloc(sendbuff + i, N * sizeof(T)));
    CUDACHECK(hipMalloc(recvbuff + i, nDev * N * sizeof(T)));
  }

  RunTest<T>(sendbuff, recvbuff, N, type, comms, dList);

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  free(sendbuff);
  free(recvbuff);
}

void usage() {
  printf("Tests nccl AllGather with user supplied arguments.\n"
      "    Usage: all_reduce_test <data size in bytes> [number of GPUs] "
      "[GPU 0] [GPU 1] ...\n\n");
}

int main(int argc, char* argv[]) {
  int nVis = 0;
  CUDACHECK(hipGetDeviceCount(&nVis));

  int N = 0;
  if (argc > 1) {
    int t = sscanf(argv[1], "%d", &N);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  } else {
    printf("Error: must specify at least data size in bytes!\n\n");
    usage();
    exit(EXIT_FAILURE);
  }

  int nDev = nVis;
  if (argc > 2) {
    int t = sscanf(argv[2], "%d", &nDev);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  }
  std::vector<int> dList(nDev);
  for (int i = 0; i < nDev; ++i)
    dList[i] = i % nVis;


  if (argc > 3) {
    if (argc - 3 != nDev) {
      printf("Error: insufficient number of GPUs in list\n\n");
      usage();
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < nDev; ++i) {
      int t = sscanf(argv[3 + i], "%d", dList.data() + i);
      if (t == 0) {
        printf("Error: %s is not an integer!\n\n", argv[2 + i]);
        usage();
        exit(EXIT_FAILURE);
      }
    }
  }

  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
  NCCLCHECK(ncclCommInitAll(comms, nDev, dList.data()));

  printf("# Using devices\n");
  for (int g=0; g<nDev; ++g) {
    int cudaDev;
    int rank;
    hipDeviceProp_t prop;
    NCCLCHECK(ncclCommCuDevice(comms[g], &cudaDev));
    NCCLCHECK(ncclCommUserRank(comms[g], &rank));
    CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
    printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev,
        prop.pciBusID, prop.name);
  }
  printf("\n");

  printf("# %10s  %12s  %6s  %7s  %5s  %5s  %7s\n",
      "bytes", "N", "type", "time", "algbw", "busbw", "delta");

  RunTests<char>(N / sizeof(char), ncclChar, comms, dList);
  RunTests<int>(N / sizeof(int), ncclInt, comms, dList);
#ifdef CUDA_HAS_HALF
  RunTests<half>(N / sizeof(half), ncclHalf, comms, dList);
#endif
  RunTests<float>(N / sizeof(float), ncclFloat, comms, dList);
  RunTests<double>(N / sizeof(double), ncclDouble, comms, dList);
  RunTests<long long>(N / sizeof(long long), ncclInt64, comms, dList);
  RunTests<unsigned long long>(N / sizeof(unsigned long long), ncclUint64, comms, dList);

  printf("\n");

  for(int i=0; i<nDev; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  char* str = getenv("NCCL_TESTS_MIN_BW");
  double check_avg_bw = str ? atof(str) : -1;
  avg_bw /= avg_count;

  printf(" Out of bounds values : %d %s\n", errors, errors ? "FAILED" : "OK");
  printf(" Avg bus bandwidth    : %g %s\n", avg_bw, check_avg_bw == -1 ? "" : (avg_bw < check_avg_bw ? "FAILED" : "OK"));
  printf("\n");
  if (errors || avg_bw < check_avg_bw)
    exit(EXIT_FAILURE);
  else 
    exit(EXIT_SUCCESS);
}

