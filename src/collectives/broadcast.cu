/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclBroadcastFunc(const void* sendbuff, void* recvbuff, const size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(NCCL_COLL,"Broadcast: opCount %lx sendbuff %p recvbuff %p count %zi datatype %d op %d root %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, datatype, op, root, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(BROADCAST_SUBSTEPS, BROADCAST_BUFCHUNKS, 1, 1, nbytes, proxyPatternFrom(root), comm));
    NCCLCHECK(saveKernel(ncclCollBroadcast, sendbuff, recvbuff, nbytes, ncclInt8, op, root, comm, stream, nbytes, 1));
  }

  return ncclSuccess;
}

/* Deprecated original "in place" function, similar to MPI */
NCCL_API(ncclResult_t, ncclBcast, void* buff, size_t count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclBcast(void* buff, size_t count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclBroadcastFunc, "Bcast", buff, buff, count, datatype,
          ncclSum, root, comm, stream);
}

NCCL_API(ncclResult_t, ncclBroadcast, const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclBroadcast(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclBroadcastFunc, "Broadcast", sendbuff, recvbuff, count, datatype,
          ncclSum, root, comm, stream);
}
