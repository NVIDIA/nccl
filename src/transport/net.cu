#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "stat_collector.h"
#include "core.h"
#include "transport.h"
#include "nvmlwrap.h"
#include "net.h"
#include "param.h"
#include "nvlink.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define NET_MAX_IFS 16

// We encode 3 bits of distance per interface into a ncclTvalue_t (64-bit)
#define NET_BITS_PER_IF 3
#define NET_BITS_PER_IF_MASK ((1<<NET_BITS_PER_IF)-1)
static_assert(sizeof(ncclTvalue_t)*8 >= NET_MAX_IFS*NET_BITS_PER_IF, "NET_MAX_IFS*NET_BITS_PER_IF must fit in a ncclTvalue_t");

struct netInfo {
  int rank;
  int ndev;
  short scores[NET_MAX_IFS];
};

struct netConnectInfo {
  ncclNetHandle_t netHandle;
};

struct netSendResources {
  void* netSendComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclSendMem* hostDevMem;
  int netDev;
  bool cudaSupport;
  struct ncclRecvMem* devNetMem;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

struct netRecvResources {
  void* netListenComm;
  void* netRecvComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclRecvMem* hostDevMem;
  int netDev;
  bool cudaSupport;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

/* Fill information necessary to exchange between ranks to choose whether or not
 * to use this transport */
ncclResult_t netFillInfo(ncclTinfo_t* opaqueInfo, int rank) {
  struct netInfo* info = (struct netInfo*)opaqueInfo;
  static_assert(sizeof(struct netInfo) <= sizeof(ncclTinfo_t), "NET Info too large");
  info->rank = rank;
  int *scores;
  NCCLCHECK(ncclNetDevices(&info->ndev, &scores));
  if (info->ndev == 0) {
    WARN("Error : Network returned 0 device");
    return ncclSystemError;
  }
  if (info->ndev > NET_MAX_IFS) info->ndev = NET_MAX_IFS;
  for (int d=0; d<info->ndev; d++) info->scores[d] = scores[d];
  free(scores);
  return ncclSuccess;
}

/* Determine if we can communicate with the peer */
ncclResult_t netCanConnect(ncclTvalue_t* ret, ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo) {
  ret[0] = 0;
  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  for (int d=0; d<myInfo->ndev; d++) {
    // Keep 3 bits of score info per dev
    ret[0] |= ((myInfo->scores[d] & NET_BITS_PER_IF_MASK)<<(NET_BITS_PER_IF*d));
  }
  return ncclSuccess;
}

static inline int groupBestStart(int nranks, int* groups, int group, ncclTvalue_t* values, int card, int minScore) {
  int bestRank = -1;
  int bestScore = 0;
  for (int rank=0; rank<nranks; rank++) {
    if (groups[rank] != group) continue;
    for (int i=0; i<nranks; i++) {
      ncclTvalue_t netValue = values[rank*nranks+i];
      if (netValue != 0) {
        ncclTvalue_t score = (netValue>>(NET_BITS_PER_IF*card)) & NET_BITS_PER_IF_MASK;
        if (score >= minScore && score > bestScore) {
          bestScore = score;
          bestRank = rank;
        }
        // All other values should be the same, stop here for this rank
        break;
      }
    }
  }
  return bestRank;
}
static inline int groupBestEnd(int nranks, int* groups, int group, int* subgroups, int startSubGroup, int startRank, ncclTvalue_t* values, int card, int minScore) {
  // For the last rank, we don't need the absolute best score, just to be within minScore.
  for (int rank=nranks-1; rank>=0; rank--) {
    if (groups[rank] != group) continue;
    if (startSubGroup != -1 && startSubGroup == subgroups[rank]) continue;
    if (startRank == rank) continue;
    for (int i=0; i<nranks; i++) {
      ncclTvalue_t netValue = values[rank*nranks+i];
      if (netValue != 0) {
        ncclTvalue_t score = (netValue>>(NET_BITS_PER_IF*card)) & NET_BITS_PER_IF_MASK;
        if (score >= minScore) {
          return rank;
        }
        // All other values should be the same, stop here for this rank
        break;
      }
    }
  }
  return -1;
}


ncclResult_t netGetRings(int nranks, int* groups, int* subgroups, ncclTvalue_t* values, int* nringsRet, int* prev, int* next, int minScore, int* nthreads) {
  int nGroups = groups[nranks-1] + 1;
  int cardUsed[NET_MAX_IFS*nGroups];
  for (int c=0; c<NET_MAX_IFS*nGroups; c++) cardUsed[c] = 0;

  for (int ring = 0; ring<*nringsRet; ring++) {
    int starts[nGroups];
    int ends[nGroups];
    for (int group = 0; group<nGroups; group++) {
      int nranksInGroup = 0;
      int nsubGroups = 0;
      for (int rank=0; rank<nranks; rank++) if (groups[rank] == group) {
          nranksInGroup++;
          nsubGroups = std::max(subgroups[rank], nsubGroups);
        }
      starts[group] = ends[group] = -1;
      // Receive on the rank closest to the NIC
      for (int card=0; card<NET_MAX_IFS; card++) {
        if (cardUsed[group*NET_MAX_IFS+card] == 1) continue;
        int start = groupBestStart(nranks, groups, group, values, card, minScore);
        // Send from any rank, but best on a different subgroup and close to the NIC also.
        int end = (nranksInGroup == 1) ? start
            : groupBestEnd(nranks, groups, group, subgroups, nsubGroups ? subgroups[start] : -1, start, values, card, minScore);
        //printf("Ring %d, Minscore %d, Card %d, group %d, start = %d, end = %d\n", ring, minScore, card, group, start, end);
        if (start != -1 && end != -1) {
          cardUsed[group*NET_MAX_IFS+card] = 1;
          starts[group] = start;
          ends[group] = end;
          break;
        }
      }
      if (starts[group] == -1 || ends[group] == -1) {
        *nringsRet = ring;
        return ncclSuccess;
      }
    }
    // Link groups together
    for (int group = 0; group<nGroups; group++) {
      int nextGroup = (group+1)%nGroups;
      next[ring*nranks+ends[group]] = starts[nextGroup];
      prev[ring*nranks+starts[nextGroup]] = ends[group];
    }
  }
  return ncclSuccess;
}

int getDev(int ringId, int nDev, short* scores) {
  int maxScore = 0;
  for (int d=0; d<nDev; d++) if (scores[d] > maxScore) maxScore = scores[d];
  int skip = ringId+1;
  while (skip) {
    for (int d=0; d<nDev; d++) {
      if (scores[d] == maxScore) {
        skip--;
        if (skip == 0) return d;
      }
    }
  }
  return 0;
}

NCCL_PARAM(NetGdrRead, "NET_GDR_READ", -2);

/* Determine if we will use this transport for this peer and return connect
 * information for this peer */
ncclResult_t netSendSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netSendResources* resources;
  NCCLCHECK(ncclCalloc(&resources, 1));
  ring->send.transportResources = resources;

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->scores);
  resources->cudaSupport = false;

  // Get user's GDR READ setting
  int gdrReadParam = ncclParamNetGdrRead();

  // Determine whether the GPU has NVLink
  int cudaDev;
  CUDACHECK(hipGetDevice(&cudaDev));
  char busId[NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE];
  CUDACHECK(hipDeviceGetPCIBusId(busId, NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE, cudaDev));
  int nvlinks = getNumNvlinks(busId);

  // Enable GDR read when:
  // 1) user sets it, or
  // 2) we are on a NVSwitch platform (i.e. no P2P traffic over PCI-E switch) AND the GPU is Volta
  bool enableGdrRead = (gdrReadParam > 0) || (nvlinks >= CONNECT_NVSWITCH && ncclCudaCompCap() > 6 && gdrReadParam != 0);
  if (enableGdrRead) {
    int flags;
    NCCLCHECK(ncclNetPtrSupport(resources->netDev, &flags));
    if (flags & NCCL_PTR_CUDA)
      resources->cudaSupport = true;
  }
  if (resources->cudaSupport)
    INFO(INIT|NET, "Net: enabling net device %d to read from rank %d", resources->netDev, myInfo->rank);

  int size = offsetof(struct ncclRecvMem, buff)+ring->buffSize;
  if (resources->cudaSupport) {
    NCCLCHECK(ncclCudaCalloc((char**)(&resources->devNetMem), size));
  }

  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostRecvMem, (void**)&resources->devHostRecvMem, size));
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostSendMem, (void**)&resources->devHostSendMem, size));

  return ncclSuccess;
}

ncclResult_t netRecvSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netRecvResources* resources;
  NCCLCHECK(ncclCalloc(&resources, 1));
  ring->recv.transportResources = resources;

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->scores);
  int flags;
  NCCLCHECK(ncclNetPtrSupport(resources->netDev, &flags));
  resources->cudaSupport = (flags & NCCL_PTR_CUDA) ? true : false;

  int sendSize = sizeof(struct ncclSendMem);
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostSendMem, (void**)&resources->devHostSendMem, sendSize));

  int recvSize = offsetof(struct ncclRecvMem, buff)+ring->buffSize;
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostRecvMem, (void**)&resources->devHostRecvMem, recvSize));

  struct netInfo* peerInfo = (struct netInfo*)peerOpaqueInfo;
  INFO(INIT|NET,"Ring %02d : %d -> %d via NET/%s/%d%s%s", ring->id, peerInfo->rank, myInfo->rank, ncclNetName(), resources->netDev,
      resources->cudaSupport ? "/GDRDMA" : "",
      (resources->hostDevMem != NULL) ? "/GDCopy" : "");
  struct netConnectInfo* info = (struct netConnectInfo*) connectInfo;
  NCCLCHECK(ncclNetListen(resources->netDev, &info->netHandle, &resources->netListenComm));
  return ncclSuccess;
}

ncclResult_t netSendConnect(struct ncclConnect* connectInfo, struct ncclConnector* send) {
  // Setup device pointers
  struct netSendResources* resources = (struct netSendResources*)send->transportResources;

  if (resources->cudaSupport) {
    send->conn.buff = resources->devNetMem->buff;
    // We don't use devMem for llMode because the CPU has to read the data
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  } else {
    send->conn.buff = resources->devHostRecvMem->buff;
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  }
  send->conn.tail = &resources->devHostRecvMem->tail;
  send->conn.opCount = &resources->devHostRecvMem->opCount;
  send->conn.fifo = resources->devHostRecvMem->sizesFifo;
  send->conn.llFifo = resources->devHostRecvMem->llSizesFifo;

  if (resources->hostDevMem == NULL) {
    send->conn.head = &resources->devHostSendMem->head;
    send->conn.llHead = &resources->devHostSendMem->llHead;
  }

  // Connect to remote peer
  struct netConnectInfo* info = (struct netConnectInfo*)connectInfo;
  NCCLCHECK(ncclNetConnect(resources->netDev, info->netHandle, &resources->netSendComm));
  return ncclSuccess;
}

/* Connect to this peer */
ncclResult_t netRecvConnect(struct ncclConnect* connectInfo, struct ncclConnector* recv) {
  // Setup device pointers
  struct netRecvResources* resources = (struct netRecvResources*)recv->transportResources;

  recv->conn.head = &resources->devHostSendMem->head;
  recv->conn.llHead = &resources->devHostSendMem->llHead;

  if (resources->cudaSupport == false) {
    recv->conn.buff = resources->devHostRecvMem->buff;
    recv->conn.llBuff = resources->devHostRecvMem->llBuff;
  }

  if (resources->hostDevMem == NULL) {
    recv->conn.tail = &resources->devHostRecvMem->tail;
    recv->conn.opCount = &resources->devHostRecvMem->opCount;
  }

  // Finish connection establishment
  NCCLCHECK(ncclNetAccept(resources->netListenComm, &resources->netRecvComm));
  NCCLCHECK(ncclNetCloseListen(resources->netListenComm));

  return ncclSuccess;
}

ncclResult_t netSendFree(void* transportResources) {
  struct netSendResources* resources = (struct netSendResources*)transportResources;
  NCCLCHECK(ncclCudaHostFree(resources->hostSendMem));
  NCCLCHECK(ncclCudaHostFree(resources->hostRecvMem));
  if (resources->cudaSupport)
    CUDACHECK(hipFree(resources->devNetMem));
  NCCLCHECK(ncclNetCloseSend(resources->netSendComm));
  free(resources);
  return ncclSuccess;
}

ncclResult_t netRecvFree(void* transportResources) {
  struct netRecvResources* resources = (struct netRecvResources*)transportResources;
  NCCLCHECK(ncclCudaHostFree(resources->hostSendMem));
  NCCLCHECK(ncclCudaHostFree(resources->hostRecvMem));
  NCCLCHECK(ncclNetCloseRecv(resources->netRecvComm));
  free(resources);
  return ncclSuccess;
}

ncclResult_t netSendProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netSendResources* resources = (struct netSendResources*) (ring->send.transportResources);
  const int llMode = args->llMode;

  volatile uint64_t* prevTail = &resources->hostRecvMem->tail;
  struct ncclSendMem* prevMem = resources->hostDevMem ? resources->hostDevMem : resources->hostSendMem;
  uint64_t* prevHead = llMode ? &prevMem->llHead : &prevMem->head;
  struct ncclRecvMem* localMem = resources->cudaSupport ? resources->devNetMem : resources->hostRecvMem;
  char* localBuff = llMode ? resources->hostRecvMem->llBuff : localMem->buff;
  int ptrType = resources->cudaSupport ? NCCL_PTR_CUDA : NCCL_PTR_HOST;
  volatile int* sizesFifo = llMode ? resources->hostRecvMem->llSizesFifo : resources->hostRecvMem->sizesFifo;
  int buffSize = llMode ? NCCL_LL_BUFF_SIZE : ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  assert(args->substeps <= SIZES_FIFO_SIZE);

  uint64_t head = llMode ? resources->llStep : 0ULL;
  uint64_t tail = llMode ? resources->llStep : 0ULL;
  uint64_t end = head + args->nsteps;

  int idle = 0;
  void* requests[args->substeps];

  int from = ring->send.proxyInfo->comm->rank;
  int to = ring->userRanks[1];

  if (!args->needProxy) goto nextColl;

  TRACE(NET,"opCount %lx head %lx tail %lx end %lx nsteps %d llMode %d", args->opCount, head, tail, end, args->nsteps, llMode);
  TRACE(NET,"opCount %lx buffSize %d sliceSize %d ptrType %d", args->opCount, buffSize, sliceSize, ptrType);

  // Update in case we skipped some collectives
  if (llMode == 0) resources->hostRecvMem->opCount = args->opCount;

  while (head < end) {
    idle++;
    if (llMode) {
      if (tail < end && tail < head + args->substeps) {
        int slot = tail%args->substeps;
        int size = sizesFifo[slot];
        if (size != 0) {
          if (size == -1) size = 0;
          uint32_t flag = tail + 1;
          int nFifoLines = DIVUP(size, sizeof(union ncclLLFifoLine));
          size = nFifoLines * sizeof(union ncclLLFifoLine);
          union ncclLLFifoLine* lines = (union ncclLLFifoLine*)(localBuff+slot*sliceSize);
          for (int i=0; i<nFifoLines; i++) {
            volatile uint32_t *f1 = &lines[i].flag1;
            volatile uint32_t *f2 = &lines[i].flag2;
            while (f1[0] != flag || f2[0] != flag);
          }
          uint64_t start_micros = now_micros();
          NCCLCHECK(ncclNetIsend(resources->netSendComm, lines, size, ptrType, requests+slot));
          uint64_t end_micros = now_micros();
          if (args->nccl_prof->do_profile) {
            commStat_t* comm_stat = create_comm_stat(NET_SEND, from, to, start_micros, end_micros, sliceSize);
            enqueue_stat(args->nccl_prof, comm_stat);
          }
          sizesFifo[slot] = size;
          tail++;
          idle = 0;
        }
      }
    } else while (tail < *prevTail) {
        // Send through network
        int slot = tail%args->substeps;
        uint64_t start_micros = now_micros();
        NCCLCHECK(ncclNetIsend(resources->netSendComm, localBuff+slot*sliceSize, sizesFifo[slot], ptrType, requests+slot));
        uint64_t end_micros = now_micros();
        if (args->nccl_prof->do_profile) {          
          commStat_t* comm_stat = create_comm_stat(NET_SEND, from, to, start_micros, end_micros, sliceSize);
          enqueue_stat(args->nccl_prof, comm_stat);
        }
        tail++;
        idle = 0;
      }
    if (head < tail) {
      int done;
      int slot = head%args->substeps;
      NCCLCHECK(ncclNetTest(requests[slot], &done, NULL));
      if (done) {
        if (llMode) {
          sizesFifo[slot] = 0;
          // Make sure size is reset to zero before we update the head.
          __sync_synchronize();
        }
        head++;
        *prevHead = head;
        idle = 0;
      }
    }
    if (idle) transportProxyIdle(idle);
  }
  // TODO(HJ): For now, we set saved nccl profiling data manually.
  if (args->nccl_prof->step > 30) {
    StatCollector* stat_collector = GetStatCollector();
    stat_collector->set_saved_in_file();
  }
  if (args->nccl_prof->do_profile) {
    StatCollector* stat_collector = GetStatCollector();
    stat_collector->save(args->nccl_prof);
  }

  // Reset
  if (llMode == 0) *prevTail = 0;

nextColl:
  if (llMode) {
    resources->llStep += args->nsteps;
    // Don't forget to ack otherwise the GPU won't be able to push data.
    *prevHead = resources->llStep;
    if (resources->llStep > resources->llLastCleaning + NCCL_LL_CLEAN_FREQ) {
      memset(localBuff, 0, NCCL_LL_BUFF_SIZE);
      resources->llStep += NCCL_LL_CHUNKS;
      *prevHead = resources->llStep;
      resources->llLastCleaning = resources->llStep;
    }
  }
  return ncclSuccess;
}

ncclResult_t netRecvProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netRecvResources* resources = (struct netRecvResources*) (ring->recv.transportResources);
  int llMode = args->llMode;

  volatile uint64_t* nextHead = llMode ? &resources->hostSendMem->llHead : &resources->hostSendMem->head;
  struct ncclRecvMem* localMem = resources->cudaSupport ? ring->devMemRecv : resources->hostRecvMem;
  char* localBuff = llMode ? localMem->llBuff : localMem->buff;
  char* nextBuff = (resources->cudaSupport == false && resources->hostDevMem) ? resources->hostDevMem->buff : NULL;
  int ptrType = resources->cudaSupport ? NCCL_PTR_CUDA : NCCL_PTR_HOST;
  uint64_t* nextTail = resources->hostDevMem ? &resources->hostDevMem->tail : &resources->hostRecvMem->tail;

  int buffSize = llMode ? NCCL_LL_BUFF_SIZE : ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  uint64_t head = llMode ? resources->llStep : 0ULL;
  uint64_t tail = llMode ? resources->llStep : 0ULL;
  uint64_t end = head + args->nsteps;

  int idle = 0;
  void* requests[args->substeps];

  int nranks = ring->recv.proxyInfo->comm->nRanks;
  int from = ring->userRanks[nranks-1];
  int to = ring->recv.proxyInfo->comm->rank;

  if (!args->needProxy) goto nextColl;

  TRACE(NET,"opCount %lx head %lx tail %lx end %lx nsteps %d llMode %d", args->opCount, head, tail, end, args->nsteps, llMode);
  TRACE(NET,"opCount %lx buffSize %d sliceSize %d ptrType %d", args->opCount, buffSize, sliceSize, ptrType);

  if (llMode == 0) {
    // Waiting for next opCount is only needed before writing nextTail.
    uint64_t* nextOpCount = resources->hostDevMem ? &resources->hostDevMem->opCount : &resources->hostRecvMem->opCount;
    transportProxyWait([=] { return *nextOpCount >= args->opCount; });
  }

  while (head < end) {
    idle++;
    if ((tail < head + args->substeps) && (tail < *nextHead + args->substeps) && (tail < end)) {
      int slot = tail%args->substeps;
      uint64_t start_micros = now_micros();
      NCCLCHECK(ncclNetIrecv(resources->netRecvComm, localBuff+slot*sliceSize, sliceSize, ptrType, requests+slot));
      uint64_t end_micros = now_micros();
      if (args->nccl_prof->do_profile) {
        commStat_t* comm_stat = create_comm_stat(NET_RECV, from, to, start_micros, end_micros, sliceSize);
        enqueue_stat(args->nccl_prof, comm_stat);
      }
      tail++;
      idle = 0;
    }
    if (tail > head) {
      int done;
      int slot = head%args->substeps;
      int size;
      NCCLCHECK(ncclNetTest(requests[slot], &done, &size));
      if (done) {
        if (nextBuff) memcpy(nextBuff+slot*sliceSize, localBuff+slot*sliceSize, size);
        head++;
        if (llMode == 0) {
          if (ptrType == NCCL_PTR_CUDA) ncclNetFlush(resources->netRecvComm, localBuff+slot*sliceSize, size);
          *nextTail = head;
        }
        idle = 0;
      }
    }
    if (idle) transportProxyIdle(idle);
  }
  // TODO(HJ): For now, we set saved nccl profiling data manually.
  if (args->nccl_prof->step > 30) {
    StatCollector* stat_collector = GetStatCollector();
    stat_collector->set_saved_in_file();
  }
  if (args->nccl_prof->do_profile) {
    StatCollector* stat_collector = GetStatCollector();
    stat_collector->save(args->nccl_prof);
  }

  // Wait for last ack and reset
  if (llMode == 0) {
    transportProxyWait([=] { return *nextHead == head; });
    *nextHead = 0;
  }

nextColl:
  if (llMode) {
    resources->llStep += args->nsteps;
    if (resources->llStep > resources->llLastCleaning + NCCL_LL_CLEAN_FREQ) {
      resources->llStep += NCCL_LL_CHUNKS;
      while (*nextHead < resources->llStep);
      resources->llLastCleaning = resources->llStep;
    }
  }
  return ncclSuccess;
}

struct ncclTransport netTransport = {
  "NET",
  netFillInfo,
  netCanConnect,
  netGetRings,
  { netSendSetup, netSendConnect, netSendFree, netSendProxy },
  { netRecvSetup, netRecvConnect, netRecvFree, netRecvProxy }
};
