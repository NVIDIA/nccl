#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and boffset for buffer sync
#define NEXT_STEP \
  step++; \
  boffset += sliceSize; \
  if (boffset == buffSize) boffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void BroadcastKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  __shared__ T* sharedNextOutput;
  __shared__ DevRing<T> ring;
  bool pushrecv = args.pushrecv;

  LoadRing<THREADS>(args.ring, &ring);
  __syncthreads();

  if (tid == 0) {
    WaitFlag prevCommOp(ring.prevOpCounter, 0);
    WaitFlag nextCommOp(ring.nextOpCounter, 0);
    prevCommOp.wait(args.opIndex);
    nextCommOp.wait(args.opIndex);
    if (pushrecv) {
      *ring.sendPtrToPrev = (T*)args.ThisOutput;
      Wait([=] {
        return *ring.recvPtrFromNext != nullptr;
      });
      sharedNextOutput = *ring.recvPtrFromNext;
      *ring.recvPtrFromNext = nullptr;
    }
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring.recvFlagFromNext, (1-NUM_BUFCHUNKS)*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring.recvFlagFromPrev, 0);
  PostFlag postDoneToPrev(ring.sendFlagToPrev, 0);
  PostFlag postReadyToNext(ring.sendFlagToNext, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T> Prims;

  const int size = args.N;
  const int rank = ring.userRank[0];
  const int nextRank = ring.userRank[1];
  const int root = args.root;
  const int buffSize = args.buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int boffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput =  args.ThisOutput;
  T * __restrict__ prevInput = ring.recvBuffer;
  T * __restrict__ nextOutput =  ring.sendBuffer;

  for (int offset = 0; offset < size; offset += sliceSize) {
    int maxOffset = size-offset;
    if (rank == root) {
      Prims::Copy(
          thisInput + offset,
          pushrecv ? sharedNextOutput + offset : nextOutput + boffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext,
          postReadyToNext);
    } else if (nextRank == root) {
      if (pushrecv) maxOffset = 0; // Only wait for signals
      Prims::Copy(
          prevInput  + boffset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitReadyFromPrev,
          postDoneToPrev);
    } else {
      if (pushrecv) {
        Prims::Copy(
            thisOutput + offset,
            sharedNextOutput + offset,
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);
      } else {
        Prims::DoubleCopy(
            prevInput + boffset,
            thisOutput + offset,
            nextOutput + boffset,
	    sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);
      }
    }
    NEXT_STEP; // Increases step, boffset
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    if (nextRank != root) {
      // Wait for last update from next then reset the flag
      waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
      *ring.recvFlagFromNext = 0;
    }

    if (rank != root) {
      // reset the flag
      *ring.recvFlagFromPrev = 0;
    }

    incrementOpCounter(&args);
  }
}

#define THREADS 256
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingBroadcast(void* buff, const int count, const int root,
    ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks != 1) {
    KernelArgs<T> args;
    ArgsSetup(&args, buff, buff, root, count, comm);
    LAUNCH_KERNEL(BroadcastKernel, THREADS, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template<typename> class RedOp>
class Broadcast {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int root, ncclComm* comm, hipStream_t stream) {
    return RingBroadcast<RedOp<T>, T>(recvbuff, count, root, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclBcast, void* buff, int count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclBcast(void* buff, int count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream) {
  return enqueue<Broadcast, FuncNull>(nullptr, buff, count, datatype, root, comm, stream);
}

