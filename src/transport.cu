/*************************************************************************
 * Copyright (c) 2016-2019, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"

extern struct ncclTransport p2pTransport;
extern struct ncclTransport shmTransport;
extern struct ncclTransport netTransport;

struct ncclTransport ncclTransports[NTRANSPORTS] = {
  p2pTransport,
  shmTransport,
  netTransport,
};

#define RECV 0
#define SEND 1

static bool NeedProxy(int type, int pattern, int root, struct ncclRing* ring, int nranks) {
  if (pattern == ncclPatternRing || pattern == ncclPatternRingTwice) return true;

  /* In chains, one rank does not need a proxy. Let's figure out which one it is */
  // Which index in the reorganized rings should we compare root against */
  const int myrank = 0, nextrank = 1, prevrank = nranks-1;
  int index = pattern == ncclPatternPipelineFrom ?
      /*                            no recv /  no send    if root = */
      /* bcast  */ (type == RECV ?   myrank : nextrank ):
      /* reduce */ (type == RECV ? prevrank :   myrank );
  int rank = ring->userRanks[index];
  return (root != rank);
}

enum { proxyRecv=0, proxySend=1 };

#define PROXYARGS_ALLOCATE_SIZE 32
struct ncclProxyPool {
  struct ncclProxyPool *next;
  struct ncclProxyArgs elems[PROXYARGS_ALLOCATE_SIZE];
};

ncclResult_t transportAllocateProxyArgs(struct ncclComm* comm, struct ncclProxyArgs** argsptr) {
  struct ncclProxyState* state = &comm->proxyState;
  struct ncclProxyArgs* elem;
  pthread_mutex_lock(&state->mutex);
  if (state->pool == NULL) {
    // Allocate a new pool of elements
    struct ncclProxyPool* newPool;
    NCCLCHECK(ncclCalloc(&newPool, 1));
    struct ncclProxyArgs* newElems = newPool->elems;
    // Chain newly allocated elements
    for (int i=0; i<PROXYARGS_ALLOCATE_SIZE; i++) {
      if (i+1 < PROXYARGS_ALLOCATE_SIZE) newElems[i].next = newElems+i+1;
    }
    // Add them all to the pool list
    state->pool = newElems;
    // Save the pool memory block for later resource release
    newPool->next = state->pools;
    state->pools = newPool;
  }
  elem = state->pool;
  state->pool = state->pool->next;
  pthread_mutex_unlock(&state->mutex);
  elem->next = elem->nextPeer = NULL;
  *argsptr = elem;
  return ncclSuccess;
}

static void ProxyAppend(struct ncclConnector* connector, struct ncclProxyArgs* args) {
  struct ncclComm* comm = connector->comm;
  struct ncclProxyState* state = &comm->proxyState;
  pthread_mutex_lock(&state->mutex);
  if (connector->proxyAppend == NULL) {
    // Nothing running for that peer. Add to the circular list
    if (state->ops == NULL) {
      // Create the list
      args->next = args;
      state->ops = args;
    } else {
      // Insert element in the list
      args->next = state->ops->next;
      state->ops->next = args;
    }
    connector->proxyAppend = args;
  } else {
    // There is an active operation already for that peer.
    // Add it to the per-peer list
    connector->proxyAppend->nextPeer = args;
    connector->proxyAppend = args;
  }
  pthread_mutex_unlock(&state->mutex);
}

template <int type>
static ncclResult_t SaveProxy(int peer, struct ncclProxyArgs* args) {
  if (peer < 0) return ncclSuccess;

  struct ncclPeer* peerComm = args->channel->peers+peer;
  struct ncclConnector* connector = type == proxyRecv ? &peerComm->recv : &peerComm->send;
  if (connector->transportComm->proxy == NULL) return ncclSuccess;

  struct ncclProxyArgs* op;
  NCCLCHECK(transportAllocateProxyArgs(connector->comm, &op));
  memcpy(op, args, sizeof(struct ncclProxyArgs));
  op->connector = connector;
  op->progress = connector->transportComm->proxy;
  op->state = ncclProxyOpReady;
  ProxyAppend(connector, op);
  return ncclSuccess;
}

ncclResult_t transportSaveProxies(struct ncclProxyArgs* args, int pattern, int root, int nranks) {
  if (pattern == ncclPatternRing || pattern == ncclPatternRingTwice || pattern == ncclPatternPipelineFrom || pattern == ncclPatternPipelineTo) {
    struct ncclRing* ring = &args->channel->ring;
    if (NeedProxy(RECV, pattern, root, ring, nranks)) NCCLCHECK(SaveProxy<proxyRecv>(ring->prev, args));
    if (NeedProxy(SEND, pattern, root, ring, nranks)) NCCLCHECK(SaveProxy<proxySend>(ring->next, args));
  }
  if (pattern == ncclPatternTreeUp || pattern == ncclPatternTreeUpDown) {
    // Tree up
    struct ncclTree* tree = &args->channel->tree;
    for (int i=0; i<NCCL_MAX_TREE_ARITY; i++) NCCLCHECK(SaveProxy<proxyRecv>(tree->down[i], args));
    NCCLCHECK(SaveProxy<proxySend>(tree->up, args));
  }
  if (pattern == ncclPatternTreeDown || pattern == ncclPatternTreeUpDown) {
    // Tree down
    struct ncclTree* tree = &args->channel->tree;
    for (int i=0; i< NCCL_MAX_TREE_ARITY; i++) NCCLCHECK(SaveProxy<proxySend>(tree->down[i], args));
    NCCLCHECK(SaveProxy<proxyRecv>(tree->up, args));
  }
  return ncclSuccess;
}

void* persistentThread(void *comm_) {
  struct ncclComm* comm = (struct ncclComm*)comm_;
  struct ncclProxyState* state = &comm->proxyState;
  struct ncclProxyArgs* op = NULL;
  ncclResult_t ret = ncclSuccess;
  int idle = 1;
  int idleSpin = 0;
  while (1) {
    do {
      if (*comm->abortFlag) return NULL;
      if (op == NULL) {
        pthread_mutex_lock(&state->mutex);
        op = state->ops;
        if (op == NULL) {
          if (state->stop) {
            // No more commands to process and proxy has been requested to stop
            pthread_mutex_unlock(&state->mutex);
            return NULL;
          }
          pthread_cond_wait(&state->cond, &state->mutex);
        }
        pthread_mutex_unlock(&state->mutex);
      }
    } while (op == NULL);
    op->idle = 0;
    if (op->state != ncclProxyOpNone) ret = op->progress(op);
    if (ret != ncclSuccess) {
      comm->fatalError = ret;
      INFO(NCCL_ALL,"%s:%d -> %d [Proxy Thread]", __FILE__, __LINE__, ret);
      return NULL;
    }
    idle &= op->idle;
    pthread_mutex_lock(&state->mutex);
    if (!idle) idleSpin = 0;
    struct ncclProxyArgs *next = op->next;
    if (next->state == ncclProxyOpNone) {
      struct ncclProxyArgs *freeOp = next;
      if (next->nextPeer) {
        // Replace next by its next per-peer element.
        next = next->nextPeer;
        if (op != freeOp) {
          next->next = freeOp->next;
          op->next = next;
        } else {
          next->next = next;
        }
      } else {
        // Remove next from circular list
        next->connector->proxyAppend = NULL;
        if (op != freeOp) {
          next = next->next;
          op->next = next;
        } else {
          next = NULL;
        }
      }
      if (freeOp == state->ops) state->ops = next;
      freeOp->next = state->pool;
      state->pool = freeOp;
    }
    op = next;
    if (op == state->ops) {
      if (idle == 1) {
        if (++idleSpin == 10) {
          sched_yield();
          idleSpin = 0;
        }
      }
      idle = 1;
    }
    pthread_mutex_unlock(&state->mutex);
  }
}

ncclResult_t transportStartProxy(struct ncclComm* comm) {
  pthread_mutex_lock(&comm->proxyState.mutex);
  if (comm->proxyState.ops != NULL)
    pthread_cond_signal(&comm->proxyState.cond);
  pthread_mutex_unlock(&comm->proxyState.mutex);
  return ncclSuccess;
}

ncclResult_t transportCreateProxy(struct ncclComm* comm) {
  if (!comm->proxyThread) {
    comm->proxyState.cond = PTHREAD_COND_INITIALIZER;
    comm->proxyState.mutex = PTHREAD_MUTEX_INITIALIZER;
    comm->proxyState.ops = NULL;
    pthread_create(&comm->proxyThread, NULL, persistentThread, comm);
  }
  return ncclSuccess;
}

ncclResult_t transportDestroyProxy(struct ncclComm* comm) {
  struct ncclProxyState* state = &comm->proxyState;

  // Request the proxy to stop and then wake it
  pthread_mutex_lock(&state->mutex);
  state->stop = true;
  pthread_cond_signal(&state->cond);
  pthread_mutex_unlock(&state->mutex);
  if (comm->proxyThread) pthread_join(comm->proxyThread, NULL);

  // Free off any memory allocated for the proxy arg pools
  pthread_mutex_lock(&state->mutex);
  struct ncclProxyState* proxyState = &comm->proxyState;
  while (proxyState->pools != NULL) {
    struct ncclProxyPool *next = proxyState->pools->next;
    free(proxyState->pools);
    proxyState->pools = next;
  }
  pthread_mutex_unlock(&state->mutex);

  return ncclSuccess;
}
