/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclAllGatherFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(NCCL_COLL,"AllGather: opCount %lx sendbuff %p recvbuff %p count %zi datatype %d op %d root %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, datatype, op, root, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(ALLGATHER_SUBSTEPS, ALLGATHER_BUFCHUNKS, comm->nRanks-1, comm->nRanks, nbytes*comm->nRanks, proxyPatternRing, comm));
    NCCLCHECK(saveKernel(ncclCollAllGather, sendbuff, recvbuff, nbytes, ncclInt8, op, root, comm, stream, nbytes*comm->nRanks, 1));
  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclAllGather, const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclAllGatherFunc, "AllGather", sendbuff, recvbuff, sendcount, datatype,
          ncclSum, 0, comm, stream);
}
