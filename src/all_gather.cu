#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllGatherKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  __shared__ T* sharedNextOutput;
  __shared__ DevRing<T> ring;
  bool pushrecv = args.pushrecv;

  LoadRing<THREADS>(args.ring, &ring);
  __syncthreads();

  if (tid == 0) {
    WaitFlag prevCommOp(ring.prevOpCounter, 0);
    WaitFlag nextCommOp(ring.nextOpCounter, 0);
    prevCommOp.wait(args.opIndex);
    nextCommOp.wait(args.opIndex);
    if (pushrecv) {
      *ring.sendPtrToPrev = (T*)args.ThisOutput;
      Wait([=] {
        return *ring.recvPtrFromNext != nullptr;
      });
      sharedNextOutput = *ring.recvPtrFromNext;
      *ring.recvPtrFromNext = nullptr;
    }
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring.recvFlagFromNext, -NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring.recvFlagFromPrev, -1*NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring.sendFlagToPrev, -1*NUM_SUBSTEPS);
  PostFlag postReadyToNext(ring.sendFlagToNext, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T> Prims;

  const int size = args.N;
  const int nranks = args.nRanks;
  const int buffSize = args.buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput =  args.ThisOutput;
  T * __restrict__ prevInput = ring.recvBuffer;
  T * __restrict__ nextOutput =  ring.sendBuffer;

  for (int chunkOffset = 0; chunkOffset < size; chunkOffset += sliceSize) {
    /////////////// begin AllGather steps ///////////////
    int offset;
    int maxOffset = size-chunkOffset;
    int rankDest;

    // step 0: push data to next GPU
    rankDest = ring.userRank[0];
    offset = chunkOffset + rankDest * size;

    if (thisInput == thisOutput) {
      Prims::Copy(
          thisInput  + offset,
          pushrecv ? sharedNextOutput + offset : nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);
    } else {
      Prims::DoubleCopy(
          thisInput  + chunkOffset,
          thisOutput + offset,
          pushrecv ? sharedNextOutput + offset : nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);
    }

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: copy to next GPU
    if (pushrecv) {
      for (int j=1; j<nranks-1; ++j) {
        rankDest = ring.userRank[nranks-j];
        offset = chunkOffset + rankDest * size;

        Prims::Copy(
            thisOutput + offset,
            sharedNextOutput + offset,
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }
    } else {
      for (int j=1; j<nranks-1; ++j) {
        rankDest = ring.userRank[nranks-j];
        offset = chunkOffset + rankDest * size;

        Prims::DoubleCopy(
            prevInput + poffset,
            thisOutput + offset,
            nextOutput + noffset,
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }

      // Make final copy from buffer to dest.
      rankDest = ring.userRank[1];
      offset = chunkOffset + rankDest * size;

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          prevInput + poffset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    // Wait for last update from next then reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
    *ring.recvFlagFromNext = 0;

    // Wait for last update from prev then reset the flag
    waitReadyFromPrev.wait(NUM_SUBSTEPS*(step+1));
    *ring.recvFlagFromPrev = 0;

    incrementOpCounter(&args);
  }
}

#define THREADS 384
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingAllGather(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, 0, count, comm);
    LAUNCH_KERNEL(AllGatherKernel, THREADS, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template<typename> class RedOp>
class AllGather {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingAllGather<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclAllGather, const void* sendbuff, int count, ncclDataType_t datatype,
    void* recvbuff, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllGather(const void* sendbuff, int count, ncclDataType_t datatype,
    void* recvbuff, ncclComm_t comm, hipStream_t stream) {
  return enqueue<AllGather, FuncNull>(sendbuff, recvbuff, count, datatype, 0, comm, stream);
}

