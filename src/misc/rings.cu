/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "param.h"

#define NCCL_MAX_SCORE 7

/* Parse user defined rings. Format is like :
 * "0 1|1 0|0 1 2 3|3 2 1 0|0 2 3 1|1 3 2 0|0 1 2 3 4 5 6 7|7 6 5 4 3 2 1 0"
 * Rings with a non-matching number of ranks are ignored so we can provide
 * rings for multiple cases.
 */
#define MAX_ENV_RANKS 512
static ncclResult_t parseRings(const char* str, int* nringsRet, int nranks, int* prev, int* next) {
  int ranks[MAX_ENV_RANKS];
  int nrings = 0;
  int rank = 0;
  int offset = 0;
  int status = 0; // 0 : between numbers, 1 : inside number
  do {
    int digit = str[offset] - '0';
    if (digit >= 0 && digit <= 9) {
      if (status == 0) {
        ranks[rank] = digit;
        status = 1;
      } else {
        ranks[rank] = ranks[rank]*10+digit;
      }
    } else {
      if (status == 1) {
        rank++;
        if (rank == MAX_ENV_RANKS) goto end;
      }
      status = 0;
      if (str[offset] == '|' || str[offset] == '\0') {
        int prevRank = ranks[rank-1];
        // Ignore rings if nranks doesn't match
        if (rank != nranks) goto newring;

        for (int r=0; r<nranks; r++) {
          int rank = ranks[r];
          // Ignore rings with ranks out of bounds
          if (rank < 0 || rank >= nranks) goto newring;
          // Ignore rings with duplicate ranks
          for (int i=0; i<r; i++)
            if (ranks[i] == rank) goto newring;

          next[nrings*nranks+prevRank] = rank;
          prev[nrings*nranks+rank] = prevRank;
          prevRank = rank;
        }
        nrings++;
newring:
        rank = 0;
      }
    }
  } while (str[offset++] != 0);
end:
  *nringsRet = nrings;
  return ncclSuccess;
}

/*
 * Ring creation algorithm
 *
 * First, we establish hierarchical coordinates depending on the way ranks can
 * communicate. After fillCoords, we have for each rank a unique 3-int array
 * {   node, pci_domain,   rank } corresponding to the three transports :
 * { 2[NET],     1[SHM], 0[P2P] }.
 * Also, we renumber ranks (to indexes) based on their growing coordinates.
 *
 * Then, we ask transports to connect groups together. We start with net, then
 * shm, then p2p. We maintain two arrays, prev and next, where values are equal
 * to -1 when ranks are not yet connected, and a rank otherwise. We never
 * connect ranks outside our group, meaning that on 4 nodes of 2 sockets of 4
 * ranks, if we are rank 13, we should see something like (provided we have a
 * single net interface, hence a single ring) :
 *
 * Connecting all nodes                                <13>
 * 2[NET] : prev 31 -1 -1 -1 -1 -1 -1 -1  7 -1 -1 -1 -1 -1 -1 -1 15 -1 -1 -1 -1 -1 -1 -1 23 -1 -1 -1 -1 -1 -1 -1
 *          next -1 -1 -1 -1 -1 -1 -1  8 -1 -1 -1 -1 -1 -1 -1 16 -1 -1 -1 -1 -1 -1 -1 24 -1 -1 -1 -1 -1 -1 -1  0
 *
 * Connecting P2P domains with shared memory           <13>
 * 1[SHM] : prev 31 -1 -1 -1 -1 -1 -1 -1  7 -1 -1 -1 11 -1 -1 -1 15 -1 -1 -1 -1 -1 -1 -1 23 -1 -1 -1 -1 -1 -1 -1
 *          next -1 -1 -1 -1 -1 -1 -1  8 -1 -1 -1 12 -1 -1 -1 16 -1 -1 -1 -1 -1 -1 -1 24 -1 -1 -1 -1 -1 -1 -1  0
 *
 * Connecting ranks (only inside the P2P domain)       <13>
 * 0[P2P] : prev 31 -1 -1 -1 -1 -1 -1 -1  7 -1 -1 -1 11 12 13 14 15 -1 -1 -1 -1 -1 -1 -1 23 -1 -1 -1 -1 -1 -1 -1
 *          next -1 -1 -1 -1 -1 -1 -1  8 -1 -1 -1 12 13 14 15 16 -1 -1 -1 -1 -1 -1 -1 24 -1 -1 -1 -1 -1 -1 -1  0
 *
 * Hence, when we ask a transport to connect groups, we provide it with a subview of the ranks (except for net
 * which always sees the full world). That way, P2P can bruteforce all combinations inside the node without
 * risking to explode in terms of combinations, and we scale better.
 *
 * Finally, we loop over Network scores to try to create rings with high scores (=locality) and decrease until
 * we get at least one ring.
 */

static void recIsConnected(int rank, int* connected, int nranks, int* matrix, int transport) {
  connected[rank] = 1;
  for (int r=0; r<nranks; r++) {
    if (connected[r] == 0 && matrix[rank*nranks+r] == transport) {
      recIsConnected(r, connected, nranks, matrix, transport);
    }
  }
}

static void isConnected(int rank, int* connected, int nranks, int* matrix, int transport) {
  for (int r=0; r<nranks; r++) connected[r] = 0;
  recIsConnected(rank, connected, nranks, matrix, transport);
}

#define NEW_IDX(rank) do { \
  rankToIdx[rank] = idx; \
  idxToRank[idx] = rank; \
  for (int t=0; t<NTRANSPORTS; t++) coords[rank*NTRANSPORTS+t] = current[t]; \
  idx++; \
} while (0)

int findConnected(int rank, int* matrix, int nranks, int transport, int* coords) {
  for (int r=0; r<nranks; r++) {
    if (coords[r*NTRANSPORTS] == -1 && matrix[rank*nranks+r] == transport) return r;
  }
  return -1;
}

static ncclResult_t fillCoords(int nranks, int* matrix, int* coords, int* rankToIdx, int* idxToRank) {
  int current[NTRANSPORTS];
  int* p2pConnected;
  NCCLCHECK(ncclCalloc(&p2pConnected, nranks));
  for (int i=0; i<NTRANSPORTS; i++) current[i] = 0;
  int curRank = 0, idx = 0;
  while (1) {
    // P2P is handled separately as there is no level below it and we need to
    // cover the case of being connected to another GPU indirectly.
    // So we detect all GPUs in the same P2P domain once and add them all at
    // once.
    isConnected(curRank, p2pConnected, nranks, matrix, 0);
    for (int r=0; r<nranks; r++) {
      if (p2pConnected[r]) {
        NEW_IDX(r);
        curRank = r;
        current[0]++;
      }
    }
    current[0] = 0;

    if (idx == nranks) {
      free(p2pConnected);
      return ncclSuccess;
    }

    // Find next group, either connected through SHM or NET.
    int rank;
    int transport = 1;
    while ((rank = findConnected(curRank, matrix, nranks, transport, coords)) == -1) {
      current[transport] = 0;
      transport++;
      if (transport == NTRANSPORTS) {
        WARN("Error : Could not find transport to connect next group\n");
        free(p2pConnected);
        return ncclInternalError; }
    }
    curRank = rank;
    current[transport]++;
  }
}

NCCL_PARAM(MinNrings, "MIN_NRINGS", 0);
NCCL_PARAM(MaxNrings, "MAX_NRINGS", 0);

/* Users can force the number of threads with an environment variable */
NCCL_PARAM(Nthreads, "NTHREADS", -2);
ncclResult_t getEnvThreads(int* nthreads) {
  int64_t nt = ncclParamNthreads();
  if (nt != -2)
    *nthreads = nt;
  return ncclSuccess;
}

static inline int copyRings(int nrings, int newNrings, int nranks, int* a, int* b, int* c, int* d) {
  if (newNrings > MAXCHANNELS) newNrings = MAXCHANNELS;
  for (int r=nrings; r<newNrings; r++) {
    for (int i=0; i<nranks; i++) {
      a[r*nranks+i] = a[(r-nrings)*nranks+i];
      b[r*nranks+i] = b[(r-nrings)*nranks+i];
      c[r*nranks+i] = c[(r-nrings)*nranks+i];
      d[r*nranks+i] = d[(r-nrings)*nranks+i];
    }
  }
  return newNrings;
}
/* Main ring creation function */
ncclResult_t ncclGetRings(int* nrings, int* nthreads, int rank, int nranks, int* transports, ncclTvalue_t* values, int* prev, int* next, int* treeIn, int* treeOut) {
  *nrings = 0;

  if (nranks == 1) return ncclSuccess;

  char* str = getenv("NCCL_RINGS");
  if (str && strlen(str)>0) {
    int ret = parseRings(str, nrings, nranks, prev, next);
    if (ret == ncclSuccess && *nrings > 0) {
      if (rank == 0) INFO(NCCL_INIT,"%d ring(s) set by environment", *nrings);
      NCCLCHECK(getEnvThreads(nthreads));
      for (int r = 0; r<*nrings; r++) {
        for (int i = 0; i<nranks; i++) {
          if (transports[i*nranks+prev[i]] == 2) treeIn[i] = 1;
          if (transports[i*nranks+next[i]] == 2) treeOut[i] = 1;
        }
      }
      return ncclSuccess;
    }
    if (rank == 0) INFO(NCCL_INIT,"No valid ring found in environment, ignoring");
    *nrings = 0;
  }

  // Compute hierarchical topology groups, indexes, and rank<->index tables
  int* coords, *globalIdxToRank, *globalRankToIdx;
  NCCLCHECK(ncclCalloc(&coords, nranks*NTRANSPORTS));
  for (int i=0; i<nranks*NTRANSPORTS; i++) coords[i] = -1;
  NCCLCHECK(ncclCalloc(&globalIdxToRank, nranks));
  NCCLCHECK(ncclCalloc(&globalRankToIdx, nranks));

  NCCLCHECK(fillCoords(nranks, transports, coords, globalRankToIdx, globalIdxToRank));

  // Start with a high score, then decrease until we find rings
  int minScore = NCCL_MAX_SCORE;
  int nringsTmp;
  int *prevTmp, *nextTmp, *idxToRank, *rankToIdx, *groups, *subgroups;
  NCCLCHECK(ncclCalloc(&prevTmp, nranks*MAXCHANNELS));
  NCCLCHECK(ncclCalloc(&nextTmp, nranks*MAXCHANNELS));
  NCCLCHECK(ncclCalloc(&idxToRank, nranks));
  NCCLCHECK(ncclCalloc(&rankToIdx, nranks));
  NCCLCHECK(ncclCalloc(&groups, nranks));
  NCCLCHECK(ncclCalloc(&subgroups, nranks));

  int nThreads;
  do {
    nThreads = *nthreads;
    for (int i=0; i<nranks*MAXCHANNELS; i++) prevTmp[i] = nextTmp[i] = -1;
    nringsTmp = MAXCHANNELS;
    // Loop over transports to connect groups
    for (int t=NTRANSPORTS-1; t>=0; t--) {
      for (int i=0; i<nranks; i++) idxToRank[i] = rankToIdx[i] = -1;

      int nidx = 0;
      for (int i=0; i<nranks; i++) {
        // Extract only ranks in the same local area as rank
        // We need to extract them in the topological order, hence we iterate over indexes, not ranks
        int r = globalIdxToRank[i];
        int sameLocal = 1;
        for (int tr = NTRANSPORTS-1; tr > t; tr--) if (coords[r*NTRANSPORTS+tr] != coords[rank*NTRANSPORTS+tr]) sameLocal = 0;
        if (!sameLocal) continue;

        groups[nidx] = coords[r*NTRANSPORTS+t];
        subgroups[nidx] = t ? coords[r*NTRANSPORTS+t-1] : nidx;
        rankToIdx[r] = nidx;
        idxToRank[nidx] = r;
        nidx++;
      }

      int ngroups = groups[nidx-1] + 1; // Coords should be ordered

      ncclTvalue_t* subvalues;
      int *subprev, *subnext;
      NCCLCHECK(ncclCalloc(&subvalues, nidx*nidx));
      NCCLCHECK(ncclCalloc(&subprev, nidx*nringsTmp));
      NCCLCHECK(ncclCalloc(&subnext, nidx*nringsTmp));
      if (ngroups > 1) {
        /* Extract subvalues */
        for (int i=0; i<nidx; i++) {
          for (int j=0; j<nidx; j++) {
            if (transports[idxToRank[i]*nranks+idxToRank[j]] == t)
              subvalues[i*nidx+j] = values[idxToRank[i]*nranks+idxToRank[j]];
            else
              subvalues[i*nidx+j] = 0;
          }
        }
        /* Extract subprev/subnext */
        for (int i=0; i<nidx*nringsTmp; i++) {
          subprev[i] = subnext[i] = -1;
        }
        for (int r=0; r<nringsTmp; r++) {
          int start = -1, end = -1;
          for (int i=0; i<nranks; i++) {
            if (rankToIdx[i] == -1) continue;
            if (prevTmp[r*nranks+i] != -1) start = i;
            if (nextTmp[r*nranks+i] != -1) end = i;
          }
          if (start != -1 && end != -1) {
            subprev[r*nidx+rankToIdx[start]] = rankToIdx[end];
            subnext[r*nidx+rankToIdx[end]] = rankToIdx[start];
          }
        }
        /* Get rings */
        NCCLCHECK(ncclTransports[t].getRings(nidx, groups, subgroups, subvalues, &nringsTmp, subprev, subnext, minScore, &nThreads));
        /* Merge subprev/subnext into prev/next */
        for (int r=0; r<nringsTmp; r++) {
          for (int i=0; i<nidx; i++) {
            if ((prevTmp[r*nranks+idxToRank[i]] == -1) && (subprev[r*nidx+i] != -1)) prevTmp[r*nranks+idxToRank[i]] = idxToRank[subprev[r*nidx+i]];
            if ((nextTmp[r*nranks+idxToRank[i]] == -1) && (subnext[r*nidx+i] != -1)) nextTmp[r*nranks+idxToRank[i]] = idxToRank[subnext[r*nidx+i]];
            if (t == NTRANSPORTS-1) {
              // Save node-level masters for trees
              treeIn[r*nranks+idxToRank[i]] = prevTmp[r*nranks+idxToRank[i]] == -1 ? 0 : 1;
              treeOut[r*nranks+idxToRank[i]] = nextTmp[r*nranks+idxToRank[i]] == -1 ? 0 : 1;
            }
          }
        }
        //for (int r=0; r<nringsTmp; r++) {
        //printf("[%d] [%d] [%d] [%d] Prev ", rank, minScore, t, r); for (int i=0; i<nranks; i++) printf("%d ", prevTmp[r*nranks+i]); printf("\n");
        //printf("[%d] [%d] [%d] [%d] Next ", rank, minScore, t, r); for (int i=0; i<nranks; i++) printf("%d ", nextTmp[r*nranks+i]); printf("\n");
        //}
      }
      free(subvalues);
      free(subprev);
      free(subnext);
      if (nringsTmp == 0) break;
    }
    minScore--;
    if (nringsTmp > *nrings) {
      *nrings = nringsTmp;
      for (int i=0; i<nranks*(*nrings); i++) {
        prev[i] = prevTmp[i];
        next[i] = nextTmp[i];
      }
    }
  } while (nringsTmp == 0 && minScore);

  free(coords);
  free(globalRankToIdx);
  free(globalIdxToRank);
  free(prevTmp);
  free(nextTmp);
  free(idxToRank);
  free(rankToIdx);
  free(groups);
  free(subgroups);

  *nthreads = nThreads;

  /* Duplicate the rings in case of multinode+NVLink */
  int nnodes = 0;
  for (int r=0; r<nranks; r++) nnodes += treeIn[r];
  int nvlink;
  NCCLCHECK(ncclNvlinkGpu(&nvlink));
  if (nnodes > 1 && nvlink) {
    *nrings = copyRings(*nrings, *nrings*2, nranks, prev, next, treeIn, treeOut);
  }

  if (*nrings == 0) {
    WARN("Could not create rings, falling back on simple ring");
    *nrings = 1;
    prev[rank] = (rank-1+nranks) % nranks;
    next[rank] = (rank+1)%nranks;
  }

  int maxNrings = ncclParamMaxNrings();
  int minNrings = ncclParamMinNrings();
  if (maxNrings > 0 && minNrings > maxNrings) {
    if (rank == 0) WARN("NCCL_MIN_NRINGS set to a value greater than NCCL_MAX_NRINGS, ignoring NCCL_MIN_NRINGS");
    minNrings = 0;
  }
  if (minNrings > MAXCHANNELS) {
    if (rank == 0) WARN("NCCL_MIN_NRINGS set to a value greater than the maximum number of rings supported (%d), limiting it to %d", MAXCHANNELS, MAXCHANNELS);
    minNrings = MAXCHANNELS;
  }
  if (maxNrings > 0 && maxNrings <= *nrings) {
    if (rank == 0) INFO(NCCL_INIT,"Limiting to %d rings per user request.", maxNrings);
    *nrings = maxNrings;
  } else {
    int defaultMinNrings = ncclCudaCompCap() == 3 ? 2 : 1;
    if (minNrings < defaultMinNrings) minNrings = defaultMinNrings;
    if (minNrings > 0 && minNrings > *nrings) {
      if (rank == 0 && minNrings > defaultMinNrings) INFO(NCCL_INIT,"Duplicating rings to %d per user request.", minNrings);
      *nrings = copyRings(*nrings, minNrings, nranks, prev, next, treeIn, treeOut);
    }
  }

  NCCLCHECK(getEnvThreads(nthreads));
  return ncclSuccess;
}
