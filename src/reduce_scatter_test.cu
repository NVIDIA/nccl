#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "nccl.h"
#include "test_utilities.h"


template<typename T>
void RunTest(T** sendbuff, T** recvbuff, const int N, const ncclDataType_t type,
    const ncclRedOp_t op, ncclComm_t* const comms, const std::vector<int>& dList) {
  // initialize data
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  T* buffer = (T*)malloc(N * nDev * sizeof(T));
  T* result = (T*)malloc(N * nDev * sizeof(T));
  memset(buffer, 0, N * nDev * sizeof(T));
  memset(result, 0, N * nDev * sizeof(T));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamCreate(s+i));
    CUDACHECK(hipMemset(recvbuff[i], 0, N * sizeof(T)));
    Randomize(sendbuff[i], N * nDev, i);

    if (i == 0) {
      CUDACHECK(hipMemcpy(result, sendbuff[i], N * nDev * sizeof(T),
          hipMemcpyDeviceToHost));
    } else {
      Accumulate<T>(result, sendbuff[i], N * nDev, op);
    }
  }

  // warm up GPU
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    ncclReduceScatter((const void*)sendbuff[i], (void*)recvbuff[i],
        std::min(N, 1024 * 1024), type, op, comms[i], s[i]);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

//  for (int n = 0; n <= N; n = (n > 0) ? n << 1 : 1)
  {
    int n = N;
    printf("%12i  %12i  %6s  %6s", (int)(n * sizeof(T)), n,
        TypeName(type).c_str(), OperationName(op).c_str());

    // do out-of-place reduction first
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      ncclReduceScatter((const void*)sendbuff[i], (void*)recvbuff[i], n, type,
          op, comms[i], s[i]);
    }

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }

    auto stop = std::chrono::high_resolution_clock::now();

    double elapsedSec =
        std::chrono::duration_cast<std::chrono::duration<double>>(
            stop - start).count();
    double algbw = (double)(n * sizeof(T)) / 1.0E9 / elapsedSec;
    double busbw = algbw * (double)(nDev - 1);

    double maxDelta = 0.0;
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      double tmpDelta = CheckDelta<T>(recvbuff[i], result+i*n, n);
      maxDelta = std::max(tmpDelta, maxDelta);
    }

    printf("  %7.3f  %5.2f  %5.2f  %7.0le", elapsedSec * 1.0E3, algbw, busbw,
        maxDelta);
  }

  {
    // now do in-place reduction
    int n = N;

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      ncclReduceScatter((const void*)sendbuff[i], (void*)sendbuff[i], n, type,
          op, comms[i], s[i]);
    }

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }

    auto stop = std::chrono::high_resolution_clock::now();

    double elapsedSec =
        std::chrono::duration_cast<std::chrono::duration<double>>(
            stop - start).count();
    double algbw = (double)(n * sizeof(T)) / 1.0E9 / elapsedSec;
    double busbw = algbw * (double)(nDev - 1);

    double maxDelta = 0.0;
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      double tmpDelta = CheckDelta<T>(sendbuff[i], result+i*n, n);
      maxDelta = std::max(tmpDelta, maxDelta);
    }

    printf("  %7.3f  %5.2f  %5.2f  %7.0le\n", elapsedSec * 1.0E3, algbw, busbw,
        maxDelta);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamDestroy(s[i]));
  }
  free(s);
  free(buffer);
  free(result);
}

template<typename T>
void RunTests(const int N, const ncclDataType_t type, ncclComm_t* const comms,
    const std::vector<int>& dList) {
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  T** sendbuff = (T**)malloc(nDev * sizeof(T*));
  T** recvbuff = (T**)malloc(nDev * sizeof(T*));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipMalloc(sendbuff + i, N * nDev * sizeof(T)));
    CUDACHECK(hipMalloc(recvbuff + i, N * sizeof(T)));
  }

  for (ncclRedOp_t op : { ncclSum, ncclProd, ncclMax, ncclMin }) {
//  for (ncclRedOp_t op : { ncclSum }) {
    RunTest<T>(sendbuff, recvbuff, N, type, op, comms, dList);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  free(sendbuff);
  free(recvbuff);
}

void usage() {
  printf("Tests nccl ReduceScatter with user supplied arguments.\n"
      "    Usage: all_reduce_test <data size in bytes> [number of GPUs] "
      "[GPU 0] [GPU 1] ...\n\n");
}

int main(int argc, char* argv[]) {
  int nVis = 0;
  CUDACHECK(hipGetDeviceCount(&nVis));

  int N = 0;
  if (argc > 1) {
    int t = sscanf(argv[1], "%d", &N);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  } else {
    printf("Error: must specify at least data size in bytes!\n\n");
    usage();
    exit(EXIT_FAILURE);
  }

  int nDev = nVis;
  if (argc > 2) {
    int t = sscanf(argv[2], "%d", &nDev);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  }
  std::vector<int> dList(nDev);
  for (int i = 0; i < nDev; ++i)
    dList[i] = i % nVis;

  if (argc > 3) {
    if (argc - 3 != nDev) {
      printf("Error: insufficient number of GPUs in list\n\n");
      usage();
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < nDev; ++i) {
      int t = sscanf(argv[3 + i], "%d", dList.data() + i);
      if (t == 0) {
        printf("Error: %s is not an integer!\n\n", argv[2 + i]);
        usage();
        exit(EXIT_FAILURE);
      }
    }
  }

  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
  ncclCommInitAll(comms, nDev, dList.data());

  printf("# Using devices\n");
  for (int g = 0; g < nDev; ++g) {
    int cudaDev;
    int rank;
    hipDeviceProp_t prop;
    ncclCommCuDevice(comms[g], &cudaDev);
    ncclCommUserRank(comms[g], &rank);
    CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
    printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev,
        prop.pciBusID, prop.name);
  }
  printf("\n");

  printf("# %10s  %12s  %6s  %6s        out-of-place                      "
      "in-place\n", "", "", "", "");
  printf("# %10s  %12s  %6s  %6s  %7s  %5s  %5s  %7s  %7s  %5s  %5s  %7s\n",
      "bytes", "N", "type", "op", "time", "algbw", "busbw", "delta", "time",
      "algbw", "busbw", "delta");

  RunTests<char>(N / sizeof(char), ncclChar, comms, dList);
  RunTests<int>(N / sizeof(int), ncclInt, comms, dList);
#ifdef CUDA_HAS_HALF
  RunTests<half>(N / sizeof(half), ncclHalf, comms, dList);
#endif
  RunTests<float>(N / sizeof(float), ncclFloat, comms, dList);
  RunTests<double>(N / sizeof(double), ncclDouble, comms, dList);
  RunTests<long long>(N / sizeof(long long), ncclInt64, comms, dList);
  RunTests<unsigned long long>(N / sizeof(unsigned long long), ncclUint64, comms, dList);

  printf("\n");

  for(int i=0; i<nDev; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  exit(EXIT_SUCCESS);
}

