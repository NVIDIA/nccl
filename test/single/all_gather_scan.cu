#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <float.h>

#include "nccl.h"
#include "test_utilities.h"
#include <nvToolsExt.h>

void showUsage(const char* bin) {
  printf("\n"
         "Usage: %s <type> <n_min> <n_max> [delta] [gpus] [gpu0 [gpu1 [...]]]\n"
         "Where:\n"
#ifdef CUDA_HAS_HALF
         "    type   =   [char|int|half|float|double|int64|uint64]\n"
#else
         "    type   =   [char|int|float|double|int64|uint64]\n"
#endif
         "    n_min  >   0\n"
         "    n_max  >=  n_min\n"
         "    delta  >   0\n\n", bin);
  return;
}

int main(int argc, char* argv[]) {
  int nvis = 0;
  CUDACHECK(hipGetDeviceCount(&nvis));
  if (nvis == 0) {
    printf("No GPUs found\n");
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  ncclDataType_t type;
  int n_min;
  int n_max;
  int delta;
  int gpus;
  int* list = NULL;

  if (argc < 4) {
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  type = strToType(argv[1]);
  if (type == nccl_NUM_TYPES) {
    printf("Invalid <type> '%s'\n", argv[1]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  n_min = strToPosInt(argv[2]);
  if (n_min < 1) {
    printf("Invalid <n_min> '%s'\n", argv[2]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  n_max = strToPosInt(argv[3]);
  if (n_max < n_min) {
    printf("Invalid <n_max> '%s'\n", argv[3]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  if (argc > 4) {
    delta = strToPosInt(argv[4]);
    if (delta < 1) {
      printf("Invalid <delta> '%s'\n", argv[4]);
      showUsage(argv[0]);
      exit(EXIT_FAILURE);
    }
  } else {
    delta = (n_max == n_min) ? 1 : (n_max - n_min+9) / 10;
  }

  if (argc > 5) {
    gpus = strToPosInt(argv[5]);
    if (gpus < 1) {
      printf("Invalid <gpus> '%s'\n", argv[5]);
      showUsage(argv[0]);
      exit(EXIT_FAILURE);
    }
  } else {
    gpus = nvis;
  }

  list = (int*)malloc(gpus*sizeof(int));

  if (argc > 6 && argc != 6+gpus) {
    printf("If given, GPU list must be fully specified.\n");
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  for(int g=0; g<gpus; ++g) {
    if(argc > 6) {
      list[g] = strToNonNeg(argv[6+g]);
      if (list[g] < 0) {
        printf("Invalid GPU%d '%s'\n", g, argv[6+g]);
        showUsage(argv[0]);
        exit(EXIT_FAILURE);
      } else if (list[g] >= nvis) {
        printf("GPU%d (%d) exceeds visible devices (%d)\n", g, list[g], nvis);
        showUsage(argv[0]);
        exit(EXIT_FAILURE);
      }
    } else {
      list[g] = g % nvis;
    }
  }

  size_t word = wordSize(type);
  size_t max_input = n_max * word;
  size_t max_output = max_input * gpus;
  void* refout;
  CUDACHECK(hipHostMalloc(&refout, max_output));

  void **input, **output;
  double** localError;
  ncclComm_t* comm;
  hipStream_t* stream;

  input = (void**)malloc(gpus*sizeof(void*));
  output = (void**)malloc(gpus*sizeof(void*));
  localError = (double**)malloc(gpus*sizeof(double*));
  comm = (ncclComm_t*)malloc(gpus*sizeof(ncclComm_t));
  stream = (hipStream_t*)malloc(gpus*sizeof(hipStream_t));

  for(int g=0; g<gpus; ++g) {
    char busid[32] = {0};
    CUDACHECK(hipDeviceGetPCIBusId(busid, 32, list[g]));
    printf("# Rank %d using device %d [%s]\n", g, list[g], busid);

    CUDACHECK(hipSetDevice(list[g]));
    CUDACHECK(hipMalloc(&input[g],  max_input));
    CUDACHECK(hipMalloc(&output[g], max_output));
    CUDACHECK(hipHostMalloc(&localError[g], sizeof(double)));
    CUDACHECK(hipStreamCreate(&stream[g]));
    makeRandom(input[g], n_max, type, 42+g);

    CUDACHECK(hipMemcpy((char*)refout+max_input*g, input[g], max_input, hipMemcpyDeviceToHost));
  }

  NCCLCHECK(ncclCommInitAll(comm, gpus, list));

  printf("       BYTES ERROR       MSEC     BW\n");

  for(int n=n_min; n<=n_max; n+=delta) {
    size_t out_bytes = word * n * gpus;

    for(int g=0; g<gpus; ++g) {
      CUDACHECK(hipSetDevice(list[g]));
      CUDACHECK(hipMemsetAsync(output[g], 0, out_bytes, stream[g]));
      CUDACHECK(hipStreamSynchronize(stream[g]));
    }

    auto start = std::chrono::high_resolution_clock::now();
    for(int g=0; g<gpus; ++g) {
      CUDACHECK(hipSetDevice(list[g]));
      NCCLCHECK(ncclAllGather(input[g], n, type, output[g], comm[g], stream[g]));
    }
    for(int g=0; g<gpus; ++g) {
      CUDACHECK(hipSetDevice(list[g]));
      CUDACHECK(hipStreamSynchronize(stream[g]));
    }
    auto stop = std::chrono::high_resolution_clock::now();
    double ms = std::chrono::duration_cast<std::chrono::duration<double>>
        (stop - start).count() * 1000.0;

    double max_error = 0.0;
    for(int slice=0; slice<gpus; ++slice) {
      void* refSlice = (void*)((char*)refout + slice*max_input);
      for(int g=0; g<gpus; ++g) {
        CUDACHECK(hipSetDevice(list[g]));
        void* mySlice = (void*)((char*)output[g] + slice*n*word);
        maxDiff(localError[g], mySlice, refSlice, n, type, stream[g]);
      }
      for(int g=0; g<gpus; ++g) {
        CUDACHECK(hipSetDevice(list[g]));
        CUDACHECK(hipStreamSynchronize(stream[g]));
        max_error = max(max_error, *localError[g]);
      }
    }

    double mb = (double)(n*word * (gpus-1)) * 1.e-6;
    double algbw = mb / ms;
    printf("%12lu %5.0le %10.3lf %6.2lf\n",
        n*word, max_error, ms, algbw);
  }

  for(int g=0; g<gpus; ++g) {
    CUDACHECK(hipSetDevice(list[g]));
    CUDACHECK(hipStreamDestroy(stream[g]));
    ncclCommDestroy(comm[g]);
    CUDACHECK(hipFree(input[g]));
    CUDACHECK(hipFree(output[g]));
    CUDACHECK(hipHostFree(localError[g]));
  }

  free(localError);
  free(output);
  free(input);
  free(comm);
  free(stream);
  CUDACHECK(hipHostFree(refout));
  exit(EXIT_SUCCESS);
}

