#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "reduce_kernel.h"
#include "common.h"

namespace {
  template<typename T, typename RedOp>
  __device__ __forceinline__ void oneRankReduce() {
    ncclWork *w = &ncclShmem.work;
    int tid = threadIdx.x;
    int tn = blockDim.x;
    #pragma unroll 1
    for(int e=0; e < NCCL_MAX_WORK_ELEMENTS && w->elems[e].header.type != ncclWorkTypeUnused; e++) {
      ncclWorkElem *we = &w->elems[e];
      intptr_t eltN = we->count;
      int bid = we->bid;
      int bn = we->nChannels;
      T const *src = (T const*)we->sendbuff;
      T *dst = (T*)we->recvbuff;

      // each block/channel gets a roughly equal segment of 16 byte packs
      constexpr int EltPerPack = 16/sizeof(T);
      intptr_t packN = (eltN + EltPerPack-1) - (eltN + EltPerPack-1)%EltPerPack;
      intptr_t i0 = (bid+0)*(packN/bn) + (bid+0 < packN%bn ? bid+0 : packN%bn);
      intptr_t i1 = (bid+1)*(packN/bn) + (bid+1 < packN%bn ? bid+1 : packN%bn);
      i0 *= EltPerPack;
      i0 = i0 < eltN ? i0 : eltN;
      i1 *= EltPerPack;
      i1 = i1 < eltN ? i1 : eltN;
      src += i0;
      dst += i0;
      ReduceOrCopyMulti<COLL_UNROLL, RedOp, T, 1, 1, 1, 1, 1>
        (tid, tn, &(we->redOpArg), true, 1, &src, 1, &dst, i1-i0);
    }
  }
}

#define INSTANTIATE(devredop, type) \
  __device__ void NCCL_ONERANK_REDUCE_NAME(devredop, type)() { \
    oneRankReduce<type, Func##devredop<type>>(); \
  }

INSTANTIATE(PreMulSum, int8_t)
INSTANTIATE(PreMulSum, uint8_t)
INSTANTIATE(PreMulSum, int32_t)
INSTANTIATE(PreMulSum, uint32_t)
INSTANTIATE(PreMulSum, int64_t)
INSTANTIATE(PreMulSum, uint64_t)
INSTANTIATE(PreMulSum, half)
#if defined(__CUDA_BF16_TYPES_EXIST__)
INSTANTIATE(PreMulSum, __hip_bfloat16)
#endif
INSTANTIATE(PreMulSum, float)
INSTANTIATE(PreMulSum, double)
