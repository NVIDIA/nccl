#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <algorithm>

#include <nvToolsExt.h>

#include "core.h"
#include "common_kernel.h"
#include "copy_kernel.h"
#include "enqueue.h"

/* HIERARCHY
 *
 * The data is split into CHUNKS, and each CHUNK is split into NUM_SUBCHUNKS
 * SUBCHUNKS, where each SUBCHUNK is processed independently. A SUBCHUNK is
 * split into numUnroll UNROLLS and each thread performs UNROLL_COUNT
 * single-data-element operations inside an UNROLL. As the name suggests, the
 * UNROLL_COUNT operations within an UNROLL are unrolled.
*/

// Number of threads used to perform copies, etc. Must be multiple of 32.
// An additional thread is used to handle threadfences, so the CUDA blocks
// have dimension NUM_THREADS+1.
#define NUM_THREADS     256

// Each thread unrolls the innermost loop of the copy or reduction operations
// to this many single-data-element instructions
#define UNROLL_COUNT    8

#define UNROLL_SIZE     (UNROLL_COUNT * NUM_THREADS)

// To hide the latency associated with the synchronization between different
// subchunks, we interleave the independent subchunks so that more data can be
// transferred while the sync is in progress. This is the number of subchunks
// that are active at the same time
#define NUM_SUBCHUNKS   4

// if this is called with CHUNK, it means that we just finished pushing the data
// of chunk CHUNK to the next GPU, so it can proceed with CHUNK
// We add 1 to chunk so that the initial flag of 0 doesn't allow the non-root
// GPUs to proceed before the flag is incremented from the upstream GPU. This
// is called by one particular consumer warp and so we select the first thread
// in the warp to set the flag.
#define SIGNAL_NEW_DATA_AVAILABLE(chunk, subchunk)                              \
    do {                                                                        \
      __threadfence_system();                                                   \
      args.NextNewDataAvailableFlag[0] = NUM_SUBCHUNKS*(chunk) + subchunk + 1;  \
    } while (0)

// This is called by all producer threads, but only thread 0 spins on the flag,
#define WAIT_FOR_NEW_DATA(chunk, subchunk)                                      \
    do {                                                                        \
      if (tid == 0) {                                                           \
        Wait([=] {                                                              \
          return ((volatile int *)args.ThisNewDataAvailableFlag)[0] >=          \
              NUM_SUBCHUNKS*(chunk) + subchunk + 1;                             \
        });                                                                     \
      }                                                                         \
      BAR(sync, 1, NUM_THREADS);                                                \
    } while (0)

// If this is called with CHUNK, it means that this GPU has just finished
// processing the chunk CHUNK and so the previous GPU can start with CHUNK + 1
#define SIGNAL_CHUNK_DONE(chunk, subchunk)                                      \
    do {                                                                        \
      args.PrevChunkDoneFlag[0] = NUM_SUBCHUNKS*(chunk) + subchunk + 1;         \
    } while (0)

// This is called by all producer threads, but only thread 0 spins on the flag,
// all threads synchronize after thread 0 is done spinning.
#define WAIT_FOR_CHUNK(chunk, subchunk)                                         \
    do {                                                                        \
      if (tid == 0) {                                                           \
        Wait([=] {                                                              \
          return ((volatile int *)args.ThisChunkDoneFlag)[0] >=                 \
              NUM_SUBCHUNKS*(chunk) + subchunk + 1 - NUM_SUBCHUNKS;             \
        });                                                                     \
      }                                                                         \
      BAR(sync, 1, NUM_THREADS);                                                \
    } while (0)

// This is called by all producer threads, but only thread 0 spins on the flag,
// all threads synchronize after thread 0 is done spinning.
#define WAIT_FOR_NEW_DATA_AND_CHUNK(chunk, subchunk)                            \
    do {                                                                        \
      if (tid == 0) {                                                           \
        Wait([=] {                                                              \
          bool newDataAvailable =                                               \
              ((volatile int *)args.ThisNewDataAvailableFlag)[0] >=             \
                  NUM_SUBCHUNKS*(chunk) + subchunk + 1;                         \
          bool chunkDone =                                                      \
              ((volatile int *)args.ThisChunkDoneFlag)[0] >=                    \
                  NUM_SUBCHUNKS*(chunk)+subchunk + 1 - NUM_SUBCHUNKS;           \
          return newDataAvailable && chunkDone;                                 \
        });                                                                     \
      }                                                                         \
      BAR(sync, 1, NUM_THREADS);                                                \
    } while (0)

__device__ inline void getSliceSizeAndOffset(int *size, int *offset, int slice,
    int numSlices, int numBigSlices, int numSmallSlices, int bigSliceN,
    int smallSliceN, int lastSliceN) {
  if (slice < numBigSlices) {
    *size = bigSliceN;
    *offset = slice * bigSliceN;
  } else {
    *size = (slice < numBigSlices + numSmallSlices) ? smallSliceN
        : ((slice == numSlices - 1) ? lastSliceN : 0);
    *offset = numBigSlices * bigSliceN + (slice - numBigSlices) * smallSliceN;
  }

//  if (threadIdx.x == 0)
//    printf("[size=%d] [offset=%d] slice=%d numSlices=%d "
//        "numBigSlices=%d numSmallSlices=%d bigSliceN=%d smallSliceN=%d "
//        "lastSliceN=%d\n", *size, *offset, slice, numSlices, numBigSlices,
//        numSmallSlices, bigSliceN, smallSliceN, lastSliceN);
}

template<typename T>
struct BroadcastKernelArgs {
  // general parameters
  int ThisId;
  int N;

  // some pre-computed sizes
  int SliceSize;
  int ChunkSize;
  int NumChunks;
  int BufferSliceStride;

  T ** ThisPtrToNextData;
  T ** PrevPtrToThisData;

  // local and remote data
  T * __restrict__ ThisData;
  volatile T * __restrict__ ThisBuffer;
  volatile T * __restrict__ NextBuffer;

  // local and remote flags
  volatile int * __restrict__ ThisNewDataAvailableFlag;
  volatile int * __restrict__ NextNewDataAvailableFlag;
  volatile int * __restrict__ ThisChunkDoneFlag;
  volatile int * __restrict__ PrevChunkDoneFlag;
};

__shared__ volatile void * nextData;
enum BcastRole {ROOT=0, MIDDLE=1, END=2};

template<int THREADS, int UNROLL, bool PUSHRECV, int ROLE, typename T>
__global__ void BroadcastKernel(const BroadcastKernelArgs<T> args) {
  if (args.N == 0) return;
  int tid = threadIdx.x;

  // First wait for args.PrevPtrToThisOutput to become nullptr to ensure that
  // the previous GPU is done with a previous collective operation.
  if (tid == 0) {
    Wait([=] {
      return *((T * volatile *)args.PrevPtrToThisData) == nullptr; // Wait for previous processor to be done
    });

    *((T * volatile *)args.PrevPtrToThisData) = (T*)args.ThisData; // Tell Previous I'm starting
    Wait([=] {
      return *((T * volatile *)args.ThisPtrToNextData) != nullptr;  // Wait till I've been told next started
    });

    if (PUSHRECV)
      nextData = *((volatile void * volatile *)args.ThisPtrToNextData); // Grab next's pointer if needed.
  }
  __syncthreads();

  for (int chunk = 0; chunk < args.NumChunks; ++chunk) {
    // calculate slice size.  for all chunks except (possibly) the last one,
    // this will just be args.SliceSize. For the last one, it may be smaller
    int bigSliceN   = args.SliceSize;
    int smallSliceN = 0;
    int lastSliceN  = 0;
    int numSlices   = NUM_SUBCHUNKS;
    int numBigSlices   = numSlices;
    int numSmallSlices = 0;

    // last chunk
    if ((chunk + 1 == args.NumChunks) && (args.N % args.ChunkSize > 0))
      CalcLastChunk<THREADS, UNROLL, T>(&bigSliceN, &smallSliceN, &lastSliceN,
          &numSlices, &numBigSlices, &numSmallSlices, args.N, args.NumChunks,
          args.ChunkSize);

    // this offset is only applied to Data pointers, not to Buffer pointers,
    // since we only have one buffer per chunk
    int chunkOffset = chunk * args.ChunkSize;

    int offset;
    int sliceSize;

    if (tid < THREADS) {
      for(int s=0; s<NUM_SUBCHUNKS; ++s) {
        getSliceSizeAndOffset(&sliceSize, &offset, s, numSlices,
            numBigSlices, numSmallSlices, bigSliceN, smallSliceN, lastSliceN);

        if (PUSHRECV) {
          if (ROLE != ROOT)
            WAIT_FOR_NEW_DATA(chunk, s);

          if (ROLE != END)
            Copy<UNROLL, THREADS>(
                (volatile T *)nextData + chunkOffset + offset,
                args.ThisData + chunkOffset + offset,
                sliceSize);
        } else { // PUSH2BUFF
          if (ROLE == ROOT) {
            WAIT_FOR_CHUNK(chunk, s);

            Copy<UNROLL, THREADS>(
                args.NextBuffer + (s * args.BufferSliceStride),
                args.ThisData + chunkOffset + offset,
                sliceSize);
          } else if (ROLE == MIDDLE) {
            WAIT_FOR_NEW_DATA_AND_CHUNK(chunk, s);

            DoubleCopy<UNROLL, THREADS>(
                args.NextBuffer + (s * args.BufferSliceStride),
                args.ThisData + chunkOffset + offset,
                args.ThisBuffer + (s * args.BufferSliceStride),
                sliceSize);
          } else { // ROLE == END
            WAIT_FOR_NEW_DATA(chunk, s);

            Copy<UNROLL, THREADS>(
                args.ThisData + chunkOffset + offset,
                args.ThisBuffer + (s * args.BufferSliceStride),
                sliceSize);
          }
        }
        __syncthreads();
      }
    } else { // Consumer thread
      for(int s=0; s<NUM_SUBCHUNKS; ++s) {
        __syncthreads();
        if (ROLE != END)
          SIGNAL_NEW_DATA_AVAILABLE(chunk, s);

        // signal chunk done if we don't push into the receive buffer and this
        // is no the last chunk and this is not root
        if ((!PUSHRECV) && (ROLE != ROOT) && (chunk + 1 < args.NumChunks)) {
          SIGNAL_CHUNK_DONE(chunk, s);
        }
      }
    }
  }

  // reset flags
  if (tid == 0) {
    args.ThisNewDataAvailableFlag[0] = 0;
    args.ThisChunkDoneFlag[0] = 0;
    *args.ThisPtrToNextData = nullptr;
  }
}

template<typename T>
ncclResult_t ncclBcastWithType(void* buff, const int count, const int root,
    ncclComm* comm, int numUnroll, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  int index = comm->ncclId;
  int rootId = comm->ringFromUser[root];

  int nextId = (index + 1) % comm->nDev;
  int prevId = (index + comm->nDev - 1) % comm->nDev;

  // There is one slice per GPU, so a slice can be at most bufferN / numGPUs,
  // where bufferN is the number of elements of type T that fit into the buffer.
  // For efficiency, we want the slice size to be a multiple of UNROLL_SIZE
  int bufferN = comm->buffSize / sizeof(T);
  // we only need buffer for k slices and k paddings
  int bufferNPerSlice = bufferN / NUM_SUBCHUNKS;
  int maxSliceSize = (bufferNPerSlice / UNROLL_SIZE) * UNROLL_SIZE;

  BroadcastKernelArgs<T> args;

  args.ThisId = index;
  args.N = count;

  args.SliceSize = numUnroll * UNROLL_SIZE * sizeof(PackType) / sizeof(T);

  // if we don't directly push into the remote receive buffer, make sure slice
  // fits into the temporary buffer
  if (!comm->useRemoteRecv) {
    // Larger transfers help QPI more than tag updates hurt P2P.
    args.SliceSize *= 8;
  }

  args.SliceSize = std::min(maxSliceSize, args.SliceSize);
  args.BufferSliceStride = args.SliceSize;
  args.ChunkSize = NUM_SUBCHUNKS * args.SliceSize;

  // avoid a case where we have one or more big chunks and one tiny one
  int remainder = args.N % args.ChunkSize;
  if ((args.N > args.ChunkSize) && (remainder > 0) &&
      (args.N < 5 * args.ChunkSize) && (2 * remainder < args.ChunkSize)) {
    args.SliceSize /= 2;
    args.ChunkSize = NUM_SUBCHUNKS * args.SliceSize;

    // round down so we end up with a big last chunk
    args.NumChunks = args.N / args.ChunkSize;
  } else {
    // round up
    args.NumChunks = (args.N + args.ChunkSize - 1) / args.ChunkSize;
  }

//  printf("sliceSize = %i, chunkSize = %i, numChunks = %i\n", args.SliceSize, args.ChunkSize, args.NumChunks);

  args.ThisPtrToNextData = (T**)&(comm->local[nextId]->recvPtrs[0]);
  args.PrevPtrToThisData = (T**)&(comm->remote[prevId]->recvPtrs[0]);

  args.ThisData = (T*)buff;
  args.ThisBuffer = (volatile T*)comm->local[prevId]->buff;
  args.NextBuffer = (volatile T*)comm->remote[nextId]->buff;

  // we need 2 * NUM_SUBCHUNKS flags, so use the first NUM_SUBCHUNKS flags
  // to signal the next GPU that new data is available and the following
  // NUM_SUBCHUNKS to signal the previous GPU that a chunk is finished
  args.ThisNewDataAvailableFlag = comm->local[prevId]->flags;
  args.NextNewDataAvailableFlag = comm->remote[nextId]->flags;
  args.ThisChunkDoneFlag = comm->local[nextId]->flags + 1;
  args.PrevChunkDoneFlag = comm->remote[prevId]->flags + 1;

  if (comm->useRemoteRecv) {
    if (index == (rootId + comm->nDev - 1) % comm->nDev) {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, true, END, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    } else if (index == rootId) {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, true, ROOT, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    } else {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, true, MIDDLE, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    }
  } else {
    if (index == (rootId + comm->nDev - 1) % comm->nDev) {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, false, END, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    } else if (index == rootId) {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, false, ROOT, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    } else {
      BroadcastKernel<NUM_THREADS, UNROLL_COUNT, false, MIDDLE, T>
          <<<1, NUM_THREADS + 1, 0, stream>>>(args);
    }
  }
  return ncclSuccess;
}

class BroadcastFunctor {
public:
  ncclResult_t operator()(const void* /*dummy sendbuff*/,
      void* buff, int count, ncclDataType_t datatype, ncclRedOp_t /*dummy operation*/,
      int root, ncclComm* comm, hipStream_t stream) {
    int numUnroll = 4;

    switch (datatype) {
    case ncclChar:
      return ncclBcastWithType<char>(buff, count, root, comm, numUnroll, stream);
    case ncclInt:
      return ncclBcastWithType<int>(buff, count, root, comm, numUnroll, stream);
#ifdef CUDA_HAS_HALF
    case ncclHalf:
      return ncclBcastWithType<half>(buff, count, root, comm, numUnroll, stream);
#endif
    case ncclFloat:
      return ncclBcastWithType<float>(buff, count, root, comm, numUnroll, stream);
    case ncclDouble:
      return ncclBcastWithType<double>(buff, count, root, comm, numUnroll, stream);
    case ncclInt64:
      return ncclBcastWithType<long long>(buff, count, root, comm, numUnroll, stream);
    case ncclUint64:
      return ncclBcastWithType<unsigned long long>(buff, count, root, comm, numUnroll, stream);
    }
    return ncclInvalidType;
  }
};

extern "C" DSOGLOBAL
ncclResult_t ncclBcast(void* buff, int count, ncclDataType_t datatype, int root,
    ncclComm_t comm, hipStream_t stream) {
  return enqueue(BroadcastFunctor(), nullptr, buff, count, datatype, ncclSum,
      root, comm, stream);
}

