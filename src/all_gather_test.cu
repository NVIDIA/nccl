#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "nccl.h"
#include "test_utilities.h"


template<typename T>
void RunTest(T** sendbuff, T** recvbuff, const int N, const ncclDataType_t type,
    ncclComm_t* const comms, const std::vector<int>& dList) {
  // initialize data
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  T* buffer = (T*)malloc(nDev * N * sizeof(T));
  T* result = (T*)malloc(nDev * N * sizeof(T));
  memset(buffer, 0, nDev * N * sizeof(T));
  memset(result, 0, nDev * N * sizeof(T));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamCreate(s+i));
    CUDACHECK(hipMemset(recvbuff[i], 0, nDev * N * sizeof(T)));
    Randomize(sendbuff[i], N, i);

    CUDACHECK(hipMemcpy(result + i * N, sendbuff[i], N * sizeof(T),
        hipMemcpyDeviceToHost));
  }

  // warm up GPU
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    ncclAllGather((const void*)sendbuff[i], std::min(32 * 1024, N), type,
        (void*)recvbuff[i], comms[i], s[i]);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //for (int n = 1; n <= N; n = n << 1)
  {
    int n = N;
    printf("%12i  %12i  %6s", (int)(n * sizeof(T)), n, TypeName(type).c_str());

    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      ncclAllGather((const void*)sendbuff[i], n, type, (void*)recvbuff[i], comms[i],
          s[i]);
    }

    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      CUDACHECK(hipStreamSynchronize(s[i]));
    }

    auto stop = std::chrono::high_resolution_clock::now();

    double elapsedSec =
        std::chrono::duration_cast<std::chrono::duration<double>>(
            stop - start).count();
    double algbw = (double)(n * sizeof(T)) / 1.0E9 * (double)(nDev - 1)
        / elapsedSec;
    double busbw = algbw;

    double maxDelta = 0.0;
    for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(dList[i]));
      double tmpDelta = CheckDelta<T>(recvbuff[i], result, nDev*N);
      maxDelta = std::max(tmpDelta, maxDelta);
    }

    printf("  %7.3f  %5.2f  %5.2f  %7.0le\n", elapsedSec * 1.0E3, algbw, busbw,
        maxDelta);
  }

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipStreamDestroy(s[i]));
  }
  free(s);
  free(buffer);
  free(result);
}

template<typename T>
void RunTests(const int N, const ncclDataType_t type, ncclComm_t* const comms,
    const std::vector<int>& dList) {
  int nDev = 0;
  ncclCommCount(comms[0], &nDev);
  T** sendbuff = (T**)malloc(nDev * sizeof(T*));
  T** recvbuff = (T**)malloc(nDev * sizeof(T*));

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipMalloc(sendbuff + i, N * sizeof(T)));
    CUDACHECK(hipMalloc(recvbuff + i, nDev * N * sizeof(T)));
  }

  RunTest<T>(sendbuff, recvbuff, N, type, comms, dList);

  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(dList[i]));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  free(sendbuff);
  free(recvbuff);
}

void usage() {
  printf("Tests nccl AllGather with user supplied arguments.\n"
      "    Usage: all_reduce_test <data size in bytes> [number of GPUs] "
      "[GPU 0] [GPU 1] ...\n\n");
}

int main(int argc, char* argv[]) {
  int nVis = 0;
  CUDACHECK(hipGetDeviceCount(&nVis));

  int N = 0;
  if (argc > 1) {
    int t = sscanf(argv[1], "%d", &N);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  } else {
    printf("Error: must specify at least data size in bytes!\n\n");
    usage();
    exit(EXIT_FAILURE);
  }

  int nDev = nVis;
  if (argc > 2) {
    int t = sscanf(argv[2], "%d", &nDev);
    if (t == 0) {
      printf("Error: %s is not an integer!\n\n", argv[1]);
      usage();
      exit(EXIT_FAILURE);
    }
  }
  std::vector<int> dList(nDev);
  for (int i = 0; i < nDev; ++i)
    dList[i] = i % nVis;


  if (argc > 3) {
    if (argc - 3 != nDev) {
      printf("Error: insufficient number of GPUs in list\n\n");
      usage();
      exit(EXIT_FAILURE);
    }

    for (int i = 0; i < nDev; ++i) {
      int t = sscanf(argv[3 + i], "%d", dList.data() + i);
      if (t == 0) {
        printf("Error: %s is not an integer!\n\n", argv[2 + i]);
        usage();
        exit(EXIT_FAILURE);
      }
    }
  }

  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nDev);
  ncclCommInitAll(comms, nDev, dList.data());

  printf("# Using devices\n");
  for (int g=0; g<nDev; ++g) {
    int cudaDev;
    int rank;
    hipDeviceProp_t prop;
    ncclCommCuDevice(comms[g], &cudaDev);
    ncclCommUserRank(comms[g], &rank);
    CUDACHECK(hipGetDeviceProperties(&prop, cudaDev));
    printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev,
        prop.pciBusID, prop.name);
  }
  printf("\n");

  printf("# %10s  %12s  %6s  %7s  %5s  %5s  %7s\n",
      "bytes", "N", "type", "time", "algbw", "busbw", "delta");

  RunTests<char>(N / sizeof(char), ncclChar, comms, dList);
  RunTests<int>(N / sizeof(int), ncclInt, comms, dList);
#if CUDART_VERSION >= 7050
  RunTests<half>(N / sizeof(half), ncclHalf, comms, dList);
#endif
  RunTests<float>(N / sizeof(float), ncclFloat, comms, dList);
  RunTests<double>(N / sizeof(double), ncclDouble, comms, dList);
  RunTests<long long>(N / sizeof(long long), ncclInt64, comms, dList);
  RunTests<unsigned long long>(N / sizeof(unsigned long long), ncclUint64, comms, dList);

  printf("\n");

  for(int i=0; i<nDev; ++i)
    ncclCommDestroy(comms[i]);
  free(comms);

  exit(EXIT_SUCCESS);
}

