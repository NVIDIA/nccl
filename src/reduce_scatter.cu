#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void ReduceScatterKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  __shared__ DevRing<T> ring;

  LoadRing<THREADS>(args.ring, &ring);
  __syncthreads();

  if (tid == 0) {
    WaitFlag prevCommOp(ring.prevOpCounter, 0);
    WaitFlag nextCommOp(ring.nextOpCounter, 0);
    prevCommOp.wait(args.opIndex);
    nextCommOp.wait(args.opIndex);
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring.recvFlagFromNext, -NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring.recvFlagFromPrev, -1*NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring.sendFlagToPrev, -1*NUM_SUBSTEPS);
  PostFlag postReadyToNext(ring.sendFlagToNext, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = args.nRanks;
  const int buffSize = args.buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput =  args.ThisOutput;
  T * __restrict__ prevInput = ring.recvBuffer;
  T * __restrict__ nextOutput =  ring.sendBuffer;

  for (int chunkOffset = 0; chunkOffset < size; chunkOffset += sliceSize) {
    /////////////// begin ReduceScatter steps ///////////////
    int offset;
    int maxOffset = size-chunkOffset;
    int rankDest;

    // step 0: push data to next GPU
    rankDest = ring.userRank[nranks-1];
    offset = chunkOffset + rankDest * size;

    Prims::Copy(
        thisInput  + offset,
        nextOutput + noffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      rankDest = ring.userRank[nranks-j];
      offset = chunkOffset + rankDest * size;

      Prims::Reduce(
          prevInput  + poffset,
          thisInput  + offset,
          nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    rankDest = ring.userRank[0];
    offset = chunkOffset + rankDest * size;

    Prims::Reduce(
        prevInput  + poffset,
        thisInput  + offset,
        thisOutput + chunkOffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP;
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    // Wait for last update from next then reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
    *ring.recvFlagFromNext = 0;

    // Wait for last update from prev then reset the flag
    waitReadyFromPrev.wait(NUM_SUBSTEPS*(step+1));
    *ring.recvFlagFromPrev = 0;

    incrementOpCounter(&args);
  }
}

#define THREADS 512
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingReduceScatter(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, 0, count, comm);
    LAUNCH_KERNEL(ReduceScatterKernel, THREADS, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class ReduceScatter {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingReduceScatter<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclReduceScatter, const void* sendbuff, void* recvbuff, int recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream);
ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, int recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  return enqueue<ReduceScatter>(sendbuff, recvbuff, recvcount, datatype, op, 0, comm, stream);
}

