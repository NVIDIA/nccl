#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "device.h"
#include "collectives.h"
#include "common.h"

__shared__ ncclShmemData ncclShmem;
#if __CUDA_ARCH__ < 700
  __shared__ ulong2 ncclShmemPerWarp[ncclShmemScratchWarpSize()*(NCCL_MAX_NTHREADS/WARP_SIZE)/sizeof(ulong2)];
#endif

struct RunWorkNop {
  __device__ void run(ncclWork *w) {}
};

__global__ void ncclDevKernel_Generic(struct ncclDevComm* comm, uint64_t channelMask, struct ncclWork* workHead) {
  ncclKernelMain<-1, RunWorkNop>(comm, channelMask, workHead);
}

__device__ void ncclDevFunc_Nop() {}
