/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "nccl.h"
#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclReduceScatterFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, ncclProf_t* nccl_prof) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(COLL,"ReduceScatter: opCount %lx sendbuff %p recvbuff %p count %zi datatype %d op %d root %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, datatype, op, root, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(REDUCESCATTER_SUBSTEPS, REDUCESCATTER_BUFCHUNKS, comm->nRanks-1, comm->nRanks, nbytes*comm->nRanks, proxyPatternRing, comm, nccl_prof));
    NCCLCHECK(saveKernel(ncclCollReduceScatter, sendbuff, recvbuff, count, datatype, op, root, comm, stream, nbytes*comm->nRanks, 1));
  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclReduceScatter, const void* sendbuff, void* recvbuff, size_t recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream, ncclProf_t* nccl_prof);
ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, size_t recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream, ncclProf_t* nccl_prof) {
  return ncclEnqueueCheck(ncclReduceScatterFunc, "ReduceScatter", sendbuff, recvbuff, recvcount, datatype,
          op, 0, comm, stream, nccl_prof);
}
