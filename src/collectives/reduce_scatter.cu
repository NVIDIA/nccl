/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclReduceScatterFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(NCCL_COLL,"ReduceScatter: opCount %lx sendbuff %p recvbuff %p count %zi datatype %d op %d root %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, datatype, op, root, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(REDUCESCATTER_SUBSTEPS, REDUCESCATTER_BUFCHUNKS, comm->nRanks-1, comm->nRanks, nbytes*comm->nRanks, proxyPatternRing, comm));
    NCCLCHECK(saveKernel(ncclCollReduceScatter, sendbuff, recvbuff, count, datatype, op, root, comm, stream, nbytes*comm->nRanks, 1));
  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclReduceScatter, const void* sendbuff, void* recvbuff, size_t recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream);
ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, size_t recvcount,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, hipStream_t stream) {
  return ncclEnqueueCheck(ncclReduceScatterFunc, "ReduceScatter", sendbuff, recvbuff, recvcount, datatype,
          op, 0, comm, stream);
}
