#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "enqueue.h"
#include "primitives.h"

#define NUM_SUBSTEPS 2
#define NUM_BUFCHUNKS 2

// Increase Step and poffset/noffset for buffer sync
#define NEXT_STEP \
  step++; \
  poffset = noffset; \
  noffset += sliceSize; \
  if (noffset == buffSize) noffset = 0;

#define ALIGN_SIZE(size, align) \
  size = ((size + (align) - 1) / (align)) * (align);

template<int THREADS, int UNROLL, class FUNC, typename T>
__launch_bounds__(THREADS+WARP_SIZE, 1)
__global__ void AllReduceKernel(const KernelArgs<T> args) {
  const int tid = threadIdx.x;
  __shared__ T* sharedNextOutput;
  __shared__ DevRing<T> ring;
  bool pushrecv = args.pushrecv;

  LoadRing<THREADS>(args.ring, &ring);
  __syncthreads();

  if (tid == 0) {
    WaitFlag prevCommOp(ring.prevOpCounter, 0);
    WaitFlag nextCommOp(ring.nextOpCounter, 0);
    prevCommOp.wait(args.opIndex);
    nextCommOp.wait(args.opIndex);
    if (pushrecv) {
      *ring.sendPtrToPrev = (T*)args.ThisOutput;
      Wait([=] {
        return *ring.recvPtrFromNext != nullptr;
      });
      sharedNextOutput = *ring.recvPtrFromNext;
      *ring.recvPtrFromNext = nullptr;
    }
  }
  __syncthreads();

  WaitFlag waitDoneFromNext(ring.recvFlagFromNext, -NUM_BUFCHUNKS*NUM_SUBSTEPS);
  WaitFlag waitReadyFromPrev(ring.recvFlagFromPrev, -1*NUM_SUBSTEPS);
  PostFlag postDoneToPrev(ring.sendFlagToPrev, -1*NUM_SUBSTEPS);
  PostFlag postReadyToNext(ring.sendFlagToNext, 0);

  typedef Primitives<THREADS, UNROLL, NUM_SUBSTEPS, T, FUNC> Prims;

  const int size = args.N;
  const int nranks = args.nRanks;
  const int buffSize = args.buffSize / sizeof(T);
  const int sliceSize = buffSize / NUM_BUFCHUNKS;
  
  int step = 0;
  int poffset, noffset = 0;

  // Compute pointers
  const T * __restrict__ thisInput = args.ThisInput;
  T * __restrict__ thisOutput =  args.ThisOutput;
  T * __restrict__ prevInput = ring.recvBuffer;
  T * __restrict__ nextOutput =  ring.sendBuffer;

  for (int chunkOffset = 0; chunkOffset < size; chunkOffset += nranks*sliceSize) {
    /////////////// begin AllReduce steps ///////////////
    int offset;
    int maxOffset;
    int slice;
    int chunkSize = min(sliceSize, DIVUP(size-chunkOffset,nranks));
    ALIGN_SIZE(chunkSize, THREADS*UNROLL);

    // step 0: push data to next GPU
    slice = ring.userRank[nranks-1];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::Copy(
        thisInput  + offset,
        nextOutput + noffset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP; // Increases step, poffset, noffset

    // k-2 steps: reduce and copy to next GPU
    for (int j=2; j<nranks; ++j) {
      slice = ring.userRank[nranks-j];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      Prims::Reduce(
          prevInput  + poffset,
          thisInput  + offset,
          nextOutput + noffset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }

    // step k-1: reduce this buffer and data, which will produce the final
    // result that we store in this data and push to the next GPU
    slice = ring.userRank[0];
    offset = chunkOffset + slice * chunkSize;
    maxOffset = min(chunkSize, size-offset);

    Prims::ReduceCopy(
        prevInput  + poffset,
        thisInput  + offset,
        pushrecv ? (sharedNextOutput + offset) : (nextOutput + noffset),
        thisOutput + offset,
        sliceSize, maxOffset,
        step,
        waitDoneFromNext, waitReadyFromPrev,
        postReadyToNext, postDoneToPrev);

    NEXT_STEP;

    if (pushrecv) {
      // k-2 steps: copy result to next GPU
      for (int j=1; j<nranks-1; ++j) {
        slice = ring.userRank[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::Copy(
            thisOutput + offset,
            sharedNextOutput + offset,
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }
    } else {
      // k-2 steps: copy result to next GPU
      for (int j=1; j<nranks-1; ++j) {
        slice = ring.userRank[nranks - j];
        offset = chunkOffset + slice * chunkSize;
        maxOffset = min(chunkSize, size-offset);

        Prims::DoubleCopy(
            prevInput + poffset,
            thisOutput + offset,
            nextOutput + noffset,
            sliceSize, maxOffset,
            step,
            waitDoneFromNext, waitReadyFromPrev,
            postReadyToNext, postDoneToPrev);

        NEXT_STEP;
      }

      // Make final copy from buffer to dest.
      slice = ring.userRank[1];
      offset = chunkOffset + slice * chunkSize;
      maxOffset = min(chunkSize, size-offset);

      // Here we need to copy from buffer to this output.
      Prims::Copy(
          prevInput + poffset,
          thisOutput + offset,
          sliceSize, maxOffset,
          step,
          waitDoneFromNext, waitReadyFromPrev,
          postReadyToNext, postDoneToPrev);

      NEXT_STEP;
    }
  }

  // wait for the last data to be pushed to us
  if (tid == 0) {
    // Wait for last update from next then reset the flag
    waitDoneFromNext.wait(NUM_SUBSTEPS*(step+NUM_BUFCHUNKS-1));
    *ring.recvFlagFromNext = 0;

    // Wait for last update from prev then reset the flag
    waitReadyFromPrev.wait(NUM_SUBSTEPS*(step+1));
    *ring.recvFlagFromPrev = 0;

    incrementOpCounter(&args);
  }
}

#define THREADS 512
#define UNROLL 8

template<class FUNC, typename T>
ncclResult_t RingAllReduce(const void* sendbuff, void* recvbuff,
    const int count, ncclComm* comm, hipStream_t stream) {
  if (count == 0)
    return ncclSuccess;

  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, count*sizeof(T), hipMemcpyDeviceToDevice, stream));
  } else {
    KernelArgs<T> args;
    ArgsSetup(&args, sendbuff, recvbuff, 0, count, comm);
    LAUNCH_KERNEL(AllReduceKernel, THREADS, UNROLL, FUNC, T, args, stream);
  }

  return ncclSuccess;
}

template<typename T, template <typename> class RedOp>
class AllReduce {
  public:
  static ncclResult_t entry(const void* sendbuff, void* recvbuff,
      int count, int /*root*/, ncclComm* comm, hipStream_t stream) {
    return RingAllReduce<RedOp<T>, T>(sendbuff, recvbuff, count, comm, stream);
  }
};

NCCL_API(ncclResult_t, ncclAllReduce, const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream);
ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, int count,
    ncclDataType_t datatype, ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  return enqueue<AllReduce>(sendbuff, recvbuff, count, datatype, op, 0, comm, stream);
}

