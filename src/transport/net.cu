#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "transport.h"
#include "nvmlwrap.h"
#include "net.h"
#include "param.h"
#include "nvlink.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#define NET_MAX_IFS 16

// We encode 3 bits of distance per interface into a ncclTvalue_t (64-bit)
#define NET_BITS_PER_IF 3
#define NET_BITS_PER_IF_MASK ((1<<NET_BITS_PER_IF)-1)
static_assert(sizeof(ncclTvalue_t)*8 >= NET_MAX_IFS*NET_BITS_PER_IF, "NET_MAX_IFS*NET_BITS_PER_IF must fit in a ncclTvalue_t");
static ncclTvalue_t getTvalue(short* distances, int ndev) {
  ncclTvalue_t tvalue = 0;
  for (int d=0; d<ndev; d++) {
    int score = 1 + PATH_SOC - distances[d];
    // Keep 3 bits of score info per dev
    tvalue |= ((score & NET_BITS_PER_IF_MASK)<<(NET_BITS_PER_IF*d));
  }
  return tvalue;
}

struct netInfo {
  int rank;
  int ndev;
  ncclTvalue_t tValue;
  short distances[NET_MAX_IFS];
};

struct netConnectInfo {
  ncclNetHandle_t netHandle;
};

struct netSendResources {
  void* netSendComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclSendMem* hostDevMem;
  int netDev;
  int useGdr;
  struct ncclRecvMem* devNetMem;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

struct netRecvResources {
  void* netListenComm;
  void* netRecvComm;
  struct ncclSendMem* hostSendMem;
  struct ncclRecvMem* hostRecvMem;
  struct ncclSendMem* devHostSendMem;
  struct ncclRecvMem* devHostRecvMem;
  struct ncclRecvMem* hostDevMem;
  int netDev;
  int useGdr;
  uint64_t llStep;
  uint64_t llLastCleaning;
};

/* Fill information necessary to exchange between ranks to choose whether or not
 * to use this transport */
ncclResult_t netFillInfo(ncclTinfo_t* opaqueInfo, int rank) {
  struct netInfo* info = (struct netInfo*)opaqueInfo;
  static_assert(sizeof(struct netInfo) <= sizeof(ncclTinfo_t), "NET Info too large");
  info->rank = rank;
  NCCLCHECK(ncclNetDevices(&info->ndev));
  if (info->ndev == 0) {
    WARN("Error : Network returned 0 device");
    return ncclSystemError;
  }
  if (info->ndev > NET_MAX_IFS) info->ndev = NET_MAX_IFS;

  // Find distance with current GPU
  int cudaDev;
  hipGetDevice(&cudaDev);
  char* cudaPath;
  NCCLCHECK(getCudaPath(cudaDev, &cudaPath));

  char line[1024];
  sprintf(line, "CUDA Dev %d, %s NIC distance : ", cudaDev, ncclNetName());
  for (int d=0; d<info->ndev; d++) {
    char* nicPath;
    ncclResult_t err = ncclNetPciPath(d, &nicPath);
    info->distances[d] = (err != ncclSuccess || nicPath == NULL || cudaPath == NULL) ? PATH_SOC : pciDistance(nicPath, cudaPath);
    sprintf(line+strlen(line), " %s", pathDists[info->distances[d]]);
    if (err == ncclSuccess) free(nicPath);
  }
  INFO(NCCL_INIT|NCCL_NET, "%s", line);
  free(cudaPath);
  return ncclSuccess;
}

/* Determine if we can communicate with the peer */
ncclResult_t netCanConnect(ncclTvalue_t* ret, ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo) {
  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  ret[0] = getTvalue(myInfo->distances, myInfo->ndev);
  return ncclSuccess;
}

static inline int groupBestStart(int nranks, int* groups, int group, ncclTvalue_t* values, int card, int minScore) {
  int bestRank = -1;
  int bestScore = 0;
  for (int rank=0; rank<nranks; rank++) {
    if (groups[rank] != group) continue;
    for (int i=0; i<nranks; i++) {
      ncclTvalue_t netValue = values[rank*nranks+i];
      if (netValue != 0) {
        ncclTvalue_t score = (netValue>>(NET_BITS_PER_IF*card)) & NET_BITS_PER_IF_MASK;
        if (score >= minScore && score > bestScore) {
          bestScore = score;
          bestRank = rank;
        }
        // All other values should be the same, stop here for this rank
        break;
      }
    }
  }
  return bestRank;
}
static inline int groupBestEnd(int nranks, int* groups, int group, int* subgroups, int startSubGroup, int startRank, ncclTvalue_t* values, int card, int minScore) {
  // For the last rank, we don't need the absolute best score, just to be within minScore.
  for (int rank=nranks-1; rank>=0; rank--) {
    if (groups[rank] != group) continue;
    if (startSubGroup != -1 && startSubGroup == subgroups[rank]) continue;
    if (startRank == rank) continue;
    for (int i=0; i<nranks; i++) {
      ncclTvalue_t netValue = values[rank*nranks+i];
      if (netValue != 0) {
        ncclTvalue_t score = (netValue>>(NET_BITS_PER_IF*card)) & NET_BITS_PER_IF_MASK;
        if (score >= minScore) {
          return rank;
        }
        // All other values should be the same, stop here for this rank
        break;
      }
    }
  }
  return -1;
}


ncclResult_t netGetRings(int nranks, int* groups, int* subgroups, ncclTvalue_t* values, int* nringsRet, int* prev, int* next, int minScore, int* nthreads) {
  int nGroups = groups[nranks-1] + 1;
  int cardUsed[NET_MAX_IFS*nGroups];
  for (int c=0; c<NET_MAX_IFS*nGroups; c++) cardUsed[c] = 0;

  for (int ring = 0; ring<*nringsRet; ring++) {
    int starts[nGroups];
    int ends[nGroups];
    for (int group = 0; group<nGroups; group++) {
      int nranksInGroup = 0;
      int nsubGroups = 0;
      for (int rank=0; rank<nranks; rank++) if (groups[rank] == group) {
          nranksInGroup++;
          nsubGroups = std::max(subgroups[rank], nsubGroups);
        }
      starts[group] = ends[group] = -1;
      // Receive on the rank closest to the NIC
      for (int card=0; card<NET_MAX_IFS; card++) {
        if (cardUsed[group*NET_MAX_IFS+card] == 1) continue;
        int start = groupBestStart(nranks, groups, group, values, card, minScore);
        // Send from any rank, but best on a different subgroup and close to the NIC also.
        int end = (nranksInGroup == 1) ? start
            : groupBestEnd(nranks, groups, group, subgroups, nsubGroups ? subgroups[start] : -1, start, values, card, minScore);
        //printf("Ring %d, Minscore %d, Card %d, group %d, start = %d, end = %d\n", ring, minScore, card, group, start, end);
        if (start != -1 && end != -1) {
          cardUsed[group*NET_MAX_IFS+card] = 1;
          starts[group] = start;
          ends[group] = end;
          break;
        }
      }
      if (starts[group] == -1 || ends[group] == -1) {
        *nringsRet = ring;
        return ncclSuccess;
      }
    }
    // Link groups together
    for (int group = 0; group<nGroups; group++) {
      int nextGroup = (group+1)%nGroups;
      next[ring*nranks+ends[group]] = starts[nextGroup];
      prev[ring*nranks+starts[nextGroup]] = ends[group];
    }
  }
  return ncclSuccess;
}

int getDev(int ringId, int nDev, short* distances) {
  int minDistance = PATH_SOC;
  for (int d=0; d<nDev; d++) if (distances[d] < minDistance) minDistance = distances[d];
  int skip = ringId+1;
  while (skip) {
    for (int d=0; d<nDev; d++) {
      if (distances[d] == minDistance) {
        skip--;
        if (skip == 0) return d;
      }
    }
  }
  return 0;
}

NCCL_PARAM(NetGdrRead, "NET_GDR_READ", -2);
NCCL_PARAM(NetGdrLevel, "NET_GDR_LEVEL", PATH_PHB);

static ncclResult_t netGetGdrSupport(int dev, int distance, int read, int* useGdr) {
  *useGdr = 0;

  int cudaDev;
  CUDACHECK(hipGetDevice(&cudaDev));

  if (read) { // For reads (sends) only enable under certain conditions
    int gdrReadParam = ncclParamNetGdrRead();
    if (gdrReadParam == 0) return ncclSuccess;
    else if (gdrReadParam < 0) { // default : enable only on DGX2
      char busId[NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE];
      CUDACHECK(hipDeviceGetPCIBusId(busId, NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE, cudaDev));
      int nvlinks = getNumNvlinks(busId);
      if (nvlinks < CONNECT_NVSWITCH || ncclCudaCompCap() < 7) return ncclSuccess;
    }
  }

  // Check if we are close enough that it makes sense to enable GDR
  int netGdrLevel = ncclParamNetGdrLevel();
  if (distance >= netGdrLevel) {
    INFO(NCCL_INIT|NCCL_NET,"NET/%s : GPU Direct RDMA Disabled for GPU %d / HCA %d (distance %d >= %d)", ncclNetName(), cudaDev, dev, distance, netGdrLevel);
    return ncclSuccess;
  }

  // Finally, check if the NIC supports it
  int flags;
  NCCLCHECK(ncclNetPtrSupport(dev, &flags));
  if (flags & NCCL_PTR_CUDA == 0) return ncclSuccess;
  *useGdr = 1;
  INFO(NCCL_INIT|NCCL_NET,"NET/%s : GPU Direct RDMA Enabled for GPU %d / HCA %d (distance %d >= %d), read %d", ncclNetName(), cudaDev, dev, distance, netGdrLevel, read);
  return ncclSuccess;
}

/* Determine if we will use this transport for this peer and return connect
 * information for this peer */
ncclResult_t netSendSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netSendResources* resources;
  NCCLCHECK(ncclCalloc(&resources, 1));
  ring->send.transportResources = resources;

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->distances);
  NCCLCHECK(netGetGdrSupport(resources->netDev, myInfo->distances[resources->netDev], 1, &resources->useGdr));

  int size = offsetof(struct ncclRecvMem, buff)+ring->buffSize;
  if (resources->useGdr) {
    NCCLCHECK(ncclCudaCalloc((char**)(&resources->devNetMem), size));
  }

  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostRecvMem, (void**)&resources->devHostRecvMem, size));
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostSendMem, (void**)&resources->devHostSendMem, size));

  return ncclSuccess;
}

ncclResult_t netRecvSetup(ncclTinfo_t* myOpaqueInfo, ncclTinfo_t* peerOpaqueInfo, struct ncclConnect* connectInfo, struct ncclRing* ring) {
  struct netRecvResources* resources;
  NCCLCHECK(ncclCalloc(&resources, 1));
  ring->recv.transportResources = resources;

  struct netInfo* myInfo = (struct netInfo*)myOpaqueInfo;
  resources->netDev = getDev(ring->id, myInfo->ndev, myInfo->distances);
  NCCLCHECK(netGetGdrSupport(resources->netDev, myInfo->distances[resources->netDev], 0, &resources->useGdr));

  int sendSize = sizeof(struct ncclSendMem);
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostSendMem, (void**)&resources->devHostSendMem, sendSize));

  int recvSize = offsetof(struct ncclRecvMem, buff)+ring->buffSize;
  NCCLCHECK(ncclCudaHostAlloc((void**)&resources->hostRecvMem, (void**)&resources->devHostRecvMem, recvSize));

  struct netInfo* peerInfo = (struct netInfo*)peerOpaqueInfo;
  INFO(NCCL_INIT|NCCL_NET,"Ring %02d : %d -> %d via NET/%s/%d%s%s", ring->id, peerInfo->rank, myInfo->rank, ncclNetName(), resources->netDev,
      resources->useGdr ? "/GDRDMA" : "",
      (resources->hostDevMem != NULL) ? "/GDCopy" : "");
  struct netConnectInfo* info = (struct netConnectInfo*) connectInfo;
  NCCLCHECK(ncclNetListen(resources->netDev, &info->netHandle, &resources->netListenComm));
  return ncclSuccess;
}

ncclResult_t netSendConnect(struct ncclConnect* connectInfo, struct ncclConnector* send) {
  // Setup device pointers
  struct netSendResources* resources = (struct netSendResources*)send->transportResources;

  if (resources->useGdr) {
    send->conn.buff = resources->devNetMem->buff;
    // We don't use devMem for llMode because the CPU has to read the data
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  } else {
    send->conn.buff = resources->devHostRecvMem->buff;
    send->conn.llBuff = resources->devHostRecvMem->llBuff;
  }
  send->conn.tail = &resources->devHostRecvMem->tail;
  send->conn.opCount = &resources->devHostRecvMem->opCount;
  send->conn.fifo = resources->devHostRecvMem->sizesFifo;
  send->conn.llFifo = resources->devHostRecvMem->llSizesFifo;

  if (resources->hostDevMem == NULL) {
    send->conn.head = &resources->devHostSendMem->head;
    send->conn.llHead = &resources->devHostSendMem->llHead;
  }

  // Connect to remote peer
  struct netConnectInfo* info = (struct netConnectInfo*)connectInfo;
  NCCLCHECK(ncclNetConnect(resources->netDev, info->netHandle, &resources->netSendComm));
  return ncclSuccess;
}

/* Connect to this peer */
ncclResult_t netRecvConnect(struct ncclConnect* connectInfo, struct ncclConnector* recv) {
  // Setup device pointers
  struct netRecvResources* resources = (struct netRecvResources*)recv->transportResources;

  recv->conn.head = &resources->devHostSendMem->head;
  recv->conn.llHead = &resources->devHostSendMem->llHead;

  if (resources->useGdr == 0) {
    recv->conn.buff = resources->devHostRecvMem->buff;
    recv->conn.llBuff = resources->devHostRecvMem->llBuff;
  }

  if (resources->hostDevMem == NULL) {
    recv->conn.tail = &resources->devHostRecvMem->tail;
    recv->conn.opCount = &resources->devHostRecvMem->opCount;
  }

  // Finish connection establishment
  NCCLCHECK(ncclNetAccept(resources->netListenComm, &resources->netRecvComm));
  NCCLCHECK(ncclNetCloseListen(resources->netListenComm));

  return ncclSuccess;
}

ncclResult_t netSendFree(void* transportResources) {
  struct netSendResources* resources = (struct netSendResources*)transportResources;
  NCCLCHECK(ncclCudaHostFree(resources->hostSendMem));
  NCCLCHECK(ncclCudaHostFree(resources->hostRecvMem));
  if (resources->useGdr)
    CUDACHECK(hipFree(resources->devNetMem));
  NCCLCHECK(ncclNetCloseSend(resources->netSendComm));
  free(resources);
  return ncclSuccess;
}

ncclResult_t netRecvFree(void* transportResources) {
  struct netRecvResources* resources = (struct netRecvResources*)transportResources;
  NCCLCHECK(ncclCudaHostFree(resources->hostSendMem));
  NCCLCHECK(ncclCudaHostFree(resources->hostRecvMem));
  NCCLCHECK(ncclNetCloseRecv(resources->netRecvComm));
  free(resources);
  return ncclSuccess;
}

ncclResult_t netSendProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netSendResources* resources = (struct netSendResources*) (ring->send.transportResources);
  const int llMode = args->llMode;

  volatile uint64_t* prevTail = &resources->hostRecvMem->tail;
  struct ncclSendMem* prevMem = resources->hostDevMem ? resources->hostDevMem : resources->hostSendMem;
  uint64_t* prevHead = llMode ? &prevMem->llHead : &prevMem->head;
  struct ncclRecvMem* localMem = resources->useGdr ? resources->devNetMem : resources->hostRecvMem;
  char* localBuff = llMode ? resources->hostRecvMem->llBuff : localMem->buff;
  int ptrType = resources->useGdr ? NCCL_PTR_CUDA : NCCL_PTR_HOST;
  volatile int* sizesFifo = llMode ? resources->hostRecvMem->llSizesFifo : resources->hostRecvMem->sizesFifo;
  int buffSize = llMode ? NCCL_LL_BUFF_SIZE : ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  assert(args->substeps <= SIZES_FIFO_SIZE);

  uint64_t head = llMode ? resources->llStep : 0ULL;
  uint64_t tail = llMode ? resources->llStep : 0ULL;
  uint64_t end = head + args->nsteps;

  int idle = 0;
  void* requests[args->substeps];

  if (!args->needProxy) goto nextColl;

  TRACE(NCCL_NET,"opCount %lx head %lx tail %lx end %lx nsteps %d llMode %d", args->opCount, head, tail, end, args->nsteps, llMode);
  TRACE(NCCL_NET,"opCount %lx buffSize %d sliceSize %d ptrType %d", args->opCount, buffSize, sliceSize, ptrType);

  // Update in case we skipped some collectives
  if (llMode == 0) resources->hostRecvMem->opCount = args->opCount;

  while (head < end) {
    idle++;
    if (llMode) {
      if (tail < end && tail < head + args->substeps) {
        int slot = tail%args->substeps;
        int size = sizesFifo[slot];
        if (size != 0) {
          if (size == -1) size = 0;
          uint32_t flag = tail + 1;
          int nFifoLines = DIVUP(size, sizeof(union ncclLLFifoLine));
          size = nFifoLines * sizeof(union ncclLLFifoLine);
          union ncclLLFifoLine* lines = (union ncclLLFifoLine*)(localBuff+slot*sliceSize);
          for (int i=0; i<nFifoLines; i++) {
            volatile uint32_t *f1 = &lines[i].flag1;
            volatile uint32_t *f2 = &lines[i].flag2;
            while (f1[0] != flag || f2[0] != flag);
          }
          NCCLCHECK(ncclNetIsend(resources->netSendComm, lines, size, ptrType, requests+slot));
          if (requests[slot] != NULL) {
            sizesFifo[slot] = size;
            tail++;
            idle = 0;
          }
        }
      }
    } else while (tail < *prevTail) {
        // Send through network
        int slot = tail%args->substeps;
        NCCLCHECK(ncclNetIsend(resources->netSendComm, localBuff+slot*sliceSize, sizesFifo[slot], ptrType, requests+slot));
        if (requests[slot] != NULL) {
          tail++;
          idle = 0;
        }
      }
    if (head < tail) {
      int done;
      int slot = head%args->substeps;
      NCCLCHECK(ncclNetTest(requests[slot], &done, NULL));
      if (done) {
        if (llMode) {
          sizesFifo[slot] = 0;
          // Make sure size is reset to zero before we update the head.
          __sync_synchronize();
        }
        head++;
        *prevHead = head;
        idle = 0;
      }
    }
    if (idle) transportProxyIdle(idle);
  }

  // Reset
  if (llMode == 0) *prevTail = 0;

nextColl:
  if (llMode) {
    resources->llStep += args->nsteps;
    // Don't forget to ack otherwise the GPU won't be able to push data.
    *prevHead = resources->llStep;
    if (resources->llStep > resources->llLastCleaning + NCCL_LL_CLEAN_FREQ) {
      memset(localBuff, 0, NCCL_LL_BUFF_SIZE);
      resources->llStep += NCCL_LL_CHUNKS;
      *prevHead = resources->llStep;
      resources->llLastCleaning = resources->llStep;
    }
  }
  return ncclSuccess;
}

ncclResult_t netRecvProxy(struct ncclProxyArgs* args) {
  struct ncclRing* ring = args->ring;
  struct netRecvResources* resources = (struct netRecvResources*) (ring->recv.transportResources);
  int llMode = args->llMode;

  volatile uint64_t* nextHead = llMode ? &resources->hostSendMem->llHead : &resources->hostSendMem->head;
  struct ncclRecvMem* localMem = resources->useGdr ? ring->devMemRecv : resources->hostRecvMem;
  char* localBuff = llMode ? localMem->llBuff : localMem->buff;
  char* nextBuff = (resources->useGdr == 0 && resources->hostDevMem) ? resources->hostDevMem->buff : NULL;
  int ptrType = resources->useGdr ? NCCL_PTR_CUDA : NCCL_PTR_HOST;
  uint64_t* nextTail = resources->hostDevMem ? &resources->hostDevMem->tail : &resources->hostRecvMem->tail;

  int buffSize = llMode ? NCCL_LL_BUFF_SIZE : ring->buffSize;
  int sliceSize = buffSize / args->substeps;

  uint64_t head = llMode ? resources->llStep : 0ULL;
  uint64_t tail = llMode ? resources->llStep : 0ULL;
  uint64_t end = head + args->nsteps;

  int idle = 0;
  void* requests[args->substeps];

  if (!args->needProxy) goto nextColl;

  TRACE(NCCL_NET,"opCount %lx head %lx tail %lx end %lx nsteps %d llMode %d", args->opCount, head, tail, end, args->nsteps, llMode);
  TRACE(NCCL_NET,"opCount %lx buffSize %d sliceSize %d ptrType %d", args->opCount, buffSize, sliceSize, ptrType);

  if (llMode == 0) {
    // Waiting for next opCount is only needed before writing nextTail.
    uint64_t* nextOpCount = resources->hostDevMem ? &resources->hostDevMem->opCount : &resources->hostRecvMem->opCount;
    transportProxyWait([=] { return *nextOpCount >= args->opCount; });
  }

  while (head < end) {
    idle++;
    if ((tail < head + args->substeps) && (tail < *nextHead + args->substeps) && (tail < end)) {
      int slot = tail%args->substeps;
      NCCLCHECK(ncclNetIrecv(resources->netRecvComm, localBuff+slot*sliceSize, sliceSize, ptrType, requests+slot));
      if (requests[slot] != NULL) {
        tail++;
        idle = 0;
      }
    }
    if (tail > head) {
      int done;
      int slot = head%args->substeps;
      int size;
      NCCLCHECK(ncclNetTest(requests[slot], &done, &size));
      if (done) {
        if (nextBuff) memcpy(nextBuff+slot*sliceSize, localBuff+slot*sliceSize, size);
        head++;
        if (llMode == 0) {
          if (ptrType == NCCL_PTR_CUDA) ncclNetFlush(resources->netRecvComm, localBuff+slot*sliceSize, size);
          *nextTail = head;
        }
        idle = 0;
      }
    }
    if (idle) transportProxyIdle(idle);
  }

  // Wait for last ack and reset
  if (llMode == 0) {
    transportProxyWait([=] { return *nextHead == head; });
    *nextHead = 0;
  }

nextColl:
  if (llMode) {
    resources->llStep += args->nsteps;
    if (resources->llStep > resources->llLastCleaning + NCCL_LL_CLEAN_FREQ) {
      resources->llStep += NCCL_LL_CHUNKS;
      while (*nextHead < resources->llStep);
      resources->llLastCleaning = resources->llStep;
    }
  }
  return ncclSuccess;
}

struct ncclTransport netTransport = {
  "NET",
  netFillInfo,
  netCanConnect,
  netGetRings,
  { netSendSetup, netSendConnect, netSendFree, netSendProxy },
  { netRecvSetup, netRecvConnect, netRecvFree, netRecvProxy }
};
