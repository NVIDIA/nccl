#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <float.h>

#include "nccl.h"
#include "test_utilities.h"
#include <nvToolsExt.h>

void showUsage(const char* bin) {
  printf("\n"
         "Usage: %s <type> <op> <n_min> <n_max> [delta] [gpus] [gpu0 [gpu1 [...]]]\n"
         "Where:\n"
#ifdef CUDA_HAS_HALF
         "    type   =   [char|int|half|float|double|int64|uint64]\n"
#else
         "    type   =   [char|int|float|double|int64|uint64]\n"
#endif
         "    op     =   [sum|prod|max|min]\n"
         "    n_min  >   0\n"
         "    n_max  >=  n_min\n"
         "    delta  >   0\n\n", bin);
  return;
}

int main(int argc, char* argv[]) {
  int nvis = 0;
  CUDACHECK(hipGetDeviceCount(&nvis));
  if (nvis == 0) {
    printf("No GPUs found\n");
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  ncclDataType_t type;
  ncclRedOp_t op;
  int n_min;
  int n_max;
  int delta;
  int gpus;
  int* list = NULL;

  if (argc < 5) {
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  type = strToType(argv[1]);
  if (type == nccl_NUM_TYPES) {
    printf("Invalid <type> '%s'\n", argv[1]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  op = strToOp(argv[2]);
  if (op == nccl_NUM_OPS) {
    printf("Invalid <op> '%s'\n", argv[2]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  n_min = strToPosInt(argv[3]);
  if (n_min < 1) {
    printf("Invalid <n_min> '%s'\n", argv[3]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  n_max = strToPosInt(argv[4]);
  if (n_max < n_min) {
    printf("Invalid <n_max> '%s'\n", argv[4]);
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  if (argc > 5) {
    delta = strToPosInt(argv[5]);
    if (delta < 1) {
      printf("Invalid <delta> '%s'\n", argv[5]);
      showUsage(argv[0]);
      exit(EXIT_FAILURE);
    }
  } else {
    delta = (n_max == n_min) ? 1 : (n_max - n_min+9) / 10;
  }

  if (argc > 6) {
    gpus = strToPosInt(argv[6]);
    if (gpus < 1) {
      printf("Invalid <gpus> '%s'\n", argv[6]);
      showUsage(argv[0]);
      exit(EXIT_FAILURE);
    }
  } else {
    gpus = nvis;
  }

  list = (int*)malloc(gpus*sizeof(int));

  if (argc > 7 && argc != 7+gpus) {
    printf("If given, GPU list must be fully specified.\n");
    showUsage(argv[0]);
    exit(EXIT_FAILURE);
  }

  for(int g=0; g<gpus; ++g) {
    if(argc > 7) {
      list[g] = strToNonNeg(argv[7+g]);
      if (list[g] < 0) {
        printf("Invalid GPU%d '%s'\n", g, argv[7+g]);
        showUsage(argv[0]);
        exit(EXIT_FAILURE);
      } else if (list[g] >= nvis) {
        printf("GPU%d (%d) exceeds visible devices (%d)\n", g, list[g], nvis);
        showUsage(argv[0]);
        exit(EXIT_FAILURE);
      }
    } else {
      list[g] = g % nvis;
    }
  }

  size_t word = wordSize(type);
  size_t max_size = n_max * word;
  void* refout;
  CUDACHECK(hipHostMalloc(&refout, max_size));

  void** input;
  void* output; // always goes on rank 0
  double* maxError;
  ncclComm_t* comm;
  hipStream_t* stream;

  input = (void**)malloc(gpus*sizeof(void*));
  comm = (ncclComm_t*)malloc(gpus*sizeof(ncclComm_t));
  stream = (hipStream_t*)malloc(gpus*sizeof(hipStream_t));

  for(int g=0; g<gpus; ++g) {
    char busid[32] = {0};
    CUDACHECK(hipDeviceGetPCIBusId(busid, 32, list[g]));
    printf("# Rank %d using device %d [%s]\n", g, list[g], busid);

    CUDACHECK(hipSetDevice(list[g]));
    CUDACHECK(hipStreamCreate(&stream[g]));
    CUDACHECK(hipMalloc(&input[g],  max_size));
    makeRandom(input[g], n_max, type, 42+g);

    if (g == 0) {
      CUDACHECK(hipMalloc(&output, max_size));
      CUDACHECK(hipHostMalloc(&maxError, sizeof(double)));
      CUDACHECK(hipMemcpy(refout, input[g], max_size, hipMemcpyDeviceToHost));
    } else {
      accVec(refout, input[g], n_max, type, op);
    }
  }

  NCCLCHECK(ncclCommInitAll(comm, gpus, list));

  printf("       BYTES ERROR       MSEC     BW\n");

  for(int n=n_min; n<=n_max; n+=delta) {
    size_t bytes = word * n;

    CUDACHECK(hipSetDevice(list[0]));
    CUDACHECK(hipMemsetAsync(output, 0, bytes, stream[0]));
    for(int g=0; g<gpus; ++g)
      CUDACHECK(hipStreamSynchronize(stream[0]));

    auto start = std::chrono::high_resolution_clock::now();
    for(int g=0; g<gpus; ++g) {
      CUDACHECK(hipSetDevice(list[g]));
      NCCLCHECK(ncclReduce(input[g], output, n, type, op, 0, comm[g], stream[g]));
    }
    for(int g=0; g<gpus; ++g) {
      CUDACHECK(hipSetDevice(list[g]));
      CUDACHECK(hipStreamSynchronize(stream[g]));
    }
    auto stop = std::chrono::high_resolution_clock::now();
    double ms = std::chrono::duration_cast<std::chrono::duration<double>>
        (stop - start).count() * 1000.0;

    CUDACHECK(hipSetDevice(list[0]));
    maxDiff(maxError, output, refout, n, type, stream[0]);
    CUDACHECK(hipStreamSynchronize(stream[0]));

    double mb = (double)bytes * 1.e-6;
    double algbw = mb / ms;
    printf("%12lu %5.0le %10.3lf %6.2lf\n",
        n*word, *maxError, ms, algbw);
  }

  for(int g=0; g<gpus; ++g) {
    CUDACHECK(hipSetDevice(list[g]));
    CUDACHECK(hipStreamDestroy(stream[g]));
    ncclCommDestroy(comm[g]);
    CUDACHECK(hipFree(input[g]));
    if(g == 0) {
      CUDACHECK(hipFree(output));
      CUDACHECK(hipHostFree(maxError));
    }
  }

  free(input);
  free(comm);
  free(stream);
  CUDACHECK(hipHostFree(refout));
  exit(EXIT_SUCCESS);
}

