/*************************************************************************
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "utils.h"
#include "debug.h"
#include <unistd.h>
#include <string.h>
#include <sys/types.h>
#include <dirent.h>

ncclResult_t getHostName(char* hostname, int maxlen) {
  if (gethostname(hostname, maxlen) != 0) {
    strncpy(hostname, "unknown", maxlen);
    return ncclSystemError;
  }
  int i = 0;
  while ((hostname[i] != '.') && (hostname[i] != '\0') && (i < maxlen-1)) i++;
  hostname[i] = '\0';
  return ncclSuccess;
}

uint64_t getHash(const char* string) {
  // Based on DJB2, result = result * 33 + char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++) {
    result = ((result << 5) + result) + string[c];
  }
  return result;
}

/* Generate a hash of the unique identifying string for this host
 * that will be unique for both bare-metal and container instances
 * Equivalent of a hash of;
 *
 * $(hostname) $(readlink /proc/self/ns/\*)
 */
uint64_t getHostHash(void) {
  char uname[1024];
  // Start off with the hostname
  (void) getHostName(uname, sizeof(uname));
  int offset = strlen(uname);

  DIR *dp = opendir("/proc/self/ns");
  if (dp != NULL) {
    struct dirent *ep;
    while (ep = readdir(dp)) {
      if (offset >= sizeof(uname) - 1) {
        break;
      }
      char ns_path[1024];
      sprintf(ns_path, "/proc/self/ns/%s", ep->d_name);
      int len = readlink(ns_path, uname+offset, sizeof(uname)-1-offset);
      if (len < 0) len = 0;
      offset += len;
    }
    (void) closedir(dp);
  }

  uname[offset]='\0';
  TRACE(INIT,"unique hostname '%s'", uname);

  return getHash(uname);
}

/* Generate a hash of the unique identifying string for this process
 * that will be unique for both bare-metal and container instances
 * Equivalent of a hash of;
 *
 * $$ $(readlink /proc/self/ns/pid)
 */
uint64_t getPidHash(void) {
  char pname[1024];
  // Start off with our pid ($$)
  sprintf(pname, "%ld", (long) getpid());
  int plen = strlen(pname);
  int len = readlink("/proc/self/ns/pid", pname+plen, sizeof(pname)-1-plen);
  if (len < 0) len = 0;

  pname[plen+len]='\0';
  TRACE(INIT,"unique PID '%s'", pname);

  return getHash(pname);
}

int parseStringList(const char* string, struct netIf* ifList, int maxList) {
  if (!string) return 0;

  const char* ptr = string;
  // Ignore "^" prefix, will be detected outside of this function
  if (ptr[0] == '^') ptr++;

  int ifNum = 0;
  int ifC = 0;
  char c;
  do {
    c = *ptr;
    if (c == ':') {
      if (ifC > 0) {
        ifList[ifNum].prefix[ifC] = '\0';
        ifList[ifNum].port = atoi(ptr+1);
        ifNum++; ifC = 0;
      }
      while (c != ',' && c != '\0') c = *(++ptr);
    } else if (c == ',' || c == '\0') {
      if (ifC > 0) {
        ifList[ifNum].prefix[ifC] = '\0';
        ifList[ifNum].port = -1;
        ifNum++; ifC = 0;
      }
    } else {
      ifList[ifNum].prefix[ifC] = c;
      ifC++;
    }
    ptr++;
  } while (c);
  return ifNum;
}

static bool matchPrefix(const char* string, const char* prefix) {
  return (strncmp(string, prefix, strlen(prefix)) == 0);
}

static bool matchPort(const int port1, const int port2) {
  if (port1 == -1) return true;
  if (port2 == -1) return true;
  if (port1 == port2) return true;
  return false;
}


bool matchIfList(const char* string, int port, struct netIf* ifList, int listSize) {
  // Make an exception for the case where no user list is defined
  if (listSize == 0) return true;

  for (int i=0; i<listSize; i++) {
    if (matchPrefix(string, ifList[i].prefix)
        && matchPort(port, ifList[i].port)) {
      return true;
    }
  }
  return false;
}
