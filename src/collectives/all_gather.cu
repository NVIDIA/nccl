/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "nccl.h"
#include "core.h"
#include "common_coll.h"
#include "enqueue.h"
#include "collectives.h"

ncclResult_t ncclAllGatherFunc(const void* sendbuff, void* recvbuff, size_t count,
    ncclDataType_t datatype, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, ncclProf_t* nccl_prof) {
  size_t nbytes = count*ncclTypeSize(datatype);
  INFO(COLL,"AllGather: opCount %lx sendbuff %p recvbuff %p count %zi datatype %d op %d root %d comm %p [nranks=%d] stream %p", comm->opCount, sendbuff, recvbuff, count, datatype, op, root, comm, comm->nRanks, stream);
  if (comm->nRanks == 1) {
    if (sendbuff != recvbuff)
      CUDACHECK(hipMemcpyAsync(recvbuff, sendbuff, nbytes, hipMemcpyDeviceToDevice, stream));
  } else {
    NCCLCHECK(transportSaveProxies(ALLGATHER_SUBSTEPS, ALLGATHER_BUFCHUNKS, comm->nRanks-1, comm->nRanks, nbytes*comm->nRanks, proxyPatternRing, comm, nccl_prof));
    NCCLCHECK(saveKernel(ncclCollAllGather, sendbuff, recvbuff, nbytes, ncclInt8, op, root, comm, stream, nbytes*comm->nRanks, 1));
  }
  return ncclSuccess;
}

NCCL_API(ncclResult_t, ncclAllGather, const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream, ncclProf_t* nccl_prof);
ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount,
    ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream, ncclProf_t* nccl_prof) {
  return ncclEnqueueCheck(ncclAllGatherFunc, "AllGather", sendbuff, recvbuff, sendcount, datatype,
          ncclSum, 0, comm, stream, nccl_prof);
}
